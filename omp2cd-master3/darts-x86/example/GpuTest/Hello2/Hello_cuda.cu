#include "hip/hip_runtime.h"
//#ifdef __cplusplus
//extern "C" {
//#endif

#include <hip/hip_runtime.h>
#include <stdio.h>
extern "C"{
#include "Hello_cuda.h"
}

const int N=32;
const int M=1000;
const int blocksize =32;

__global__
void hello(int *a){
	int tx=threadIdx.x;
	printf ("tx: %d \n",tx);
	a[tx]=tx*tx;	
}

extern "C"
void Hello_cuda()
{
	size_t sz =N*sizeof(int);
	int *a= (int *)malloc(sz);
	for (int i=0;i<N;++i){
			a[i]=2;
	}
	
	int *d_addr;
	hipMalloc( (void **)&d_addr,sz );
	hipMemcpy( d_addr, a, sz, hipMemcpyHostToDevice );

	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(d_addr);
	hipMemcpy( a, d_addr, sz, hipMemcpyDeviceToHost );
	hipFree( d_addr );

	for (int i=0;i<N;++i){
			if(a[i]!= i*i){
				fprintf(stderr,"a[%d] = %d != %d !\n",i,a[i],i*i)	;
			}
	}
	free(a);

}



//#ifdef __cplusplus
//}
//#endif
