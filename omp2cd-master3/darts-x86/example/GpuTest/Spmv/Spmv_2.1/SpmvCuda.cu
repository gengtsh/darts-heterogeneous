#include "hip/hip_runtime.h"
#include "cudacommon.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include "OptionParser.h"
#include "ResultDatabase.h"
#include "Spmv.h"
#include "util.h"
#include "conf.h"
#include "hipsparse.h"

using namespace std;


texture<float, 1> vecTex;  // vector textures
texture<int2, 1>  vecTexD;

// Texture Readers (used so kernels can be templated)
struct texReaderSP {
   __device__ __forceinline__ float operator()(const int idx) const
   {
       return tex1Dfetch(vecTex, idx);
   }
};

struct texReaderDP {
   __device__ __forceinline__ double operator()(const int idx) const
   {
       int2 v = tex1Dfetch(vecTexD, idx);
#if (__CUDA_ARCH__ < 130)
       // Devices before arch 130 don't support DP, and having the
       // __hiloint2double() intrinsic will cause compilation to fail.
       // This return statement added as a workaround -- it will compile,
       // but since the arch doesn't support DP, it will never be called
       return 0;
#else
       return __hiloint2double(v.y, v.x);
#endif
   }
};

template <typename floatType>
void memcpyHostToDevice(floatType *dst, floatType *src, int size ){
    CUDA_SAFE_CALL(hipMemcpy(dst, src, size * sizeof(floatType),hipMemcpyHostToDevice));
};

template <typename floatType>
void memcpyDeviceTexture(const void* devPtr, size_t size ){

    if (sizeof(floatType) == sizeof(float))
    {
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
        CUDA_SAFE_CALL(hipBindTexture(0, vecTex, devPtr, channelDesc,size * sizeof(float)));
    }else {
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
        CUDA_SAFE_CALL(hipBindTexture(0, vecTexD, devPtr, channelDesc,size * sizeof(int2)));
    }
};

template void memcpyHostToDevice<double>(double *dst, double *src, int size );
template void memcpyHostToDevice<int>(int *dst, int *src, int size );

template void memcpyDeviceTexture<double>(const void* devPtr, size_t size );
template void memcpyDeviceTexture<int>(const void* devPtr, size_t size );


__global__ void
minusVal(int * __restrict__ a, const int val, const int n)
{
    //printf("blockIdx.x = %d, blockDim.x = %d, threadIdx.x = %d \n",blockIdx.x,blockDim.x,threadIdx.x);
    //printf("threadIdx.x = %d\n",threadIdx.x);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) a[i] -= val;

}

template <typename floatType>
__global__ void
printVal(floatType * __restrict__ a, const int n)
{
    //printf("blockIdx.x = %d, blockDim.x = %d, threadIdx.x = %d \n",blockIdx.x,blockDim.x,threadIdx.x);
    //printf("threadIdx.x = %d\n",threadIdx.x);
     
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(blockIdx.x == 0 && threadIdx.x< n){
        printf("device val[%d] = %lf\n", i,(double)a[i]);
    }
}

// Forward declarations for kernels
template <typename fpType, typename texReader>
__global__ void
shoc_spmv_csr_scalar_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out);


template <typename fpType, typename texReader>
__global__ void
shoc_spmv_csr_scalar_section_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out,
                       const int secStart);

template <typename fpType, typename texReader>
__global__ void
spmv_csr_vector_kernel(const fpType * __restrict__ val,
             	       const int    * __restrict__ cols,
		               const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out);

template <typename fpType, typename texReader>
__global__ void
spmv_csr_vector_section_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out,
                       const int secStart);


template <typename fpType, typename texReader>
__global__ void
spmv_ellpackr_kernel(const fpType * __restrict__ val,
		             const int    * __restrict__ cols,
		             const int    * __restrict__ rowLengths,
                     const int dim, fpType * __restrict__ out);

template <typename fpType>
__global__ void
zero(fpType * __restrict__ a, const int size);


template <typename floatType>
void SHOC_csrTestScalar(ResultDatabase* resultDB, OptionParser* op, CSRMM<floatType> *csrHost, CSRMM<floatType> *csrDevice ){

    int deviceStart = csrDevice->getStartPoint();
    int *h_rowDelimiters = csrHost->getRowDelimiters()+deviceStart;
    int secStart = h_rowDelimiters[0] ;
    floatType *h_val = csrHost->getVal()+secStart;
    int *h_cols = csrHost->getCols()+secStart;
    floatType *h_vec = csrHost->getVec()+deviceStart;
    floatType *h_out = csrHost->getOut()+deviceStart;

    int numRows = csrDevice->getNumRows();
    //int numNonZeroes = csrDevice->getNumNonZeroes();
    int numNonZeroes = h_rowDelimiters[numRows]-secStart;

    //std::cout<<"secStart: "<<secStart<<std::endl;

    floatType *d_val = csrDevice->getVal();
    int *d_cols = csrDevice->getCols();
    int *d_rowDelimiters = csrDevice->getRowDelimiters();
    floatType *d_vec = csrDevice->getVec();
    floatType *d_out = csrDevice->getOut();

#ifdef CUDA_RECORD
    // Setup events for timing
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));
   
    // Transfer data to device
    CUDA_SAFE_CALL(hipEventRecord(start, 0));
#endif
    CUDA_SAFE_CALL(hipMemcpy(d_val, h_val,   numNonZeroes * sizeof(floatType),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_cols, h_cols, numNonZeroes * sizeof(int),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_rowDelimiters, h_rowDelimiters,(numRows+1) * sizeof(int), hipMemcpyHostToDevice));
    
#ifdef CUDA_RECORD
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));


    float iTransferTime, oTransferTime;
    CUDA_SAFE_CALL(hipEventElapsedTime(&iTransferTime, start, stop));
    iTransferTime *= 1.e-3;
#endif
    // Bind texture for position
    string suffix;
    if (sizeof(floatType) == sizeof(float)){
        suffix = "-SP";
    }else {
        suffix = "-DP";
    }

    // Setup thread configuration
    int nBlocksScalar = (int) ceil((floatType) numRows / BLOCK_SIZE);
    int nBlocksVector = (int) ceil(numRows /(floatType)(BLOCK_SIZE / WARP_SIZE));
    int passes = op->getOptionInt("passes");
    int iters  = op->getOptionInt("iterations");
    

#ifdef CUDA_RECORD
    // Results description info
    char atts[TEMP_BUFFER_SIZE];
    sprintf(atts, "%d_elements_%d_rows",numNonZeroes, numRows);
    string prefix = "";
    double gflop = 2 * (double) numNonZeroes / 1e9;
#endif

#ifdef DARTS_DEBUG
    cout << "CSR Scalar Kernel\n";
#endif
    //cout<<"passes is : " <<passes<<", iters is "<< iters<<std::endl;

    //for (int k=0; k<passes; k++)
    //{
        // Run Scalar Kernel
    
#ifdef CUDA_RECORD
        CUDA_SAFE_CALL(hipEventRecord(start, 0));
#endif
        //for (int j = 0; j < iters; j++)
        //{
            if(suffix == "-DP"){
                shoc_spmv_csr_scalar_section_kernel<floatType, texReaderDP><<<nBlocksScalar, BLOCK_SIZE>>>
            (d_val, d_cols, d_rowDelimiters, numRows, d_out,secStart);
            }else{
                shoc_spmv_csr_scalar_section_kernel<floatType, texReaderSP><<<nBlocksScalar, BLOCK_SIZE>>>
            (d_val, d_cols, d_rowDelimiters, numRows, d_out,secStart);
            }
        //}
       
#ifdef CUDA_RECORD
        CUDA_SAFE_CALL(hipEventRecord(stop, 0));
        CUDA_SAFE_CALL(hipEventSynchronize(stop));
        float scalarKernelTime;
        CUDA_SAFE_CALL(hipEventElapsedTime(&scalarKernelTime, start, stop));
        // Transfer data back to host
        CUDA_SAFE_CALL(hipEventRecord(start, 0));
#endif
        CUDA_SAFE_CALL(hipMemcpy(h_out, d_out, numRows * sizeof(floatType),hipMemcpyDeviceToHost));
        
#ifdef CUDA_RECORD
        CUDA_SAFE_CALL(hipEventRecord(stop, 0));
        CUDA_SAFE_CALL(hipEventSynchronize(stop));
        CUDA_SAFE_CALL(hipEventElapsedTime(&oTransferTime, start, stop));
#endif
        hipDeviceSynchronize();
        
#ifdef CUDA_RECORD
        oTransferTime *= 1.e-3;

        scalarKernelTime = (scalarKernelTime / (float)iters) * 1.e-3;
        double totalTransfer = iTransferTime + oTransferTime;
        string startPoint = std::to_string(csrDevice->getStartPoint());
        string testName = prefix+"CSR-Scalar"+suffix+"-startPoint-"+startPoint;
    
        resultDB->AddResult(testName, atts, "Gflop/s",gflop/(scalarKernelTime));
        resultDB->AddResult(testName, atts, "Gflop/s",gflop / (scalarKernelTime+totalTransfer));
        //resultDB->AddResult(testName+"_PCIe", atts, "Gflop/s",gflop / (scalarKernelTime+totalTransfer));
#endif
    //}
}



template <typename floatType>
void SHOC_csrStreamTestScalar(ResultDatabase* resultDB, OptionParser* op, CSRMM<floatType> *csrHost, CSRMM<floatType> *csrDevice ){

    int deviceStart = csrDevice->getStartPoint();
    int *h_rowDelimiters = csrHost->getRowDelimiters()+deviceStart;
    int secStart = h_rowDelimiters[0] ;
    floatType *h_val = csrHost->getVal()+secStart;
    int *h_cols = csrHost->getCols()+secStart;
    //floatType *h_vec = csrHost->getVec()+deviceStart;
    //floatType *h_vec = csrHost->getVec()+secStart;
    floatType *h_out = csrHost->getOut()+deviceStart;

    int numRows = csrDevice->getNumRows();
    //int numNonZeroes = csrDevice->getNumNonZeroes();
    int numNonZeroes = h_rowDelimiters[numRows]-secStart;
#ifdef DARTS_DEBUG 
    std::cout<<"deviceStart: "<<deviceStart<<",secStart: "<<secStart<<",numRows: "<<numRows<<std::endl;
#endif
    floatType *d_val = csrDevice->getVal();
    int *d_cols = csrDevice->getCols();
    int *d_rowDelimiters = csrDevice->getRowDelimiters();
    floatType *d_vec = csrDevice->getVec();
    floatType *d_out = csrDevice->getOut();


    // Bind texture for position
    string suffix;
    if (sizeof(floatType) == sizeof(float)){
        suffix = "-SP";
    }else {
        suffix = "-DP";
    }
#ifdef DARTS_DEBUG
    cout << "CSR Stream Scalar Kernel\n";
#endif

    int nStream = 32;
    hipStream_t *stream;
    hipEvent_t  *cuEvent;
    stream = new hipStream_t[nStream];
    cuEvent = new hipEvent_t[nStream];

    int chunk = numRows/nStream;
    int *sNumRows = new int[nStream];
    int *sNumNonZeroes = new int[nStream];
    int *svcStart = new int[nStream];
    int *srStart   = new int[nStream];
    int *ssStart   = new int[nStream];
    
    // Setup thread configuration
    int *nBlocksScalar = new int[nStream]; 

    for(int i=0; i<nStream; ++i){
        //CUDA_SAFE_CALL(hipStreamCreateWithFlags(&stream[i],hipStreamNonBlocking));
        CUDA_SAFE_CALL(hipStreamCreate(&stream[i]));
        //CUDA_SAFE_CALL(hipEventCreate(&cuEvent[i]));
        CUDA_SAFE_CALL(hipEventCreateWithFlags(&cuEvent[i],hipEventDisableTiming));
    
        sNumRows[i]     = (i==(nStream-1))?(numRows-i*chunk):(chunk);
        sNumNonZeroes[i]= (i==(nStream-1))?(h_rowDelimiters[numRows]-h_rowDelimiters[i*chunk]):(h_rowDelimiters[(i+1)*chunk]-h_rowDelimiters[i*chunk]);
        svcStart[i]    = h_rowDelimiters[i*chunk]-h_rowDelimiters[0] ;
        srStart[i]     = i*chunk;
        ssStart[i]     = h_rowDelimiters[i*chunk];
        nBlocksScalar[i] = (int) ceil((floatType) sNumRows[i] / BLOCK_SIZE);

    }

    for(int i=0; i<nStream; ++i){
#ifdef DARTS_DEBUG
        std::cout<<"stream: "<<i<<std::endl;
        std::cout<<"svcStart["<<i<<"] = "<<svcStart[i]<<",svcStart["<<i<<"] = "<<svcStart[i]<<",srStart["<<i<<"] = "<<srStart[i]<<",sNumRows["<<i<<"] = "<<sNumRows[i]<<",sNumNonZeroes["<<i<<"] = "<<sNumNonZeroes[i]<<std::endl;
        std::cout<<"d_val addr:  "<<d_val<<std::endl; 
#endif
        CUDA_SAFE_CALL(hipMemcpyAsync(d_val + svcStart[i] , h_val+svcStart[i],   sNumNonZeroes[i] * sizeof(floatType),hipMemcpyHostToDevice,stream[i]));
        CUDA_SAFE_CALL(hipMemcpyAsync(d_cols+ svcStart[i], h_cols+svcStart[i], sNumNonZeroes[i] * sizeof(int),hipMemcpyHostToDevice,stream[i]));
        CUDA_SAFE_CALL(hipMemcpyAsync(d_rowDelimiters + srStart[i], h_rowDelimiters+srStart[i],(sNumRows[i]+1) * sizeof(int), hipMemcpyHostToDevice,stream[i]));

        if(suffix == "-DP"){
            shoc_spmv_csr_scalar_section_kernel<floatType, texReaderDP><<<nBlocksScalar[i], BLOCK_SIZE, 0, stream[i]>>>
        (d_val+svcStart[i], d_cols+svcStart[i], d_rowDelimiters+srStart[i], sNumRows[i], d_out+srStart[i],ssStart[i]);
        }else{
            shoc_spmv_csr_scalar_section_kernel<floatType, texReaderSP><<<nBlocksScalar[i], BLOCK_SIZE,0, stream[i]>>>
        (d_val+svcStart[i], d_cols+svcStart[i], d_rowDelimiters+srStart[i], sNumRows[i], d_out+srStart[i],ssStart[i]);
        }
        CUDA_SAFE_CALL(hipMemcpyAsync(h_out+srStart[i], d_out+srStart[i], sNumRows[i] * sizeof(floatType),hipMemcpyDeviceToHost,stream[i]));


    }

    //CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    //std::cout<<"h_out addr: "<<h_out<<",h_out[1]="<<h_out[1]<<std::endl; 
    
    delete [] sNumRows ;
    delete [] sNumNonZeroes ;
    delete [] svcStart;
    delete [] srStart; 
    delete [] ssStart; 
    delete [] nBlocksScalar ; 

    for(int i=0; i<nStream; ++i){
        CUDA_SAFE_CALL(hipStreamDestroy(stream[i]));
        CUDA_SAFE_CALL(hipEventDestroy(cuEvent[i]));
    }
    delete [] stream;
    delete [] cuEvent;


#ifdef CUDA_RECORD

#endif
}
    

template <typename floatType>
void CuSparse_csrTest(ResultDatabase* resultDB, OptionParser* op, CSRMM<floatType> *csrHost, CSRMM<floatType> *csrDevice ){

    int deviceStart = csrDevice->getStartPoint();
    int *h_rowDelimiters = csrHost->getRowDelimiters()+deviceStart;
    int secStart = h_rowDelimiters[0] ;
    floatType *h_val = csrHost->getVal()+secStart;
    int *h_cols = csrHost->getCols()+secStart;
    floatType *h_vec = csrHost->getVec()+deviceStart;
    floatType *h_out = csrHost->getOut()+deviceStart;

    int numRows = csrDevice->getNumRows();
    //int numNonZeroes = csrDevice->getNumNonZeroes();
    int numNonZeroes = h_rowDelimiters[numRows]-secStart;

    //std::cout<<"secStart: "<<secStart<<std::endl;

    floatType *d_val = csrDevice->getVal();
    int *d_cols = csrDevice->getCols();
    int *d_rowDelimiters = csrDevice->getRowDelimiters();
    floatType *d_vec = csrDevice->getVec();
    floatType *d_out = csrDevice->getOut();

#ifdef CUDA_RECORD
    // Setup events for timing
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));
   
    // Transfer data to device
    CUDA_SAFE_CALL(hipEventRecord(start, 0));
#endif
    CUDA_SAFE_CALL(hipMemcpy(d_val, h_val,   numNonZeroes * sizeof(floatType),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_cols, h_cols, numNonZeroes * sizeof(int),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_rowDelimiters, h_rowDelimiters,(numRows+1) * sizeof(int), hipMemcpyHostToDevice));
    
#ifdef CUDA_RECORD
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));


    float iTransferTime, oTransferTime;
    CUDA_SAFE_CALL(hipEventElapsedTime(&iTransferTime, start, stop));
    iTransferTime *= 1.e-3;
#endif

    // Bind texture for position
    string suffix;
    if (sizeof(floatType) == sizeof(float)){
        suffix = "-SP";
    }else {
        suffix = "-DP";
    }
    
    
    /* cusparse APIs */
    //hipsparseStatus_t status;
    int devId;
    hipDeviceProp_t prop;
    CUDA_SAFE_CALL(hipGetDevice(&devId));
    CUDA_SAFE_CALL(hipGetDeviceProperties( &prop, devId)) ;

    /* initialize cusparse library */
    hipsparseHandle_t handle=0;
    CUSPARSE_SAFE_CALL(hipsparseCreate(&handle));

#if CUDA_V10
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX,vecY;
    void* dBuffer  = NULL;
    size_t bufferSize = 0;

    hipDataType cuValueType;//for cuda v10
    if (suffix == "-SP"){
        cuValueType = HIP_R_32F;
    }else if(suffix == "-DP"){
        cuValueType = HIP_R_64F;
    }
    /*create sparse matrix A in CSR format */
    CUSPARSE_SAFE_CALL(hipsparseCreateCsr(&matA,numRows,numRows,numNonZeroes,d_rowDelimiters,d_cols,d_val,HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,cuValueType));
    /*create dense vector X */
    CUSPARSE_SAFE_CALL(hipsparseCreateDnVec(&vecX,numRows,d_vec,cuValueType));
    /*create dense vector Y */
    CUSPARSE_SAFE_CALL(hipsparseCreateDnVec(&vecY,numRows,d_out,cuValueType));

#elif CUDA_V9
    /* create and setup matrix descriptor */
    hipsparseMatDescr_t descr = 0;
    CUSPARSE_SAFE_CALL(hipsparseCreateMatDescr(&descr));
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

#endif

    const floatType alpha = 1;
    const floatType beta  = 0;

    int passes = op->getOptionInt("passes");
    int iters  = op->getOptionInt("iterations");
    

#ifdef CUDA_RECORD
    // Results description info
    char atts[TEMP_BUFFER_SIZE];
    sprintf(atts, "%d_elements_%d_rows",numNonZeroes, numRows);
    string prefix = "";
    double gflop = 2 * (double) numNonZeroes / 1e9;
#endif

#ifdef DARTS_DEBUG
    cout << "CSR (cuSparse) Scalar Kernel\n";
#endif
    
#ifdef CUDA_RECORD
        CUDA_SAFE_CALL(hipEventRecord(start, 0));
#endif

#if CUDA_V10
       // /*execuse SpMV */ 
       CUDA_SAFE_CALL(hipsparseSpMV(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,&alpha,maxA,vecX,&beta,vecY,cuValueType, HIPSPARSE_MV_ALG_DEFAULT,dBuffer)); 

#elif CUDA_V9       
        if(suffix == "-DP"){
            CUSPARSE_SAFE_CALL(hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,numRows,numRows,numNonZeroes,(const double *)&alpha,descr,(const double *)d_val,d_rowDelimiters,d_cols,(const double*)d_vec,(const double*)&beta,(double*)d_out));
        }else{
            CUSPARSE_SAFE_CALL(hipsparseScsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,numRows,numRows,numNonZeroes,(const float *)&alpha,descr,(const float *)d_val,d_rowDelimiters,d_cols,(const float*)d_vec,(const float*)&beta,(float*)d_out));
        }
#endif

#ifdef CUDA_RECORD
        CUDA_SAFE_CALL(hipEventRecord(stop, 0));
        CUDA_SAFE_CALL(hipEventSynchronize(stop));
        float scalarKernelTime;
        CUDA_SAFE_CALL(hipEventElapsedTime(&scalarKernelTime, start, stop));
        // Transfer data back to host
        CUDA_SAFE_CALL(hipEventRecord(start, 0));
#endif
        CUDA_SAFE_CALL(hipMemcpy(h_out, d_out, numRows * sizeof(floatType),hipMemcpyDeviceToHost));
        
#ifdef CUDA_RECORD
        CUDA_SAFE_CALL(hipEventRecord(stop, 0));
        CUDA_SAFE_CALL(hipEventSynchronize(stop));
        CUDA_SAFE_CALL(hipEventElapsedTime(&oTransferTime, start, stop));
#endif
        hipDeviceSynchronize();

        /* destroy handle */
        CUSPARSE_SAFE_CALL(hipsparseDestroy(handle));

#if CUDA_V10 
        /* destroy matrix/vec descriptor */
        CUSPARSE_SAFE_CALL(hipsparseDestroySpMat(matA));
        CUSPARSE_SAFE_CALL(hipsparseDestroyDnVec(vecX));
        CUSPARSE_SAFE_CALL(hipsparseDestroyDnVec(vecY));

#elif CUDA_V9
        /* destroy matrix descriptor */
        CUSPARSE_SAFE_CALL(hipsparseDestroyMatDescr(descr));

#endif

#ifdef CUDA_RECORD
        oTransferTime *= 1.e-3;

        scalarKernelTime = (scalarKernelTime / (float)iters) * 1.e-3;
        double totalTransfer = iTransferTime + oTransferTime;
        string startPoint = std::to_string(csrDevice->getStartPoint());
        string testName = prefix+"CSR-Scalar"+suffix+"-startPoint-"+startPoint;
    
        resultDB->AddResult(testName, atts, "Gflop/s",gflop/(scalarKernelTime));
        resultDB->AddResult(testName, atts, "Gflop/s",gflop / (scalarKernelTime+totalTransfer));
        //resultDB->AddResult(testName+"_PCIe", atts, "Gflop/s",gflop / (scalarKernelTime+totalTransfer));
#endif
    //}
}


template <typename floatType>
void CuSparse_csrStreamTest(ResultDatabase* resultDB, OptionParser* op, CSRMM<floatType> *csrHost, CSRMM<floatType> *csrDevice ){

    int deviceStart = csrDevice->getStartPoint();
    int *h_rowDelimiters = csrHost->getRowDelimiters()+deviceStart;
    int secStart = h_rowDelimiters[0] ;
    floatType *h_val = csrHost->getVal()+secStart;
    int *h_cols = csrHost->getCols()+secStart;
    floatType *h_vec = csrHost->getVec();
    floatType *h_out = csrHost->getOut()+deviceStart;

    int numRows = csrDevice->getNumRows();
    int numCols = csrDevice->getNumCols();
    
    //numRows = 5000;
    //int numNonZeroes = h_rowDelimiters[numRows]-secStart;
#ifdef DARTS_DEBUG 
    std::cout<<"deviceStart: "<<deviceStart<<",secStart: "<<secStart<<",numRows: "<<numRows<<std::endl;
#endif
    floatType *d_val = csrDevice->getVal();
    int *d_cols = csrDevice->getCols();
    int *d_rowDelimiters = csrDevice->getRowDelimiters();
    floatType *d_vec = csrDevice->getVec();
    floatType *d_out = csrDevice->getOut();


    // Bind texture for position
    string suffix;
    if (sizeof(floatType) == sizeof(float)){
        suffix = "-SP";
    }else {
        suffix = "-DP";
    }
#ifdef DARTS_DEBUG
    cout << "CSR Stream Scalar Kernel\n";
#endif

    int nStream = 32;// less than MAXSTREAM
    hipStream_t *stream;
    hipEvent_t  *cuEvent;
    stream = new hipStream_t[nStream];
    cuEvent = new hipEvent_t[nStream];

    int chunk = numRows/nStream;
    int *sNumRows = new int[nStream];
    int *sNumNonZeroes = new int[nStream];
    int *svcStart = new int[nStream];
    int *srStart   = new int[nStream];
    int *soStart   = new int[nStream];
    int *ssStart   = new int[nStream];
   
    floatType **d_val_sec               = new floatType*[nStream];
    floatType **d_out_sec               = new floatType*[nStream]; 
    int **d_cols_sec                    = new int*[nStream];
    int **d_rowDelimiters_sec           = new int*[nStream];

    floatType **h_val_sec               = new floatType*[nStream];
    floatType **h_out_sec               = new floatType*[nStream]; 
    int **h_cols_sec                    = new int*[nStream];
    int **h_rowDelimiters_sec           = new int*[nStream];

    int *nBlocks_sec                    = new int[nStream]; 

    /* cusparse APIs */
 
#ifdef DARTS_DEBUG
    int devId;
    hipDeviceProp_t prop;
    CUDA_SAFE_CALL(hipGetDevice(&devId));
    CUDA_SAFE_CALL(hipGetDeviceProperties( &prop, devId)) ;
    std::cout<<"cuda device async Eng count: "<<prop.asyncEngineCount<<std::endl;
#endif

#if CUDA_V10
    /* create matrix and vec descriptor */
    hipsparseSpMatDescr_t *matA = new hipsparseSpMatDescr_t [nStream];
    hipsparseDnVecDescr_t *vecY = new hipsparseDnVecDescr_t [nStream];
#endif
    
    hipsparseHandle_t *handle = new hipsparseHandle_t [nStream] ;
    for(int i=0; i<nStream; ++i){
        //CUDA_SAFE_CALL(hipStreamCreateWithFlags(&stream[i],hipStreamNonBlocking));
        //CUDA_SAFE_CALL(hipEventCreateWithFlags(&cuEvent[i],hipEventDisableTiming));
        CUDA_SAFE_CALL(hipStreamCreate(&stream[i]));
        CUDA_SAFE_CALL(hipEventCreate(&cuEvent[i]));
    
        sNumRows[i]     = (i==(nStream-1))?(numRows-i*chunk):(chunk);
        if(sNumRows[i] == 0) continue;
        sNumNonZeroes[i]= (i==(nStream-1))?(h_rowDelimiters[numRows]-h_rowDelimiters[i*chunk]):(h_rowDelimiters[(i+1)*chunk]-h_rowDelimiters[i*chunk]);
        svcStart[i]    = h_rowDelimiters[i*chunk]-h_rowDelimiters[0] ;
        srStart[i]     = i*(chunk+1);
        soStart[i]     = i*chunk;
        ssStart[i]     = h_rowDelimiters[i*chunk];


        d_val_sec[i]              = d_val + svcStart[i] ;
        d_cols_sec[i]             = d_cols + svcStart[i];
        d_rowDelimiters_sec[i]   = d_rowDelimiters + srStart[i] ;
        d_out_sec[i]              = d_out + soStart[i]; 

        h_val_sec[i]              = h_val + svcStart[i];
        h_cols_sec[i]             = h_cols + svcStart[i];
        h_rowDelimiters_sec[i]    = h_rowDelimiters + soStart[i];
        h_out_sec[i]              = h_out + soStart[i]; 
        nBlocks_sec[i]            = ceil(sNumRows[i]/(float)BLOCK_SIZE) ; 

        CUSPARSE_SAFE_CALL(hipsparseCreate(&handle[i]));
        CUSPARSE_SAFE_CALL(hipsparseSetStream(handle[i],stream[i]));

#if CUDA_V10
        /*create sparse matrix A in CSR format */
        CUSPARSE_SAFE_CALL(hipsparseCreateCsr(&matA[i],sNumRows[i],numCols,sNumNonZeroes[i],d_rowDelimiters_sec[i],d_cols_sec[i],d_val_sec[i],HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,cuValueType));
        /*create dense vector Y */
        CUSPARSE_SAFE_CALL(hipsparseCreateDnVec(&vecY[i],sNumRows[i],d_out_sec[i],cuValueType));

#endif
    }

#if CUDA_V10
    void* dBuffer  = NULL;
    hipDataType cuValueType;
    if (suffix == "-SP"){
        cuValueType = HIP_R_32F;
    }else if(suffix == "-DP"){
        cuValueType = HIP_R_64F;
    }

    /*create dense vector X */
    hipsparseDnVecDescr_t vecX;
    CUSPARSE_SAFE_CALL(hipsparseCreateDnVec(&vecX,numRows,d_vec,cuValueType));

#endif

    const floatType alpha = 1;
    const floatType beta  = 0;

    ///* initialize cusparse library */
    //hipsparseHandle_t handle ;
    //CUSPARSE_SAFE_CALL(hipsparseCreate(&handle));
    
    hipsparseMatDescr_t  descr;
    CUSPARSE_SAFE_CALL(hipsparseCreateMatDescr(&descr));
    CUSPARSE_SAFE_CALL(hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL));
    CUSPARSE_SAFE_CALL(hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO));
    int sNumRowDelimiter = 0;
    for(int i=0; i<nStream; ++i){
     
        //CUSPARSE_SAFE_CALL(hipsparseSetStream(handle,stream[i])); 
#ifdef DARTS_DEBUG
        std::cout<<"stream: "<<i<<std::endl;
        std::cout<<"svcStart["<<i<<"] = "<<svcStart[i]<<",srStart["<<i<<"] = "<<srStart[i]<<",soStart["<<i<<"] = "<<soStart[i]<<",sNumRows["<<i<<"] = "<<sNumRows[i]<<",sNumNonZeroes["<<i<<"] = "<<sNumNonZeroes[i]<<std::endl;
#endif

        CUDA_SAFE_CALL(hipMemcpyAsync(d_val + svcStart[i] , h_val+svcStart[i],   sNumNonZeroes[i] * sizeof(floatType),hipMemcpyHostToDevice,stream[i]));
        CUDA_SAFE_CALL(hipMemcpyAsync(d_cols+ svcStart[i], h_cols+svcStart[i], sNumNonZeroes[i] * sizeof(int),hipMemcpyHostToDevice,stream[i]));
        CUDA_SAFE_CALL(hipMemcpyAsync(d_rowDelimiters + srStart[i], h_rowDelimiters+soStart[i],(sNumRows[i]+1) * sizeof(int), hipMemcpyHostToDevice,stream[i]));
        
        minusVal<<<nBlocks_sec[i],BLOCK_SIZE,0,stream[i]>>>(d_rowDelimiters_sec[i],ssStart[i],sNumRows[i]+1); 


#if CUDA_V10
       // /*execuse SpMV */ 
       CUDA_SAFE_CALL(hipsparseSpMV(handle[i],HIPSPARSE_OPERATION_NON_TRANSPOSE,&alpha,maxA[i],vecX,&beta,vecY[i],cuValueType, HIPSPARSE_MV_ALG_DEFAULT,dBuffer)); 

#elif CUDA_V9

        if(suffix == "-DP"){
            CUSPARSE_SAFE_CALL(hipsparseDcsrmv(handle[i],HIPSPARSE_OPERATION_NON_TRANSPOSE,sNumRows[i],numCols,sNumNonZeroes[i],(const double *)&alpha,descr,(const double *)(d_val_sec[i]),d_rowDelimiters_sec[i],d_cols_sec[i],(const double*)d_vec,(const double*)&beta,(double*)(d_out_sec[i])));
        }else{
            CUSPARSE_SAFE_CALL(hipsparseScsrmv(handle[i],HIPSPARSE_OPERATION_NON_TRANSPOSE,sNumRows[i],numCols,sNumNonZeroes[i],(const float *)&alpha,descr,(const float *)(d_val_sec[i]),d_rowDelimiters_sec[i],d_cols_sec[i],(const float*)d_vec,(const float*)&beta,(float*)(d_out_sec[i])));
        }
#endif

#ifdef DARTS_DEBUG 
        ////CUDA_SAFE_CALL(hipDeviceSynchronize());
        //int num = 20;
        //printVal<floatType><<<nBlocks,BLOCK_SIZE,0, stream[i]>>>(d_val+svcStart[i], num);
        //printVal<int><<<nBlocks,BLOCK_SIZE,0,stream[i]>>>(d_cols+svcStart[i], num);
        //printVal<int><<<nBlocks,BLOCK_SIZE,0,stream[i]>>>(d_rowDelimiters+srStart[i], num);
        //
        //for(int j=0; j<num; ++j){
        //    printVal<floatType><<<nBlocks,BLOCK_SIZE,0,stream[i]>>>(d_vec+(h_cols+svcStart[i])[j] , 1);
        //}
        //printVal<floatType><<<nBlocks,BLOCK_SIZE,0,stream[i]>>>(d_out+soStart[i], num);

#endif
        //CUDA_SAFE_CALL(hipEventRecord(cuEvent[i],stream[i]));
        //CUDA_SAFE_CALL(hipStreamWaitEvent(stream[i],cuEvent[i],0));
        
        CUDA_SAFE_CALL(hipMemcpyAsync(h_out_sec[i], d_out_sec[i], sNumRows[i] * sizeof(floatType),hipMemcpyDeviceToHost,stream[i]));

#ifdef DARTS_DEBUG 
        //CUDA_SAFE_CALL(hipDeviceSynchronize());
        //for(int j=0; j<10; ++j){
        //    std::cout<<"h_out["<<j<<"] = "<<(h_out+soStart[i])[j]<<std::endl;
        //}
#endif

    }

    //CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipDeviceSynchronize());

#ifdef DARTS_DEBUG 
    //std::cout<<"h_out[0] = "<< h_out[0]<<",h_out[1] = "<< h_out[1]<<", h_out[2] = "<<h_out[2]<<std::endl;
#endif

#if CUDA_V10 
    /* destroy matrix/vec descriptor */
    CUSPARSE_SAFE_CALL(hipsparseDestroyDnVec(vecX));
    for(int i =0; i<nStream;++i){ 
        CUSPARSE_SAFE_CALL(hipsparseDestroySpMat(matA[i]));
        CUSPARSE_SAFE_CALL(hipsparseDestroyDnVec(vecY[i]));
    }
    delete [] matA;
    delete [] vecY;
#elif CUDA_V9
    /* destroy matrix descriptor */
    CUSPARSE_SAFE_CALL(hipsparseDestroyMatDescr(descr));
    
#endif

    /* destroy handle */
    //CUSPARSE_SAFE_CALL(hipsparseDestroy(handle));
    for(int i =0; i<nStream;++i){ 
        CUSPARSE_SAFE_CALL(hipsparseDestroy(handle[i]));
    }
    delete [] handle;
    delete [] sNumRows ;
    delete [] sNumNonZeroes ;
    delete [] svcStart;
    delete [] srStart; 
    delete [] ssStart; 
    delete [] d_val_sec;           
    delete [] d_cols_sec;          
    delete [] d_rowDelimiters_sec;
    delete [] d_out_sec;          

    delete [] h_val_sec;           
    delete [] h_cols_sec;           
    delete [] h_rowDelimiters_sec;
    delete [] h_out_sec;           
    delete [] nBlocks_sec; 
    
    for(int i=0; i<nStream; ++i){
        CUDA_SAFE_CALL(hipStreamDestroy(stream[i]));
        CUDA_SAFE_CALL(hipEventDestroy(cuEvent[i]));
    }
    delete [] stream;
    delete [] cuEvent;


#ifdef CUDA_RECORD

#endif
}



template <typename floatType>
void csrTestVector(ResultDatabase* resultDB, OptionParser* op, CSRMM<floatType> *csrHost, CSRMM<floatType> *csrDevice ){

    int deviceStart = csrDevice->getStartPoint();
    int *h_rowDelimiters = csrHost->getRowDelimiters()+deviceStart;
    int secStart = h_rowDelimiters[0] ;
    floatType *h_val = csrHost->getVal()+secStart;
    int *h_cols = csrHost->getCols()+secStart;
    floatType *h_vec = csrHost->getVec()+deviceStart;
    floatType *h_out = csrHost->getOut()+deviceStart;

    int numRows = csrDevice->getNumRows();
    int numNonZeroes = csrDevice->getNumNonZeroes();

    //std::cout<<"secStart: "<<secStart<<std::endl;

    floatType *d_val = csrDevice->getVal();
    int *d_cols = csrDevice->getCols();
    int *d_rowDelimiters = csrDevice->getRowDelimiters();
    floatType *d_vec = csrDevice->getVec();
    floatType *d_out = csrDevice->getOut();

    // Setup events for timing
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));
    
    // Transfer data to device
    CUDA_SAFE_CALL(hipEventRecord(start, 0));
    CUDA_SAFE_CALL(hipMemcpy(d_val, h_val,   numNonZeroes * sizeof(floatType),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_cols, h_cols, numNonZeroes * sizeof(int),hipMemcpyHostToDevice));
   // CUDA_SAFE_CALL(hipMemcpy(d_vec, h_vec, numRows * sizeof(floatType),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_rowDelimiters, h_rowDelimiters,(numRows+1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));


    float iTransferTime, oTransferTime;
    CUDA_SAFE_CALL(hipEventElapsedTime(&iTransferTime, start, stop));
    iTransferTime *= 1.e-3;
    // Bind texture for position
    string suffix;
    if (sizeof(floatType) == sizeof(float)){
     //   hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
     //   CUDA_SAFE_CALL(hipBindTexture(0, vecTex, d_vec, channelDesc,numRows * sizeof(float)));
        suffix = "-SP";

    }
    else {
    //    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
    //    CUDA_SAFE_CALL(hipBindTexture(0, vecTexD, d_vec, channelDesc,numRows * sizeof(int2)));
        suffix = "-DP";
    }

    // Setup thread configuration
    int nBlocksScalar = (int) ceil((floatType) numRows / BLOCK_SIZE);
    int nBlocksVector = (int) ceil(numRows /(floatType)(BLOCK_SIZE / WARP_SIZE));
    int passes = op->getOptionInt("passes");
    int iters  = op->getOptionInt("iterations");
    
    // Results description info
    char atts[TEMP_BUFFER_SIZE];
    sprintf(atts, "%d_elements_%d_rows", numNonZeroes, numRows);
    string prefix = "";
    double gflop = 2 * (double) numNonZeroes / 1e9;
    cout << "CSR vector Kernel\n";

    //cout<<"passes is : " <<passes<<", iters is "<< iters<<std::endl;

    //for (int k=0; k<passes; k++)
    //{
        // Run Scalar Kernel
        CUDA_SAFE_CALL(hipEventRecord(start, 0));
        //for (int j = 0; j < iters; j++)
        //{
            if(suffix == "-DP"){
                spmv_csr_vector_section_kernel<floatType, texReaderDP><<<nBlocksVector, BLOCK_SIZE>>>
            (d_val, d_cols, d_rowDelimiters, numRows, d_out,secStart);
            }else{
                spmv_csr_vector_section_kernel<floatType, texReaderSP><<<nBlocksScalar, BLOCK_SIZE>>>
            (d_val, d_cols, d_rowDelimiters, numRows, d_out,secStart);
            }
        //}
        CUDA_SAFE_CALL(hipEventRecord(stop, 0));
        CUDA_SAFE_CALL(hipEventSynchronize(stop));
        float vectorKernelTime;
        CUDA_SAFE_CALL(hipEventElapsedTime(&vectorKernelTime, start, stop));
        // Transfer data back to host
        CUDA_SAFE_CALL(hipEventRecord(start, 0));
        CUDA_SAFE_CALL(hipMemcpy(h_out, d_out, numRows * sizeof(floatType),hipMemcpyDeviceToHost));
        CUDA_SAFE_CALL(hipEventRecord(stop, 0));
        CUDA_SAFE_CALL(hipEventSynchronize(stop));
        CUDA_SAFE_CALL(hipEventElapsedTime(&oTransferTime, start, stop));
        hipDeviceSynchronize();
        
        vectorKernelTime = (vectorKernelTime / (float)iters) * 1.e-3;
        string testName = prefix+"CSR-Vector"+suffix;
        double totalTransfer = iTransferTime + oTransferTime;
        
        resultDB->AddResult(testName, atts, "Gflop/s",gflop/(vectorKernelTime));
        resultDB->AddResult(testName+"_PCIe", atts, "Gflop/s",gflop / (vectorKernelTime+totalTransfer));
    
    //}
}


// ****************************************************************************
// Function: shoc_spmv_csr_scalar_kernel
//
// Purpose:
//   Computes sparse matrix - vector multiplication on the GPU using
//   the CSR data storage format, using a thread per row of the sparse
//   matrix; based on Bell (SC09) and Baskaran (IBM Tech Report)
//
// Arguments:
//   val: array holding the non-zero values for the matrix
//   cols: array of column indices for each element of the sparse matrix
//   rowDelimiters: array of size dim+1 holding indices to rows of the matrix
//                  last element is the index one past the last
//                  element of the matrix
//   dim: number of rows in the matrix
//   out: output - result from the spmv calculation
//
// Returns:  nothing
//           out indirectly through a pointer
//
// Programmer: Lukasz Wesolowski
// Creation: June 28, 2010
//
// Modifications:
//
// ****************************************************************************
template <typename fpType, typename texReader>
__global__ void
shoc_spmv_csr_scalar_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out)
{
    int myRow = blockIdx.x * blockDim.x + threadIdx.x;
    texReader vecTexReader;
    if (myRow < dim)
    {
        fpType t = 0.0f;
        int start = rowDelimiters[myRow];
        int end = rowDelimiters[myRow+1];
        for (int j = start; j < end; j++)
        {
            int col = cols[j];
            t += val[j] * vecTexReader(col);
#ifdef DARTS_DEBUG
            if(threadIdx.x <20&&blockIdx.x ==0){
                printf("val[%d]=%lf, vecTexReader(%d)=%lf\n",j,val[j],col,vecTexReader(col));
            }
#endif
        }
        out[myRow] = t;
    }
}



template <typename fpType, typename texReader>
__global__ void
shoc_spmv_csr_scalar_section_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out,
                       const int secStart)
{
    int myRow = blockIdx.x * blockDim.x + threadIdx.x;
    texReader vecTexReader;
    if (myRow < dim)
    {
        fpType t = 0.0f;
        int start = rowDelimiters[myRow]-secStart;
        int end = rowDelimiters[myRow+1]-secStart;
        for (int j = start; j < end; j++)
        {
            int col = cols[j];
            t += val[j] * vecTexReader(col);

#ifdef DARTS_DEBUG
            //if(threadIdx.x <10&&blockIdx.x ==0){
            //    printf("val[%ld]=%g, vecTexReader(%ld)=%g\n",j,val[j],col,vecTexReader(col));
            //}
#endif
        }
        out[myRow] = t;

#ifdef DARTS_DEBUG
            //if(threadIdx.x <10&&blockIdx.x ==0){
            //    printf("out[%ld]=%g\n",out[myRow]);
            //}
#endif
    }
}


// ****************************************************************************
// Function: spmv_csr_vector_kernel
//
// Purpose:
//   Computes sparse matrix - vector multiplication on the GPU using
//   the CSR data storage format, using a warp per row of the sparse
//   matrix; based on Bell (SC09) and Baskaran (IBM Tech Report)
//
// Arguments:
//   val: array holding the non-zero values for the matrix
//   cols: array of column indices for each element of the sparse matrix
//   rowDelimiters: array of size dim+1 holding indices to rows of the matrix
//                  last element is the index one past the last
//                  element of the matrix
//   dim: number of rows in the matrix
//   out: output - result from the spmv calculation
//
// Returns:  nothing
//           out indirectly through a pointer
//
// Programmer: Lukasz Wesolowski
// Creation: June 28, 2010
//
// Modifications:
//
// ****************************************************************************
template <typename fpType, typename texReader>
__global__ void
spmv_csr_vector_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out)
{
    // Thread ID in block
    int t = threadIdx.x;
    // Thread ID within warp
    int id = t & (warpSize-1);
    int warpsPerBlock = blockDim.x / warpSize;
    // One row per warp
    int myRow = (blockIdx.x * warpsPerBlock) + (t / warpSize);
    // Texture reader for the dense vector
    texReader vecTexReader;

    __shared__ volatile fpType partialSums[BLOCK_SIZE];

    if (myRow < dim)
    {
        int warpStart = rowDelimiters[myRow];
        int warpEnd = rowDelimiters[myRow+1];
        fpType mySum = 0;
        for (int j = warpStart + id; j < warpEnd; j += warpSize)
        {
            int col = cols[j];
            mySum += val[j] * vecTexReader(col);
        }
        partialSums[t] = mySum;

        // Reduce partial sums
        if (id < 16) partialSums[t] += partialSums[t+16];
        if (id <  8) partialSums[t] += partialSums[t+ 8];
        if (id <  4) partialSums[t] += partialSums[t+ 4];
        if (id <  2) partialSums[t] += partialSums[t+ 2];
        if (id <  1) partialSums[t] += partialSums[t+ 1];

        // Write result
        if (id == 0)
        {
            out[myRow] = partialSums[t];
        }
    }
}



template <typename fpType, typename texReader>
__global__ void
spmv_csr_vector_section_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out,
                       const int secStart)
{
    // Thread ID in block
    int t = threadIdx.x;
    // Thread ID within warp
    int id = t & (warpSize-1);
    int warpsPerBlock = blockDim.x / warpSize;
    // One row per warp
    int myRow = (blockIdx.x * warpsPerBlock) + (t / warpSize);
    // Texture reader for the dense vector
    texReader vecTexReader;

    __shared__ volatile fpType partialSums[BLOCK_SIZE];

    if (myRow < dim)
    {
        int warpStart = rowDelimiters[myRow]-secStart;
        int warpEnd = rowDelimiters[myRow+1]-secStart;
        fpType mySum = 0;
        for (int j = warpStart + id; j < warpEnd; j += warpSize)
        {
            int col = cols[j];
            mySum += val[j] * vecTexReader(col);
        }
        partialSums[t] = mySum;

        // Reduce partial sums
        if (id < 16) partialSums[t] += partialSums[t+16];
        if (id <  8) partialSums[t] += partialSums[t+ 8];
        if (id <  4) partialSums[t] += partialSums[t+ 4];
        if (id <  2) partialSums[t] += partialSums[t+ 2];
        if (id <  1) partialSums[t] += partialSums[t+ 1];

        // Write result
        if (id == 0)
        {
            out[myRow] = partialSums[t];
        }
    }
}



// ****************************************************************************
// Function: spmv_ellpackr_kernel
//
// Purpose:
//   Computes sparse matrix - vector multiplication on the GPU using
//   the ELLPACK-R data storage format; based on Vazquez et al (Univ. of
//   Almeria Tech Report 2009)
//
// Arguments:
//   val: array holding the non-zero values for the matrix in column
//   major format and padded with zeros up to the length of longest row
//   cols: array of column indices for each element of the sparse matrix
//   rowLengths: array storing the length of each row of the sparse matrix
//   dim: number of rows in the matrix
//   out: output - result from the spmv calculation
//
// Returns:  nothing directly
//           out indirectly through a pointer
//
// Programmer: Lukasz Wesolowski
// Creation: June 29, 2010
//
// Modifications:
//
// ****************************************************************************
template <typename fpType, typename texReader>
__global__ void
spmv_ellpackr_kernel(const fpType * __restrict__ val,
                     const int    * __restrict__ cols,
                     const int    * __restrict__ rowLengths,
                     const int dim, fpType * __restrict__ out)
{
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    texReader vecTexReader;

    if (t < dim)
    {
        fpType result = 0.0f;
        int max = rowLengths[t];
        for (int i = 0; i < max; i++)
        {
            int ind = i*dim+t;
            result += val[ind] * vecTexReader(cols[ind]);
        }
        out[t] = result;
    }
}

template <typename fpType>
__global__ void
zero(fpType * __restrict__ a, const int size)
{
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    if (t < size) a[t] = 0;
}


template void SHOC_csrTestScalar<double>(ResultDatabase* resultDB, OptionParser* op, CSRMM<double> *csrHost, CSRMM<double> *csrDevice );
template void SHOC_csrTestScalar<float>(ResultDatabase* resultDB, OptionParser* op, CSRMM<float> *csrHost, CSRMM<float> *csrDevice );


template void SHOC_csrStreamTestScalar<double>(ResultDatabase* resultDB, OptionParser* op, CSRMM<double> *csrHost, CSRMM<double> *csrDevice );
template void SHOC_csrStreamTestScalar<float>(ResultDatabase* resultDB, OptionParser* op, CSRMM<float> *csrHost, CSRMM<float> *csrDevice );




template void CuSparse_csrTest<double>(ResultDatabase* resultDB, OptionParser* op, CSRMM<double> *csrHost, CSRMM<double> *csrDevice );
template void CuSparse_csrTest<float>(ResultDatabase* resultDB, OptionParser* op, CSRMM<float> *csrHost, CSRMM<float> *csrDevice );

template void CuSparse_csrStreamTest<double>(ResultDatabase* resultDB, OptionParser* op, CSRMM<double> *csrHost, CSRMM<double> *csrDevice );
//template void CuSparse_csrStreamTest<float>(ResultDatabase* resultDB, OptionParser* op, CSRMM<float> *csrHost, CSRMM<float> *csrDevice );


template void csrTestVector<double>(ResultDatabase* resultDB, OptionParser* op, CSRMM<double> *csrHost, CSRMM<double> *csrDevice );
template void csrTestVector<float>(ResultDatabase* resultDB, OptionParser* op, CSRMM<float> *csrHost, CSRMM<float> *csrDevice );
