#include "hip/hip_runtime.h"
#include "cudacommon.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include "OptionParser.h"
#include "ResultDatabase.h"
#include "Spmv.h"
#include "util.h"
#include "conf.h"

using namespace std;


texture<float, 1> vecTex;  // vector textures
texture<int2, 1>  vecTexD;

// Texture Readers (used so kernels can be templated)
struct texReaderSP {
   __device__ __forceinline__ float operator()(const int idx) const
   {
       return tex1Dfetch(vecTex, idx);
   }
};

struct texReaderDP {
   __device__ __forceinline__ double operator()(const int idx) const
   {
       int2 v = tex1Dfetch(vecTexD, idx);
#if (__CUDA_ARCH__ < 130)
       // Devices before arch 130 don't support DP, and having the
       // __hiloint2double() intrinsic will cause compilation to fail.
       // This return statement added as a workaround -- it will compile,
       // but since the arch doesn't support DP, it will never be called
       return 0;
#else
       return __hiloint2double(v.y, v.x);
#endif
   }
};

template <typename floatType>
void memcpyHostToDevice(floatType *dst, floatType *src, int size ){
    CUDA_SAFE_CALL(hipMemcpy(dst, src, size * sizeof(floatType),hipMemcpyHostToDevice));
};

template <typename floatType>
void memcpyDeviceTexture(const void* devPtr, size_t size ){

    if (sizeof(floatType) == sizeof(float))
    {
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
        CUDA_SAFE_CALL(hipBindTexture(0, vecTex, devPtr, channelDesc,size * sizeof(float)));
    }else {
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
        CUDA_SAFE_CALL(hipBindTexture(0, vecTexD, devPtr, channelDesc,size * sizeof(int2)));
    }
};

template void memcpyHostToDevice<double>(double *dst, double *src, int size );
template void memcpyHostToDevice<int>(int *dst, int *src, int size );

template void memcpyDeviceTexture<double>(const void* devPtr, size_t size );
template void memcpyDeviceTexture<int>(const void* devPtr, size_t size );


// Forward declarations for kernels
template <typename fpType, typename texReader>
__global__ void
spmv_csr_scalar_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out);


template <typename fpType, typename texReader>
__global__ void
spmv_csr_scalar_section_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out,
                       const int secStart);

template <typename fpType, typename texReader>
__global__ void
spmv_csr_vector_kernel(const fpType * __restrict__ val,
             	       const int    * __restrict__ cols,
		               const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out);

template <typename fpType, typename texReader>
__global__ void
spmv_csr_vector_section_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out,
                       const int secStart);


template <typename fpType, typename texReader>
__global__ void
spmv_ellpackr_kernel(const fpType * __restrict__ val,
		             const int    * __restrict__ cols,
		             const int    * __restrict__ rowLengths,
                     const int dim, fpType * __restrict__ out);

template <typename fpType>
__global__ void
zero(fpType * __restrict__ a, const int size);


template <typename floatType>
void csrTestScalar(ResultDatabase* resultDB, OptionParser* op, CSRMM<floatType> *csrHost, CSRMM<floatType> *csrDevice ){

    int deviceStart = csrDevice->getStartPoint();
    int *h_rowDelimiters = csrHost->getRowDelimiters()+deviceStart;
    int secStart = h_rowDelimiters[0] ;
    floatType *h_val = csrHost->getVal()+secStart;
    int *h_cols = csrHost->getCols()+secStart;
    floatType *h_vec = csrHost->getVec()+deviceStart;
    floatType *h_out = csrHost->getOut()+deviceStart;

    int numRows = csrDevice->getNumRows();
    //int numNonZeroes = csrDevice->getNumNonZeroes();
    int numNonZeroes = h_rowDelimiters[numRows]-secStart;

    //std::cout<<"secStart: "<<secStart<<std::endl;

    floatType *d_val = csrDevice->getVal();
    int *d_cols = csrDevice->getCols();
    int *d_rowDelimiters = csrDevice->getRowDelimiters();
    floatType *d_vec = csrDevice->getVec();
    floatType *d_out = csrDevice->getOut();

#ifdef CUDA_RECORD
    // Setup events for timing
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));
   
    // Transfer data to device
    CUDA_SAFE_CALL(hipEventRecord(start, 0));
#endif
    CUDA_SAFE_CALL(hipMemcpy(d_val, h_val,   numNonZeroes * sizeof(floatType),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_cols, h_cols, numNonZeroes * sizeof(int),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_rowDelimiters, h_rowDelimiters,(numRows+1) * sizeof(int), hipMemcpyHostToDevice));
    
#ifdef CUDA_RECORD
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));


    float iTransferTime, oTransferTime;
    CUDA_SAFE_CALL(hipEventElapsedTime(&iTransferTime, start, stop));
    iTransferTime *= 1.e-3;
#endif
    // Bind texture for position
    string suffix;
    if (sizeof(floatType) == sizeof(float)){
        suffix = "-SP";
    }else {
        suffix = "-DP";
    }

    // Setup thread configuration
    int nBlocksScalar = (int) ceil((floatType) numRows / BLOCK_SIZE);
    int nBlocksVector = (int) ceil(numRows /(floatType)(BLOCK_SIZE / WARP_SIZE));
    int passes = op->getOptionInt("passes");
    int iters  = op->getOptionInt("iterations");
    

#ifdef CUDA_RECORD
    // Results description info
    char atts[TEMP_BUFFER_SIZE];
    sprintf(atts, "%d_elements_%d_rows",numNonZeroes, numRows);
    string prefix = "";
    double gflop = 2 * (double) numNonZeroes / 1e9;
#endif

#ifdef DARTS_DEBUG
    cout << "CSR Scalar Kernel\n";
#endif
    //cout<<"passes is : " <<passes<<", iters is "<< iters<<std::endl;

    //for (int k=0; k<passes; k++)
    //{
        // Run Scalar Kernel
    
#ifdef CUDA_RECORD
        CUDA_SAFE_CALL(hipEventRecord(start, 0));
#endif
        //for (int j = 0; j < iters; j++)
        //{
            if(suffix == "-DP"){
                spmv_csr_scalar_section_kernel<floatType, texReaderDP><<<nBlocksScalar, BLOCK_SIZE>>>
            (d_val, d_cols, d_rowDelimiters, numRows, d_out,secStart);
            }else{
                spmv_csr_scalar_section_kernel<floatType, texReaderSP><<<nBlocksScalar, BLOCK_SIZE>>>
            (d_val, d_cols, d_rowDelimiters, numRows, d_out,secStart);
            }
        //}
       
#ifdef CUDA_RECORD
        CUDA_SAFE_CALL(hipEventRecord(stop, 0));
        CUDA_SAFE_CALL(hipEventSynchronize(stop));
        float scalarKernelTime;
        CUDA_SAFE_CALL(hipEventElapsedTime(&scalarKernelTime, start, stop));
        // Transfer data back to host
        CUDA_SAFE_CALL(hipEventRecord(start, 0));
#endif
        CUDA_SAFE_CALL(hipMemcpy(h_out, d_out, numRows * sizeof(floatType),hipMemcpyDeviceToHost));
        
#ifdef CUDA_RECORD
        CUDA_SAFE_CALL(hipEventRecord(stop, 0));
        CUDA_SAFE_CALL(hipEventSynchronize(stop));
        CUDA_SAFE_CALL(hipEventElapsedTime(&oTransferTime, start, stop));
#endif
        hipDeviceSynchronize();
        
#ifdef CUDA_RECORD
        oTransferTime *= 1.e-3;

        scalarKernelTime = (scalarKernelTime / (float)iters) * 1.e-3;
        double totalTransfer = iTransferTime + oTransferTime;
        string startPoint = std::to_string(csrDevice->getStartPoint());
        string testName = prefix+"CSR-Scalar"+suffix+"-startPoint-"+startPoint;
    
        resultDB->AddResult(testName, atts, "Gflop/s",gflop/(scalarKernelTime));
        resultDB->AddResult(testName, atts, "Gflop/s",gflop / (scalarKernelTime+totalTransfer));
        //resultDB->AddResult(testName+"_PCIe", atts, "Gflop/s",gflop / (scalarKernelTime+totalTransfer));
#endif
    //}
}


template <typename floatType>
void csrTestVector(ResultDatabase* resultDB, OptionParser* op, CSRMM<floatType> *csrHost, CSRMM<floatType> *csrDevice ){

    int deviceStart = csrDevice->getStartPoint();
    int *h_rowDelimiters = csrHost->getRowDelimiters()+deviceStart;
    int secStart = h_rowDelimiters[0] ;
    floatType *h_val = csrHost->getVal()+secStart;
    int *h_cols = csrHost->getCols()+secStart;
    floatType *h_vec = csrHost->getVec()+deviceStart;
    floatType *h_out = csrHost->getOut()+deviceStart;

    int numRows = csrDevice->getNumRows();
    int numNonZeroes = csrDevice->getNumNonZeroes();

    //std::cout<<"secStart: "<<secStart<<std::endl;

    floatType *d_val = csrDevice->getVal();
    int *d_cols = csrDevice->getCols();
    int *d_rowDelimiters = csrDevice->getRowDelimiters();
    floatType *d_vec = csrDevice->getVec();
    floatType *d_out = csrDevice->getOut();

    // Setup events for timing
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));
    
    // Transfer data to device
    CUDA_SAFE_CALL(hipEventRecord(start, 0));
    CUDA_SAFE_CALL(hipMemcpy(d_val, h_val,   numNonZeroes * sizeof(floatType),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_cols, h_cols, numNonZeroes * sizeof(int),hipMemcpyHostToDevice));
   // CUDA_SAFE_CALL(hipMemcpy(d_vec, h_vec, numRows * sizeof(floatType),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_rowDelimiters, h_rowDelimiters,(numRows+1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));


    float iTransferTime, oTransferTime;
    CUDA_SAFE_CALL(hipEventElapsedTime(&iTransferTime, start, stop));
    iTransferTime *= 1.e-3;
    // Bind texture for position
    string suffix;
    if (sizeof(floatType) == sizeof(float)){
     //   hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
     //   CUDA_SAFE_CALL(hipBindTexture(0, vecTex, d_vec, channelDesc,numRows * sizeof(float)));
        suffix = "-SP";

    }
    else {
    //    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
    //    CUDA_SAFE_CALL(hipBindTexture(0, vecTexD, d_vec, channelDesc,numRows * sizeof(int2)));
        suffix = "-DP";
    }

    // Setup thread configuration
    int nBlocksScalar = (int) ceil((floatType) numRows / BLOCK_SIZE);
    int nBlocksVector = (int) ceil(numRows /(floatType)(BLOCK_SIZE / WARP_SIZE));
    int passes = op->getOptionInt("passes");
    int iters  = op->getOptionInt("iterations");
    
    // Results description info
    char atts[TEMP_BUFFER_SIZE];
    sprintf(atts, "%d_elements_%d_rows", numNonZeroes, numRows);
    string prefix = "";
    double gflop = 2 * (double) numNonZeroes / 1e9;
    cout << "CSR vector Kernel\n";

    //cout<<"passes is : " <<passes<<", iters is "<< iters<<std::endl;

    //for (int k=0; k<passes; k++)
    //{
        // Run Scalar Kernel
        CUDA_SAFE_CALL(hipEventRecord(start, 0));
        //for (int j = 0; j < iters; j++)
        //{
            if(suffix == "-DP"){
                spmv_csr_vector_section_kernel<floatType, texReaderDP><<<nBlocksVector, BLOCK_SIZE>>>
            (d_val, d_cols, d_rowDelimiters, numRows, d_out,secStart);
            }else{
                spmv_csr_vector_section_kernel<floatType, texReaderSP><<<nBlocksScalar, BLOCK_SIZE>>>
            (d_val, d_cols, d_rowDelimiters, numRows, d_out,secStart);
            }
        //}
        CUDA_SAFE_CALL(hipEventRecord(stop, 0));
        CUDA_SAFE_CALL(hipEventSynchronize(stop));
        float vectorKernelTime;
        CUDA_SAFE_CALL(hipEventElapsedTime(&vectorKernelTime, start, stop));
        // Transfer data back to host
        CUDA_SAFE_CALL(hipEventRecord(start, 0));
        CUDA_SAFE_CALL(hipMemcpy(h_out, d_out, numRows * sizeof(floatType),hipMemcpyDeviceToHost));
        CUDA_SAFE_CALL(hipEventRecord(stop, 0));
        CUDA_SAFE_CALL(hipEventSynchronize(stop));
        CUDA_SAFE_CALL(hipEventElapsedTime(&oTransferTime, start, stop));
        hipDeviceSynchronize();
        
        vectorKernelTime = (vectorKernelTime / (float)iters) * 1.e-3;
        string testName = prefix+"CSR-Vector"+suffix;
        double totalTransfer = iTransferTime + oTransferTime;
        
        resultDB->AddResult(testName, atts, "Gflop/s",gflop/(vectorKernelTime));
        resultDB->AddResult(testName+"_PCIe", atts, "Gflop/s",gflop / (vectorKernelTime+totalTransfer));
    
    //}
}


// ****************************************************************************
// Function: spmv_csr_scalar_kernel
//
// Purpose:
//   Computes sparse matrix - vector multiplication on the GPU using
//   the CSR data storage format, using a thread per row of the sparse
//   matrix; based on Bell (SC09) and Baskaran (IBM Tech Report)
//
// Arguments:
//   val: array holding the non-zero values for the matrix
//   cols: array of column indices for each element of the sparse matrix
//   rowDelimiters: array of size dim+1 holding indices to rows of the matrix
//                  last element is the index one past the last
//                  element of the matrix
//   dim: number of rows in the matrix
//   out: output - result from the spmv calculation
//
// Returns:  nothing
//           out indirectly through a pointer
//
// Programmer: Lukasz Wesolowski
// Creation: June 28, 2010
//
// Modifications:
//
// ****************************************************************************
template <typename fpType, typename texReader>
__global__ void
spmv_csr_scalar_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out)
{
    int myRow = blockIdx.x * blockDim.x + threadIdx.x;
    texReader vecTexReader;
    if (myRow < dim)
    {
        fpType t = 0.0f;
        int start = rowDelimiters[myRow];
        int end = rowDelimiters[myRow+1];
        for (int j = start; j < end; j++)
        {
            int col = cols[j];
            t += val[j] * vecTexReader(col);
#ifdef DARTS_DEBUG
            if(threadIdx.x <20&&blockIdx.x ==0){
                printf("val[%d]=%lf, vecTexReader(%d)=%lf\n",j,val[j],col,vecTexReader(col));
            }
#endif
        }
        out[myRow] = t;
    }
}



template <typename fpType, typename texReader>
__global__ void
spmv_csr_scalar_section_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out,
                       const int secStart)
{
    int myRow = blockIdx.x * blockDim.x + threadIdx.x;
    texReader vecTexReader;
    if (myRow < dim)
    {
        fpType t = 0.0f;
        int start = rowDelimiters[myRow]-secStart;
        int end = rowDelimiters[myRow+1]-secStart;
        for (int j = start; j < end; j++)
        {
            int col = cols[j];
            t += val[j] * vecTexReader(col);

#ifdef DARTS_DEBUG
           // if(threadIdx.x <20&&blockIdx.x ==0){
           //     printf("val[%d]=%lf, vecTexReader(%d)=%lf\n",j,val[j],col,vecTexReader(col));
           // }
#endif
        }
        out[myRow] = t;
    }
}


// ****************************************************************************
// Function: spmv_csr_vector_kernel
//
// Purpose:
//   Computes sparse matrix - vector multiplication on the GPU using
//   the CSR data storage format, using a warp per row of the sparse
//   matrix; based on Bell (SC09) and Baskaran (IBM Tech Report)
//
// Arguments:
//   val: array holding the non-zero values for the matrix
//   cols: array of column indices for each element of the sparse matrix
//   rowDelimiters: array of size dim+1 holding indices to rows of the matrix
//                  last element is the index one past the last
//                  element of the matrix
//   dim: number of rows in the matrix
//   out: output - result from the spmv calculation
//
// Returns:  nothing
//           out indirectly through a pointer
//
// Programmer: Lukasz Wesolowski
// Creation: June 28, 2010
//
// Modifications:
//
// ****************************************************************************
template <typename fpType, typename texReader>
__global__ void
spmv_csr_vector_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out)
{
    // Thread ID in block
    int t = threadIdx.x;
    // Thread ID within warp
    int id = t & (warpSize-1);
    int warpsPerBlock = blockDim.x / warpSize;
    // One row per warp
    int myRow = (blockIdx.x * warpsPerBlock) + (t / warpSize);
    // Texture reader for the dense vector
    texReader vecTexReader;

    __shared__ volatile fpType partialSums[BLOCK_SIZE];

    if (myRow < dim)
    {
        int warpStart = rowDelimiters[myRow];
        int warpEnd = rowDelimiters[myRow+1];
        fpType mySum = 0;
        for (int j = warpStart + id; j < warpEnd; j += warpSize)
        {
            int col = cols[j];
            mySum += val[j] * vecTexReader(col);
        }
        partialSums[t] = mySum;

        // Reduce partial sums
        if (id < 16) partialSums[t] += partialSums[t+16];
        if (id <  8) partialSums[t] += partialSums[t+ 8];
        if (id <  4) partialSums[t] += partialSums[t+ 4];
        if (id <  2) partialSums[t] += partialSums[t+ 2];
        if (id <  1) partialSums[t] += partialSums[t+ 1];

        // Write result
        if (id == 0)
        {
            out[myRow] = partialSums[t];
        }
    }
}



template <typename fpType, typename texReader>
__global__ void
spmv_csr_vector_section_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out,
                       const int secStart)
{
    // Thread ID in block
    int t = threadIdx.x;
    // Thread ID within warp
    int id = t & (warpSize-1);
    int warpsPerBlock = blockDim.x / warpSize;
    // One row per warp
    int myRow = (blockIdx.x * warpsPerBlock) + (t / warpSize);
    // Texture reader for the dense vector
    texReader vecTexReader;

    __shared__ volatile fpType partialSums[BLOCK_SIZE];

    if (myRow < dim)
    {
        int warpStart = rowDelimiters[myRow]-secStart;
        int warpEnd = rowDelimiters[myRow+1]-secStart;
        fpType mySum = 0;
        for (int j = warpStart + id; j < warpEnd; j += warpSize)
        {
            int col = cols[j];
            mySum += val[j] * vecTexReader(col);
        }
        partialSums[t] = mySum;

        // Reduce partial sums
        if (id < 16) partialSums[t] += partialSums[t+16];
        if (id <  8) partialSums[t] += partialSums[t+ 8];
        if (id <  4) partialSums[t] += partialSums[t+ 4];
        if (id <  2) partialSums[t] += partialSums[t+ 2];
        if (id <  1) partialSums[t] += partialSums[t+ 1];

        // Write result
        if (id == 0)
        {
            out[myRow] = partialSums[t];
        }
    }
}



// ****************************************************************************
// Function: spmv_ellpackr_kernel
//
// Purpose:
//   Computes sparse matrix - vector multiplication on the GPU using
//   the ELLPACK-R data storage format; based on Vazquez et al (Univ. of
//   Almeria Tech Report 2009)
//
// Arguments:
//   val: array holding the non-zero values for the matrix in column
//   major format and padded with zeros up to the length of longest row
//   cols: array of column indices for each element of the sparse matrix
//   rowLengths: array storing the length of each row of the sparse matrix
//   dim: number of rows in the matrix
//   out: output - result from the spmv calculation
//
// Returns:  nothing directly
//           out indirectly through a pointer
//
// Programmer: Lukasz Wesolowski
// Creation: June 29, 2010
//
// Modifications:
//
// ****************************************************************************
template <typename fpType, typename texReader>
__global__ void
spmv_ellpackr_kernel(const fpType * __restrict__ val,
                     const int    * __restrict__ cols,
                     const int    * __restrict__ rowLengths,
                     const int dim, fpType * __restrict__ out)
{
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    texReader vecTexReader;

    if (t < dim)
    {
        fpType result = 0.0f;
        int max = rowLengths[t];
        for (int i = 0; i < max; i++)
        {
            int ind = i*dim+t;
            result += val[ind] * vecTexReader(cols[ind]);
        }
        out[t] = result;
    }
}

template <typename fpType>
__global__ void
zero(fpType * __restrict__ a, const int size)
{
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    if (t < size) a[t] = 0;
}



template void csrTestScalar<double>(ResultDatabase* resultDB, OptionParser* op, CSRMM<double> *csrHost, CSRMM<double> *csrDevice );
template void csrTestScalar<int>(ResultDatabase* resultDB, OptionParser* op, CSRMM<int> *csrHost, CSRMM<int> *csrDevice );

template void csrTestVector<double>(ResultDatabase* resultDB, OptionParser* op, CSRMM<double> *csrHost, CSRMM<double> *csrDevice );
template void csrTestVector<int>(ResultDatabase* resultDB, OptionParser* op, CSRMM<int> *csrHost, CSRMM<int> *csrDevice );
