#include "hip/hip_runtime.h"
extern "C" {
#include <hip/hip_runtime.h>
#include "conf.h"
#include "stencil.h"
}
#include <stdio.h>
#define ROTATE_DOWN(val,MAX) ((val-1==-1)?MAX-1:val-1)
#define ROTATE_UP(val,MAX) ((val+1)%MAX)
/**
  * GPU Device kernel for the for 2D stencil
  * First attempt during hackaton
  * M = Rows, N = Cols INCLUDING HALOS
  * In this version now we replace the size of the shared memory to be just 3 rows (actually 1+HALO*2) rows 
  */

//__global__ void gpu_stencil2D_4pt_hack4(double * dst, double * src, int M, int N)
//{
////	printf("kernel begin!\n");
//	//Declaring the shared memory array for source
//	extern	__shared__ double shared_mem[] ;
//
//	//indexes
//	int i, j, k,curRow;
//                           //Cols   *  numRows/Tile * tileIndex  
//	int base_global_row = ( N ) * ( GRID_TILE_Y * blockIdx.y ); 
//	int base_global_col = blockDim.x*blockIdx.x;
//	int base_global_idx = base_global_row + base_global_col ;
//	int center = 1,north = 0,south = 2; //indexes for the current location in the shared memory
//	int t = threadIdx.x;
//	
//	//copy the shared memory to fill the pipeline
//	bool rowLeft = (blockIdx.y==(gridDim.y-1))&&(M%GRID_TILE_Y<3)&&(M%GRID_TILE_Y>0);
//	int  numRowLeft =(rowLeft)?(3-M%GRID_TILE_Y):0;
//	bool noColsLeft = (base_global_col +t )<N;
//	bool noColsLeft2= (base_global_col+t+2)<N;
//	for (i = 0 ; i < 1+HALO*2-numRowLeft ; i ++ ){
//		k = base_global_idx+i*N+t;
//		j = i*(blockDim.x+2) + t;
//		shared_mem [j] = (noColsLeft)?src[k]:0.0;
//		if((t<2) &&(noColsLeft)){
//			shared_mem[j+blockDim.x]=src[k+blockDim.x];
//		}
//	}
//		
//	__syncthreads();
//
//	int tt = (((blockIdx.y+1)*GRID_TILE_Y)>M)?(M%GRID_TILE_Y): GRID_TILE_Y;
//	int ss = (((M%GRID_TILE_Y)==1)&&(blockIdx.y ==(gridDim.y-2)))?-1:0;
//
//	int lastRow = ((blockIdx.y == (gridDim.y-1))?-1:1)+tt +ss ;
////	printf("lastRow:%d \n",lastRow );
//	//Pipelined copy one row and process it
//	for ( curRow = HALO; curRow < lastRow; curRow+=1 )
//	{
//		//Stencil computation
//							//top             + bottom              + left                + right
//		j = threadIdx.x+HALO;
//		if(noColsLeft2){
//			dst[base_global_idx + curRow*N + j] =(shared_mem[north*(blockDim.x+2)+j] + shared_mem[south*(blockDim.x+2)+j] + shared_mem[center*(blockDim.x+2)+j-1] + shared_mem[center*(blockDim.x+2)+j+1] )/5.5;
//		}
//		__syncthreads();
//		//We are copying from src to shared memory.
//		k=base_global_idx+(curRow+2)*N+threadIdx.x;
//		if(k<M*N){
//			shared_mem [north*(blockDim.x+2)+threadIdx.x] =(noColsLeft)? src[k]:0.0;
//		}
//		if((t<2)&&(noColsLeft)&&(k<M*N)){
//			shared_mem[north*(blockDim.x+2)+threadIdx.x+blockDim.x]=src[k+blockDim.x];
//		}	
//		center = ROTATE_UP(center,3);
//		south  = ROTATE_UP(south,3);
//		north  = ROTATE_UP(north,3);
//		__syncthreads();
//	}
//
////	printf("kernel finish!\n");
//}



__global__ void gpu_stencil2D_4pt_hack4(double * dst, double * src, int M, int N)
{
#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.x==0)){
		printf("kernel begin!\n");
	}
#endif
	//Declaring the shared memory array for source
	extern	__shared__ double shared_mem[] ;

	//indexes
	int i,j, k,curRow;
                           //Cols   *  numRows/Tile * tileIndex  
	int base_global_row = ( GRID_TILE_Y * blockIdx.y ); 
	int base_global_col = blockDim.x*blockIdx.x;
	int base_global_idx = N*base_global_row + base_global_col ;
	int center = 1,north = 0,south = 2; //indexes for the current location in the shared memory
	int t = threadIdx.x;
	
	//copy the shared memory to fill the pipeline
	bool legalCol = (base_global_col +t )<N;
	bool legalCol2= (base_global_col+t+2)<N;
	bool legalColn= (base_global_col+t+blockDim.x)<N;
	for (i = 0 ; i < 1+HALO*2 ; i ++ ){
		k = base_global_idx+i*N+t;
		j = i*(blockDim.x+2) + t;
		bool legalRow = (base_global_row+i)<M;
		shared_mem [j] =legalRow?( legalCol?src[k]:0.0):0.0;
		if((t<2)&&legalColn&&legalRow){
			shared_mem[j+blockDim.x]=src[k+blockDim.x];
		}
	}
		
	__syncthreads();

	//Pipelined copy one row and process it
	for ( curRow = HALO; curRow < GRID_TILE_Y+1; curRow+=1 )
	{
		//Stencil computation
		//top + bottom + left + right

		j = threadIdx.x+HALO;
		bool legalRow1 =( base_global_row+curRow+1)<M;
		if((legalCol2)&&(legalRow1)){
			dst[base_global_idx + curRow*N + j] =(shared_mem[north*(blockDim.x+2)+j] + shared_mem[south*(blockDim.x+2)+j] + shared_mem[center*(blockDim.x+2)+j-1] + shared_mem[center*(blockDim.x+2)+j+1] )/5.5;
		}
		__syncthreads();
		//We are copying from src to shared memory.
		int nextRow2 = base_global_row+curRow+2;
		bool legalRow2 = nextRow2<M;
		k = base_global_col+nextRow2*N+t;

		shared_mem [north*(blockDim.x+2)+t] =(legalRow2&&legalCol)?src[k]:0.0;

		if((t<2)&&legalColn&&legalRow2){
			shared_mem[north*(blockDim.x+2)+t+blockDim.x]=src[k+blockDim.x];
		}	
		
		center = ROTATE_UP(center,3);
		south  = ROTATE_UP(south,3);
		north  = ROTATE_UP(north,3);
		__syncthreads();
	}
#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.x==0)){
		printf("kernel finish!\n");
	}
#endif
}


__global__ void gpu_stencil2D_4pt_hack5_cp_rows(double * dst, double * shared_cols, double *shared_rows,int tile_y,int M, int N){


#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.x==0)){
		printf("copy rows begin!\n");
	}
#endif

	int base_global_row = (tile_y  * blockIdx.y ); 
	int base_global_col = blockDim.x*blockIdx.x;
	int base_global_idx = N*base_global_row + base_global_col ;
	int nextRow = base_global_row+1;
	bool legalNextRow = (nextRow<M)?1:0;
	int t = threadIdx.x;
	bool legalCurCol = (base_global_col + t)<N;
	int idx = (base_global_row/tile_y)*2*N + t+base_global_col;
	int idx_nextrow = idx + N;
	if(legalCurCol){
		shared_rows[idx] = dst[base_global_idx + t];
	}
	if(legalNextRow&&legalCurCol){
		shared_rows[idx_nextrow] = dst[base_global_idx + N+t];
	}
	__syncthreads();


#ifdef CUDA_CUDA_DEBUG
//	if(threadIdx.x==0){
//		printf("blockIdx.x = %d,blockIdx.y = %d\n",blockIdx.x,blockIdx.y);
//	}
//	if(blockIdx.y==1 && threadIdx.x==0){
//		printf("addr: %d\n",idx_nextrow);
//	}
	if(blockIdx.y==0 && blockIdx.x==2 && (t==0 || t==1)){	
		printf("addr:%d, val = %f\n", idx_nextrow,shared_rows[idx_nextrow]);
	}
#endif

#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.x==0)){
		printf("copy rows finish!\n");
	}
#endif
}

__global__ void gpu_stencil2D_4pt_hack5_cp_cols(double * dst, double * shared_cols, double *shared_rows,int tile_x,int tile_y, int M, int N){

#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.y==0)){
		printf("copy cols begin!\n");
	}
#endif

	int base_global_row = tile_y  * blockIdx.y; 
	int base_global_col = tile_x  * blockIdx.x;
	int base_global_idx = N*base_global_row + base_global_col ;
	int nextCol = base_global_col+1;
	bool legalNextCol = (nextCol<N);
	int t = threadIdx.y;
	int idx = 2*M*blockIdx.x + t + base_global_row;
	int idx_nextCol = idx + M ;
	bool legalCurRow = (base_global_row + t)<M;
	if(legalCurRow){
		shared_cols[idx] = dst[base_global_idx + t*N];
	}
	if(legalNextCol && legalCurRow){
		shared_cols[idx_nextCol] = dst[base_global_idx + t*N+1];
	}
	__syncthreads();


#ifdef CUDA_CUDA_DEBUG
//	if(threadIdx.y==0){
//		printf("blockDimy = %d\n",blockDim.y);
//	}
	if(blockIdx.x==1 && t<5){
		printf("addr: %d ,%f,\n",idx_nextCol,shared_cols[idx_nextCol]);
	}
#endif

#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.y==0)){
		printf("copy cols finish!\n");
	}
#endif
}

__global__ void gpu_stencil2D_4pt_hack5(double * dst, double * shared_cols, double *shared_rows,int tile_y,int M, int N)
{
#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.x==0)){
		printf("kernel begin!\n");
	}
#endif
	//Declaring the shared memory array for source
	extern	__shared__ double shared_mem[] ;

	//indexes
	int i,j, k,curRow;
                           //Cols   *  numRows/Tile * tileIndex  
	int base_global_row = ( tile_y * blockIdx.y ); 
	int base_global_col = blockDim.x*blockIdx.x;
	int base_global_idx = N*base_global_row + base_global_col ;
	int center = 1,north = 0,south = 2; //indexes for the current location in the shared memory
	int t = threadIdx.x;

	//copy the shared memory to fill the pipeline
	bool legalCol = (base_global_col + t )<N;
	bool legalCol1 = (base_global_col + t +1)<N;
	bool legalCol2= (base_global_col+t+2)<N;
	bool legalColn= (base_global_col+t+blockDim.x)<N;

	shared_mem [t] = shared_rows[base_global_col + t + blockIdx.y * N*2];
	if(t==2 || t==3){
		shared_mem [blockDim.x + t-2] = shared_rows[base_global_col+blockIdx.y*N*2+blockDim.x  + t-2];
	}
#ifdef CUDA_CUDA_DEBUG
//	if(blockIdx.y==0 && blockIdx.x==1 && (t==2||t==3)){
//		printf("addr: %d,val: %f\n",blockDim.x+t-2,shared_mem[blockDim.x+t-2]);
//	}
#endif
	for (i = 1 ; i < 1+HALO*2 ; i ++ ){
		k = base_global_idx+i*N+t;
		j = i*(blockDim.x+2) + t;
		bool legalRow = (base_global_row+i)<M;
		shared_mem [j+1] =legalRow?( legalCol1?dst[k+1]:0.0):0.0;
	
		if((t==1)&&legalColn&&legalRow){
			shared_mem[j+blockDim.x]=(blockIdx.x == (gridDim.x-1))?dst[k+blockDim.x]:shared_cols[blockIdx.x*2*M+3*M+i+base_global_row];
		}
		if(t==0){
			shared_mem[j] = shared_cols[blockIdx.x*2*M+base_global_row+i];
		}
	}
		
	__syncthreads();


#ifdef CUDA_CUDA_DEBUG
	if(blockIdx.y==0 && blockIdx.x==1 && (t==1||t==0)){
		printf("addr: %d,val: %f\n",blockDim.x+(blockDim.x+2)+t,shared_mem[blockDim.x + (blockDim.x+2)+t]);

		printf("addr: %d,val: %f\n",2*(blockDim.x+2)+blockDim.x+t,shared_mem[2*(blockDim.x+2) + blockDim.x+t]);
	}
#endif

	//Pipelined copy one row and process it
	for ( curRow = HALO; curRow < tile_y; curRow+=1 )
	{
		//Stencil computation
		//top + bottom + left + right

		j = threadIdx.x+HALO;
		bool legalRow1 =( base_global_row+curRow+1)<M;
		if((legalCol2)&&(legalRow1)){
			dst[base_global_idx + curRow*N + j] =(shared_mem[north*(blockDim.x+2)+j] + shared_mem[south*(blockDim.x+2)+j] + shared_mem[center*(blockDim.x+2)+j-1] + shared_mem[center*(blockDim.x+2)+j+1] )/5.5;
		}
		__syncthreads();
		
		//We are copying from src to shared memory.
		int nextRow2 = base_global_row+curRow+2;
		bool legalRow2 = nextRow2<M;
		k = base_global_col+nextRow2*N+t;

		shared_mem [north*(blockDim.x+2)+t+1] =(legalRow2&&legalCol1)?dst[k+1]:0.0;

		if((t==1)&&legalColn&&legalRow2){
			shared_mem[north*(blockDim.x+2)+t+blockDim.x]=(blockIdx.x == (gridDim.x-1))?dst[k+blockDim.x]:shared_cols[blockIdx.x*2*M+3*M+nextRow2];
		}	
		if((t==0)&&legalRow2){
			shared_mem[north*(blockDim.x+2)+t] = shared_cols[nextRow2+blockIdx.x*2*M];
		}
		
		center = ROTATE_UP(center,3);
		south  = ROTATE_UP(south,3);
		north  = ROTATE_UP(north,3);
		__syncthreads();
	}
	int lastRow1 = base_global_row+curRow+1;
	bool legalLastRow1 = (lastRow1)<M;
	
	if(legalLastRow1){
		
		shared_mem[south*(blockDim.x+2)+t] = shared_rows[base_global_col + t + blockIdx.y * N*2+N*3] ;

		if(t==2 || t==3){
			shared_mem [south*(blockDim.x+2) +blockDim.x + t-2] = shared_rows[base_global_col+blockIdx.y*N*2+3*N+blockDim.x  + t-2];
		}
#ifdef CUDA_CUDA_DEBUG
//		if((blockIdx.x==1)&&((t==2)||(t==3))&&(blockIdx.y==0)){ 
//			printf("addr: %d,val: %f\n",base_global_col+blockIdx.y*N*2+3*N+blockDim.x  + t-2 ,shared_mem[south*(blockDim.x+2)+blockDim.x+t-2]);
//		}
#endif
		__syncthreads();
	}
	if((legalCol2)&& legalLastRow1){
			dst[base_global_idx + curRow*N + j] =(shared_mem[north*(blockDim.x+2)+j] + shared_mem[south*(blockDim.x+2)+j] + shared_mem[center*(blockDim.x+2)+j-1] + shared_mem[center*(blockDim.x+2)+j+1] )/5.5;
	
	}


#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.x==0)){
		printf("kernel finish!\n");
	}
#endif
}


__global__ void gpu_stencil2D_4pt_hack2(double * dst, double * src, int M, int N)
{
//	printf("kernel begin!\n");
	//Declaring the shared memory array for source
	__shared__ double shared_mem[ 1 + HALO*2 ] [ GRID_TILE_X + HALO*2]; //1 is the row I am modifying
	//double * shSrc = shared_mem;

	//indexes
	int i, j, curRow;
                           //Cols   *  numRows/Tile * tileIndex  
	int base_global_idx = ( N ) * ( GRID_TILE_Y * blockIdx.y ) + GRID_TILE_X*blockIdx.x;
	
	int center = 1,north = 0,south = 2; //indexes for the current location in the shared memory

	//copy the shared memory to fill the pipeline
	for (i = 0 ; i < 1+HALO*2 ; i ++ )
		for (j = threadIdx.x ; j < GRID_TILE_X+2*HALO ; j+=blockDim.x)
		{
			shared_mem [i][j] = src[base_global_idx + i*N + j];
		}
	__syncthreads();
	//Pipelined copy one row and process it
	for ( curRow = HALO; curRow < GRID_TILE_Y; curRow+=1 )
	{
		//Stencil computation
		for (j = threadIdx.x + HALO ; j < GRID_TILE_X+HALO ; j+=blockDim.x)
		{
							//top             + bottom              + left                + right
			dst[base_global_idx + curRow*N + j] = (shared_mem[north][j] + shared_mem[south][j] + shared_mem[center][j-1] + shared_mem[center][j+1] )/5.5;
		}
		
		__syncthreads();
		//We are copying from dst to shared memory.
		for (j = threadIdx.x ; j < GRID_TILE_X+2*HALO ; j+=blockDim.x)
		{
			shared_mem [north][j] = src[base_global_idx + (curRow+2)*N + j];
		}
	
		center = ROTATE_UP(center,3);
		south  = ROTATE_UP(south,3);
		north = ROTATE_UP(north,3);
		__syncthreads();
	}

	//Dranning the pipeline
	for (j = threadIdx.x + HALO ; j < GRID_TILE_X+HALO ; j+=blockDim.x)
	{
							//top             + bottom              + left                + right
		dst[base_global_idx + curRow*N + j] = (shared_mem[north][j] + shared_mem[south][j] + shared_mem[center][j-1] + shared_mem[center][j+1] )/5.5;
	}
	__syncthreads();

//	printf("kernel finish!\n");
}



///**
//  * GPU Device kernel for the for 2D stencil
//  * First attempt during hackaton
//  * M = Rows, N = Cols INCLUDING HALOS
//  */
//__global__ void gpu_stencil2D_4pt_hack1(double * dst, double * src, int M, int N)
//{
//
//	//Declaring the shared memory array for source
//	__shared__ double shared_mem[GRID_TILE_Y + HALO*2 ] [ GRID_TILE_X + HALO*2];
//	//double * shSrc = shared_mem;
//
//	//indexes
//	int i, j;
//
//                           //Cols   *  numRows/Tile * tileIndex  
//	int base_global_idx = ( N ) * ( GRID_TILE_Y * blockIdx.y ) + GRID_TILE_X*blockIdx.x;
//
//	//We are copying from dst to shared memory.
//	for (i = 0 ; i < GRID_TILE_Y+2*HALO ; i ++ )
//		for (j = threadIdx.x ; j < GRID_TILE_X+2*HALO ; j+=blockDim.x)
//		{
//			shared_mem [i][j] = src[base_global_idx + i*N + j];
//		}
//
//	__syncthreads();
//
//	//Stencil computation
//	for (i = HALO ; i < GRID_TILE_Y+HALO ; i ++ )
//		for (j = threadIdx.x + HALO ; j < GRID_TILE_X+HALO ; j+=blockDim.x)
//		{
//			                                //top             + bottom              + left                + right
//			dst[base_global_idx + i*N + j] = (shared_mem[i-1][j] + shared_mem[i+1][j] + shared_mem[i][j-1] + shared_mem[i][j+1] )/5.5;
//		}
//
//	__syncthreads();
//}




/**
  * GPU Device kernel for the for 2D stencil
  * M = Rows, N = Cols
  */
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
	//Declaring the shared memory array for source
	extern __shared__ double shared_mem[];
	double * shSrc = shared_mem;

	//indexes
	int i, j;

	//neighbor's values 
	double north, south, east, west;



	//SharedMem Collumns Dimension
	int smColDim = HALO*2+blockDim.y*TILE_SIZE;
	int smRowDim = HALO*2+blockDim.x*TILE_SIZE;

	//Copying to shared memory

	//Inner part
	for ( i = 0 ; i < TILE_SIZE ; i++ )
	{
		for ( j = 0 ; j < TILE_SIZE ; j++ )
		{
			int globalIndex=HALO*N+blockIdx.x*blockDim.x*TILE_SIZE*N+threadIdx.x*TILE_SIZE*N+i*N+blockIdx.y*blockDim.y*TILE_SIZE+threadIdx.y*TILE_SIZE+j+HALO;
			int shMemIndex=HALO*smColDim+threadIdx.x*smColDim*TILE_SIZE+i*smColDim+HALO+threadIdx.y*TILE_SIZE+j;
			shSrc[shMemIndex]=src[globalIndex];
		}
	}

	//Halos

	if (threadIdx.x == 0 && threadIdx.y == 0 ) 
	{

		int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;
		//For Bottom and top row
		for ( i = 0 ; i < HALO ; i++ )
		{
			for ( j = 0 ; j < smColDim ; j++ )
			{
				indexTopHalo = (blockIdx.x*blockDim.x*TILE_SIZE+i)*N + (blockIdx.y*blockDim.y*TILE_SIZE) + j;
				indexBottomHalo = (HALO + (blockIdx.x+1)*blockDim.x*TILE_SIZE)*N + (blockIdx.y*blockDim.y*TILE_SIZE)+j;
				shSrc[i*smColDim+j] = src[indexTopHalo];
				shSrc[(HALO+blockDim.x*TILE_SIZE+i)*smColDim + j] = src[indexBottomHalo];
			}
		}
		
		//For right and left Columns
		for ( i = 0 ; i < HALO ; i++ )
		{
			for ( j = 0 ; j < smRowDim-HALO*2; j ++ )
			{
				indexLeftHalo = (HALO+blockIdx.x*blockDim.x*TILE_SIZE+j)*N + (blockIdx.y*blockDim.y*TILE_SIZE)+i;
				indexRightHalo = (HALO+blockIdx.x*blockDim.x*TILE_SIZE+j)*N + ((blockIdx.y+1)*blockDim.y*TILE_SIZE)+HALO+i;
				shSrc[(HALO+j)*smColDim+i] = src[indexLeftHalo];
				shSrc[(HALO+j+1)*smColDim-HALO+i] = src[indexRightHalo];
			}
		}
	}

	__syncthreads();



	for ( i = 0 ; i < TILE_SIZE ; i++ )
	{
		for ( j = 0 ; j < TILE_SIZE ; j++ )
		{
			int globalIndex=HALO*N+blockIdx.x*blockDim.x*TILE_SIZE*N+threadIdx.x*TILE_SIZE*N+i*N+blockIdx.y*blockDim.y*TILE_SIZE+threadIdx.y*TILE_SIZE+j+HALO;
			int shMemIndex=HALO*smColDim+threadIdx.x*smColDim*TILE_SIZE+i*smColDim+HALO+threadIdx.y*TILE_SIZE+j;


			//Getting the neighbohrs
			north = shSrc[shMemIndex-smColDim];
			south = shSrc[shMemIndex+smColDim];
			east  = shSrc[shMemIndex+1];
			west  = shSrc[shMemIndex-1];
			//Real Stencil operation
			dst[globalIndex] = ( north + south + east + west )/5.5;
//			dst[globalIndex] = ( north + south + east + west )/4;
		}
	}

	__syncthreads();
}




/**
 *  Naïve 4pt stencil code for 2D arrays. 
 */
void
stencil2D4pt ( double* __restrict__ dst,    double* __restrict__ src, 
               const size_t     n_rows, const size_t     n_cols,
               const size_t     n_tsteps )
{
    typedef double (*Array2D)[n_cols];
    volatile Array2D DST = (Array2D) dst,
            SRC = (Array2D) src;
    for (size_t ts = 0; ts < n_tsteps; ++ts) {
        for (size_t i = 1; i < n_rows-1; ++i) {
            for (size_t j = 1; j < n_cols-1; ++j) {
                DST[i][j] = (SRC[i-1][j] + SRC[i+1][j] + SRC[i][j-1] + SRC[i][j+1])/5.5;
            }
        }
        SWAP_PTR(&DST,&SRC);
    }
}

extern "C"
void
stencil2D4pt_gpu( double * __restrict__ dst, double* __restrict__ src,
		  const size_t M, const size_t N, 
		  const size_t NUM_ITERATIONS)//M Rows by N Columns
{
		
	double size = sizeof(double) * M * N;

	//device memory allocation
	double * d_dst, * d_src;
	hipMalloc( (void **) &d_dst, size);
	hipMalloc( (void **) &d_src, size);
	
	//dimmensions for indexes
	// TODO the -2 is to remove the borders
	dim3 dimBlock(MAX_BLOCK_DIM,MAX_BLOCK_DIM);
	int gridx = (N-2)/(MAX_BLOCK_DIM*TILE_SIZE) + (((N-2)%(MAX_BLOCK_DIM*TILE_SIZE) == 0)? 0:1 ) ;
	int gridy = (M-2)/(MAX_BLOCK_DIM*TILE_SIZE) + (((M-2)%(MAX_BLOCK_DIM*TILE_SIZE) == 0)? 0:1 ) ;
	dim3 dimGrid(gridx,gridy);

	//Shared memory size = inside + halo
	int shMemSize=MAX_BLOCK_DIM*TILE_SIZE*MAX_BLOCK_DIM*TILE_SIZE*sizeof(double)+(HALO*MAX_BLOCK_DIM*TILE_SIZE+HALO*HALO)*4*sizeof(double);
	
	//Hackaton dimensions
	dim3 dimGrid_hack1((N-HALO*2)/GRID_TILE_X,(M-HALO*2)/GRID_TILE_Y);

	//Copying the device memory
	hipMemcpy(d_src, src, size, hipMemcpyHostToDevice);
	hipMemcpy(d_dst, dst, size, hipMemcpyHostToDevice);

	//printf("CUDA Stencil Code running... cycles = %d. dim = %d by %d \n",NUM_ITERATIONS,M,N);
	
    int time_step = NUM_ITERATIONS;

    while (time_step-- > 0) 
    {
    	//gpu_stencil2D_4pt<<<dimGrid,dimBlock,shMemSize>>>(d_dst,d_src,M,N);
		//gpu_stencil2D_4pt_hack1<<<dimGrid_hack1,NUM_THREADS>>>(d_dst,d_src,M,N); //JOSE Hackathon!
		//printf("before: d_src[10] = %ld",d_src[10]);

		gpu_stencil2D_4pt_hack2<<<dimGrid_hack1,NUM_THREADS>>>(d_dst,d_src,M,N);
		//Inline swapping.
		
		//printf("after: d_src[10] = %ld",d_src[10]);
		double * temp;
		if ( NUM_ITERATIONS%2 ==0 || time_step !=0)
		{
			temp=d_src;
			d_src=d_dst;
			d_dst=temp;
		}
	}
	
	
	//Copying memory back from device to DRAM
	//hipMemcpy(src, d_src, size, hipMemcpyDeviceToHost);
	hipMemcpy(dst, d_dst, size, hipMemcpyDeviceToHost);
	hipMemcpy(src, d_src, size, hipMemcpyDeviceToHost);
	
	//Free device memory
	hipFree(d_src); hipFree(d_dst);
}

//void*
//stencil_run(void* arg)
//{
//    stencil_t* stencil = (stencil_t*)arg;
//    STENCIL_COMPUTE(stencil->stencil,stencil->arg);
//    return NULL;
//}


void gpu_kernel4(dim3 dimGrid,dim3 dimBlock,double * d_dst, double * d_src, int M, int N){
		int sharedMemSize = sizeof(double)*(1+HALO*2)*(dimBlock.x+2);
#ifdef CUDA_DARTS_DEBUG
		printf("sharedMemSize: %d B, total sharedMemSize: %d B\n",sharedMemSize, sharedMemSize*dimGrid.x*dimGrid.y);
#endif
		gpu_stencil2D_4pt_hack4<<<dimGrid,dimBlock,sharedMemSize>>>(d_dst,d_src,M,N);
#ifdef CUDA_DARTS_DEBUG
		printf("gpu kernel return to host, but kernel haven't finished!\n");
#endif

}
void gpu_kernel5(dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedCols, double * sharedRows, int tile_y,int M, int N){
		int sharedMemSize = sizeof(double)*(1+HALO*2)*(dimBlock.x+2);
#ifdef CUDA_DARTS_DEBUG
		printf("sharedMemSize: %d B, total sharedMemSize: %d B\n",sharedMemSize, sharedMemSize*dimGrid.x*dimGrid.y);
#endif
		gpu_stencil2D_4pt_hack5<<<dimGrid,dimBlock,sharedMemSize>>>(d_dst,sharedCols,sharedRows,tile_y,M,N);
#ifdef CUDA_DARTS_DEBUG
		printf("gpu kernel return to host, but kernel haven't finished!\n");
#endif

}


void gpu_kernel5_cp_rows(dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedCols, double * sharedRows, int tile_y,int M, int N){

		gpu_stencil2D_4pt_hack5_cp_rows<<<dimGrid,dimBlock>>>(d_dst,sharedCols,sharedRows,tile_y,M,N);
}

void gpu_kernel5_cp_cols(dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedCols, double * sharedRows, int tile_x,int tile_y,int M, int N){

		gpu_stencil2D_4pt_hack5_cp_cols<<<dimGrid,dimBlock>>>(d_dst,sharedCols,sharedRows,tile_x,tile_y,M,N);
}


void gpu_kernel5_stream(hipStream_t &stream, dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedCols, double * sharedRows, int tile_y,int M, int N){
		int sharedMemSize = sizeof(double)*(1+HALO*2)*(dimBlock.x+2);
#ifdef CUDA_DARTS_DEBUG
		printf("Kernel5 stream: sharedMemSize: %d B, total sharedMemSize: %d B\n",sharedMemSize, sharedMemSize*dimGrid.x*dimGrid.y);
#endif

        gpu_stencil2D_4pt_hack5<<<dimGrid,dimBlock,sharedMemSize,stream>>>(d_dst,sharedCols,sharedRows,tile_y,M,N);
#ifdef CUDA_DARTS_DEBUG
		printf("gpu kernel return to host, but kernel haven't finished!\n");
#endif


}


void gpu_kernel5_stream_cp_rows(hipStream_t &stream ,dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedCols, double * sharedRows, int tile_y,int M, int N){

#ifdef CUDA_DARTS_DEBUG
		printf("Kernel5 stream: copy Rows. \n");
#endif
        gpu_stencil2D_4pt_hack5_cp_rows<<<dimGrid,dimBlock,0,stream>>>(d_dst,sharedCols,sharedRows,tile_y,M,N);

}

void gpu_kernel5_stream_cp_cols(hipStream_t &stream,dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedCols, double * sharedRows, int tile_x,int tile_y,int M, int N){

#ifdef CUDA_DARTS_DEBUG
		printf("Kernel5 stream: copy cols. \n");
#endif
        gpu_stencil2D_4pt_hack5_cp_cols<<<dimGrid,dimBlock,0,stream>>>(d_dst,sharedCols,sharedRows,tile_x,tile_y,M,N);

}

void gpu_kernel1(dim3 dimGrid_hack1,double * d_dst, double * d_src, int M, int N){
		gpu_stencil2D_4pt_hack2<<<dimGrid_hack1,NUM_THREADS>>>(d_dst,d_src,M,N);
}

void gpu_kernel3(hipStream_t &stream,dim3 dimGrid_hack1,double * d_dst, double * d_src, int M, int N){
		gpu_stencil2D_4pt_hack2<<<dimGrid_hack1,NUM_THREADS,0,stream>>>(d_dst,d_src,M,N);

}

void gpu_kernel2(dim3 dimGrid_hack1,double *dst, double *src, double size, size_t ts, double * d_dst, double * d_src, int M, int N){
	double * tmp;
	while (--ts!=0){
		printf("ts:%ld \n", ts);
		gpu_stencil2D_4pt_hack2<<<dimGrid_hack1,NUM_THREADS>>>(d_dst,d_src,M,N);
		tmp = d_src;
		d_src = d_dst;
		d_dst=tmp;
	}
}
bool checkGpu(hipStream_t *stream, size_t n){
    for (size_t i=0;i<n;++i){
  
#ifdef CUDA_DARTS_DEBUG
		printf("checkGpu: %d \n",i);
#endif
        if (hipSuccess != hipStreamQuery(stream[i]))
            return false;
    }
    return true;
}



__global__ void gpu_stencil37_hack1_cp_slices(double * dst, double * shared_rows, double *shared_cols,double *shared_slices,int n_rows, int n_cols,int n_slices,int tile_x,int tile_y, int tile_z){

#ifdef CUDA_DARTS_DEBUG
    if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.x==0)){
		printf("copy slices begin!\n");
        printf("gridDim.x=%d,gridDim.y=%d,gridDim.z=%d\n",gridDim.x,gridDim.y,gridDim.z);
        printf("blockDim.x=%d,blockDim.y=%d,blockDim.z=%d\n",blockDim.x,blockDim.y,blockDim.z);
        printf("tile_x=%d,tile_y=%d,tile_z=%d\n",tile_x,tile_y,tile_z);
	}
#endif
    int base_global_slice = tile_z * blockIdx.z;
	int base_global_row   = tile_y * blockIdx.y;
	int base_global_col   = blockDim.x * blockIdx.x;

	int area = n_rows*n_cols;
    int base_global_idx = base_global_slice*area + base_global_row * n_cols + base_global_col;
    
    int nextSlice = base_global_slice+1;
    bool legalNextSlice = (nextSlice<n_slices);
	int tx = threadIdx.x;
	bool legalCurCol = (base_global_col + tx)<n_cols;
    
    for(int ty=0;ty<tile_y;++ty){ 
        bool legalCurRow = (base_global_row + ty)<n_rows;
        int idx = blockIdx.z*area*2 + (base_global_row+ty)*n_cols + base_global_col+tx ;
        int idx_dst = base_global_idx + ty*n_cols+tx;
    	if(legalCurCol&&legalCurRow){
    		shared_slices[idx] = dst[idx_dst];
    	}
    	if(legalNextSlice&&legalCurCol&&legalCurRow){
    		shared_slices[idx+area] = dst[idx_dst+area];
    	}

    }
    __syncthreads();

#ifdef CUDA_CUDA_DEBUG
	if(blockIdx.z ==0 && blockIdx.y==0 && blockIdx.x==1 ){
	//	printf("shared_slices: addr:%d, val = %f\n",n_cols*n_rows + threadIdx.x,shared_slices[n_cols*n_rows+threadIdx.x]);
	    if(threadIdx.x==0||threadIdx.x==1||threadIdx.x==2){
            int addr = n_cols*n_rows + blockDim.x*blockIdx.x+threadIdx.x;
            int addr1 = n_cols*n_rows + blockDim.x*blockIdx.x+threadIdx.x+n_cols;
            int addr2 = n_cols*n_rows + blockDim.x*blockIdx.x+threadIdx.x+n_cols*2;
	    	printf("blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,shared_slices: addr= %d, val= %f\n",blockIdx.x, blockIdx.y, blockIdx.z, addr,shared_slices[addr]);
	    	printf("blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,shared_slices: addr= %d, val= %f\n",blockIdx.x, blockIdx.y, blockIdx.z, addr1,shared_slices[addr1]);
	    	printf("blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,shared_slices: addr= %d, val= %f\n",blockIdx.x, blockIdx.y, blockIdx.z, addr2,shared_slices[addr2]);
        }
    }
#endif

#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.x==0)){
		printf("copy slices end!\n");
	}
#endif
}


__global__ void gpu_stencil37_hack1_cp_rows(double * dst, double * shared_rows, double *shared_cols,double *shared_slices,int n_rows, int n_cols,int n_slices,int tile_x,int tile_y, int tile_z){

#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.x==0)){
		printf("copy rows begin\n");
        printf("gridDim.x=%d,gridDim.y=%d,gridDim.z=%d\n",gridDim.x,gridDim.y,gridDim.z);
        printf("blockDim.x=%d,blockDim.y=%d,blockDim.z=%d\n",blockDim.x,blockDim.y,blockDim.z);
        printf("tile_x=%d,tile_y=%d,tile_z=%d\n",tile_x,tile_y,tile_z);
	}
#endif
    int base_global_slice = tile_z * blockIdx.z;
	int base_global_row   = tile_y  * blockIdx.y;
	int base_global_col   = blockDim.x*blockIdx.x;

	int dst_area = n_rows*n_cols;
    int s_area = gridDim.y*n_cols*2; 
    
    int base_global_idx = base_global_slice*dst_area + base_global_row * n_cols + base_global_col;
    
    int nextRow = base_global_row+1;
	bool legalNextRow = nextRow<n_rows;

    int tx = threadIdx.x;
	bool legalCurCol = (base_global_col + tx)<n_cols;
    
    for(int tz=0;tz<tile_z;++tz){ 
        bool legalCurSlice = (base_global_slice + tz)<n_slices;
        int idx_dst =base_global_idx + tz*dst_area+ tx  ;
        int idx = (base_global_slice+tz)*s_area + blockIdx.y*n_cols*2+blockIdx.x*blockDim.x+ tx  ;
        if(legalCurCol && legalCurSlice){
    		shared_rows[idx] = dst[idx_dst];
    	}
        if(legalCurCol && legalCurSlice && legalNextRow){
    		shared_rows[idx+n_cols] = dst[idx_dst+n_cols];
    	}


    }
    __syncthreads();

#ifdef CUDA_CUDA_DEBUG
	if(blockIdx.y==0 && blockIdx.x==0 &&blockIdx.z==0 ){
        if((threadIdx.x==0 || threadIdx.x==1 || threadIdx.x==2 ) && threadIdx.y==0){
            
            int addr0 = base_global_idx+0*dst_area+threadIdx.x;
            int addr  = base_global_slice+blockIdx.x*blockDim.x + threadIdx.x;
            int addr1 = s_area*(base_global_slice+1)+n_cols+blockIdx.x*blockDim.x+ threadIdx.x;
            int addr2 = s_area*(base_global_slice+2)+n_cols+blockIdx.x*blockDim.x+ threadIdx.x;
		    printf("blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,dst      : z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,0,addr0,dst[addr0]);
		    printf("blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,shared_rows: z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,0,addr,shared_rows[addr]);
		    printf("blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,shared_rows: z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,1,addr1,shared_rows[addr1]);
		    printf("blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,shared_rows: z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,2,addr2,shared_rows[addr2]);
        }
        if(threadIdx.x==0 && threadIdx.y==0){
            int addr =  2*s_area+n_cols+256;
            int addr1 = 2*dst_area+n_cols+256;
            printf("shared_rows: addr:%d, val:%f\n", addr, shared_rows[addr]);  
            printf("dst        : addr:%d, val:%f\n", addr1, dst[addr1]);  
        }
	}
#endif

#ifdef CUDA_DARTS_DEBUG
	
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.x==0)){
		printf("copy rows end!\n");
	}
#endif
}


__global__ void gpu_stencil37_hack1_cp_cols(double * dst, double * shared_rows, double *shared_cols,double *shared_slices,int n_rows, int n_cols,int n_slices,int tile_x,int tile_y, int tile_z){

#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.y==0)&& threadIdx.x==0 && threadIdx.z==0){
		printf("copy cols begin\n");
        printf("gridDim.x=%d,gridDim.y=%d,gridDim.z=%d\n",gridDim.x,gridDim.y,gridDim.z);
        printf("blockDim.x=%d,blockDim.y=%d,blockDim.z=%d\n",blockDim.x,blockDim.y,blockDim.z);
        printf("tile_x=%d,tile_y=%d,tile_z=%d\n",tile_x,tile_y,tile_z);
	}
#endif
    int base_global_slice = tile_z * blockIdx.z;
	int base_global_row   = blockDim.y * blockIdx.y;
	int base_global_col   = tile_x * blockIdx.x;

	int area_dst = n_rows*n_cols;
    int area_shared = gridDim.x*n_rows*2; 
    
#ifdef CUDA_CUDA_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.y==0&&threadIdx.x==0&&threadIdx.z==0)){
        printf("area_shared=%d\n",area_shared);
	}
#endif
    int base_global_idx = base_global_slice*area_dst + base_global_row * n_cols + base_global_col;
    
    int nextCol= base_global_col+1;
	bool legalNextCol = (nextCol<n_cols)?1:0;
    
    int ty = threadIdx.y;
	bool legalCurRow = (base_global_row + ty)<n_rows;
    
    for(int tz=0;tz<tile_z;++tz){ 
        bool legalCurSlice = (base_global_slice + tz)<n_slices;
        int idx_dst =base_global_idx + tz*area_dst + ty*n_cols ;
        int idx = (base_global_slice+tz)*area_shared + blockIdx.x*2*n_rows+blockIdx.y*blockDim.y+ty; 

        if(legalCurRow && legalCurSlice){
    		shared_cols[idx] = dst[idx_dst];
    	}
        if(legalCurRow && legalCurSlice && legalNextCol){
    		shared_cols[idx + n_rows] = dst[idx_dst + 1];
        }

        __syncthreads();
    }
    __syncthreads();

#ifdef CUDA_CUDA_DEBUG
	if(blockIdx.z ==0 && blockIdx.y==0 && blockIdx.x==0 && (threadIdx.x==0)){
//		printf("shared_cols: addr:%d, val = %f\n", threadIdx.y,shared_cols[threadIdx.y]);
	}
#endif

#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.y==0 && threadIdx.x==0 && threadIdx.z==0)){
		printf("copy cols end!\n");
	}
#endif
}


void gpu_kernel37_cp_slices(dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedCols, double * sharedRows, double * sharedSlices, int n_rows, int n_cols, int n_slices,int tile_x,int tile_y, int tile_z){

//#ifdef CUDA_DARTS_DEBUG
//		printf("gpu_kernel37 copy slices begin!\n");
//        printf("dimBlock.x: %d, dimBlock.y: %d,dimBlock.z: %d\n",dimBlock.x,dimBlock.y,dimBlock.z);
//        printf("dimGrid.x: %d, dimGrid.y: %d,dimGrid.z: %d\n",dimGrid.x,dimGrid.y,dimGrid.z);
//#endif
		gpu_stencil37_hack1_cp_slices<<<dimGrid,dimBlock>>>(d_dst,sharedRows,sharedCols,sharedSlices,n_rows,n_cols,n_slices,tile_x,tile_y,tile_z);

//#ifdef CUDA_DARTS_DEBUG
//		printf("gpu_kernel37 copy slices finish!\n");
//#endif
}

void gpu_kernel37_cp_rows(dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedRows,double * sharedCols, double * sharedSlices, int n_rows, int n_cols, int n_slices,int tile_x,int tile_y, int tile_z){

//#ifdef CUDA_DARTS_DEBUG
//		printf("gpu_kernel37 copy rows begin!\n");
//#endif
		gpu_stencil37_hack1_cp_rows<<<dimGrid,dimBlock>>>(d_dst,sharedRows,sharedCols,sharedSlices,n_rows,n_cols,n_slices,tile_x,tile_y,tile_z);

//#ifdef CUDA_DARTS_DEBUG
//		printf("gpu_kernel37 copy rows finish!\n");
//#endif
}

void gpu_kernel37_cp_cols(dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedRows, double * sharedCols, double * sharedSlices, int n_rows, int n_cols, int n_slices,int tile_x,int tile_y, int tile_z){

//#ifdef CUDA_DARTS_DEBUG
//		printf("gpu_kernel37 copy cols begin!\n");
//#endif
		gpu_stencil37_hack1_cp_cols<<<dimGrid,dimBlock>>>(d_dst,sharedRows,sharedCols,sharedSlices,n_rows,n_cols,n_slices,tile_x,tile_y,tile_z);
//#ifdef CUDA_DARTS_DEBUG
//		printf("gpu_kernel37 copy cols finish!\n");
//#endif
}

void gpu_kernel37_cp_slices_stream(hipStream_t &stream,dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedCols, double * sharedRows, double * sharedSlices, int n_rows, int n_cols, int n_slices,int tile_x,int tile_y, int tile_z){

//#ifdef CUDA_DARTS_DEBUG
//		printf("gpu_kernel37 copy slices begin!\n");
//        printf("dimBlock.x: %d, dimBlock.y: %d,dimBlock.z: %d\n",dimBlock.x,dimBlock.y,dimBlock.z);
//        printf("dimGrid.x: %d, dimGrid.y: %d,dimGrid.z: %d\n",dimGrid.x,dimGrid.y,dimGrid.z);
//#endif
		gpu_stencil37_hack1_cp_slices<<<dimGrid,dimBlock,0,stream>>>(d_dst,sharedRows,sharedCols,sharedSlices,n_rows,n_cols,n_slices,tile_x,tile_y,tile_z);

//#ifdef CUDA_DARTS_DEBUG
//		printf("gpu_kernel37 copy slices finish!\n");
//#endif
}

void gpu_kernel37_cp_rows_stream(hipStream_t &stream, dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedRows,double * sharedCols, double * sharedSlices, int n_rows, int n_cols, int n_slices,int tile_x,int tile_y, int tile_z){

//#ifdef CUDA_DARTS_DEBUG
//		printf("gpu_kernel37 copy rows begin!\n");
//#endif
		gpu_stencil37_hack1_cp_rows<<<dimGrid,dimBlock,0,stream>>>(d_dst,sharedRows,sharedCols,sharedSlices,n_rows,n_cols,n_slices,tile_x,tile_y,tile_z);

//#ifdef CUDA_DARTS_DEBUG
//		printf("gpu_kernel37 copy rows finish!\n");
//#endif
}

void gpu_kernel37_cp_cols_stream(hipStream_t &stream,dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedRows, double * sharedCols, double * sharedSlices, int n_rows, int n_cols, int n_slices,int tile_x,int tile_y, int tile_z){

//#ifdef CUDA_DARTS_DEBUG
//		printf("gpu_kernel37 copy cols begin!\n");
//#endif
		gpu_stencil37_hack1_cp_cols<<<dimGrid,dimBlock,0,stream>>>(d_dst,sharedRows,sharedCols,sharedSlices,n_rows,n_cols,n_slices,tile_x,tile_y,tile_z);
//#ifdef CUDA_DARTS_DEBUG
//		printf("gpu_kernel37 copy cols finish!\n");
//#endif
}

__global__ void gpu_stencil37_hack2(double * dst, double * shared_rows, double * shared_cols, double * shared_slices,int n_rows,int n_cols, int n_slices,int tile_x, int tile_y, int tile_z){

#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.x==0)&&(threadIdx.y==0)){
		printf("3D kernel begin!\n");
//        printf("blockIdx.x = %d,blockIdx.y = %d, blockIdx.z = %d\n", blockIdx.x,blockIdx.y,blockIdx.z);
//        printf("threadIdx.x = %d,threadIdx.y = %d, threadIdx.z = %d \n", threadIdx.x,threadIdx.y,threadIdx.z);
        printf("gridDim.x=%d,gridDim.y=%d,gridDim.z=%d\n",gridDim.x,gridDim.y,gridDim.z);
        printf("blockDim.x=%d,blockDim.y=%d,blockDim.z=%d\n",blockDim.x,blockDim.y,blockDim.z);
        printf("tile_x=%d,tile_y=%d,tile_z=%d\n",tile_x,tile_y,tile_z);
	}
#endif
    
    int base_global_slice = tile_z * blockIdx.z;
	int base_global_row   = tile_y * blockIdx.y;
	int base_global_col   = tile_x * blockIdx.x;
    int global_area = n_rows*n_cols;
   
    int base_global_idx = base_global_slice * global_area + base_global_row * n_cols + base_global_col;
    int num_rows = ((base_global_row + tile_y+2)<n_rows)?(tile_y+2):(n_rows-base_global_row);
    int num_cols = ((base_global_col + tile_x+2)<n_cols)?(tile_x+2):(n_cols-base_global_col);
    int num_slices = ((base_global_slice + tile_z+2)<n_slices)?(tile_z+2):(n_slices-base_global_slice);

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int tx1 = threadIdx.x + 1;
    int ty1 = threadIdx.y + 1;
	bool legalCol    = (base_global_col + tx   )<n_cols;
	bool legalCol1   = (base_global_col + tx + 1)<n_cols;
	bool legalCol2   = (base_global_col + tx + 2)<n_cols;
    bool legalColN   = (base_global_col + blockDim.x     )<n_cols;
    bool legalColN1  = (base_global_col + blockDim.x + 1 )<n_cols;
    bool legalColN2  = (base_global_col + blockDim.x + 2 )<n_cols;
    bool legalColNX  = (base_global_col + blockDim.x + tx)<n_cols;
    
    bool legalSlice1  = (base_global_slice + 1       )<n_slices;
    bool legalSlice2  = (base_global_slice + 2       )<n_slices;
    bool legalSliceN  = (base_global_slice + tile_z  )<n_slices;
    bool legalSliceN1 = (base_global_slice + tile_z+1)<n_slices;
   
    bool legalRow   = (base_global_row + ty    )<n_rows;
    bool legalRow1  = (base_global_row + ty + 1)<n_rows;
    bool legalRow2  = (base_global_row + ty + 2)<n_rows;
    bool legalRowN  = (base_global_row + blockDim.y     )<n_rows;
    bool legalRowN1 = (base_global_row + blockDim.y + 1 )<n_rows;
    bool legalRowN2 = (base_global_row + blockDim.y + 2 )<n_rows;
    bool legalRowNY = (base_global_row + blockDim.y + ty)<n_rows;
    
    //Declaring the shared memory array for source
	extern	__shared__ double shared_mem[] ;
    
    
    //====================================copy first 3 slices to shared_mem[]=================================//
    int s_stride_x = blockDim.x + 2;
    int s_stride_y = blockDim.y + 2;
    int shared_area = s_stride_x*s_stride_y; 
    int sslices_area = global_area;
    int slices_idx = (blockIdx.z)*global_area*2 +blockIdx.y*tile_y*n_cols + blockIdx.x*tile_x;
    //----copy first two slices from shared_slices----//
    //--copy x: 1~blockDim.x , y: 1~blockDimy
    shared_mem[(ty+1)*s_stride_x+tx+1] = (legalRow1&&legalCol1)?shared_slices[slices_idx+(ty+1)*n_cols+tx+1]:0;
    shared_mem[(ty+1)*s_stride_x+tx+1+shared_area] = (legalRow1&&legalCol1&&legalSlice1)?shared_slices[slices_idx+(ty+1)*n_cols+tx+1+sslices_area]:0;

    //--copy y=0,y=blockDim.y+1, x=1~blockDim.x --//
    if(ty ==0){
        shared_mem[ty*s_stride_x+ tx+1] = (legalRow&&legalCol1)?shared_slices[slices_idx+ty*n_cols+tx+1]:0;
        shared_mem[ty*s_stride_x+ tx+1+shared_area] = (legalRow&&legalCol1&&legalSlice1)?shared_slices[slices_idx+ty*n_cols+tx+1+sslices_area]:0;
    }
    if(ty==1){
        shared_mem[(blockDim.y+ty)*s_stride_x+ tx+1] = (legalRowNY&&legalCol1)?shared_slices[slices_idx+(ty+blockDim.y)*n_cols+tx+1]:0;
        shared_mem[(blockDim.y+ty)*s_stride_x+ tx+1+shared_area] = (legalRowNY&&legalCol1&&legalSlice1)?shared_slices[slices_idx+(ty+blockDim.y)*n_cols+tx+1+sslices_area]:0;
    }
    //--copy x= 0, x=blockDim.x+1, y=1~blockDim.x--//
    if(tx==0){
        shared_mem[(ty+1)*s_stride_x + tx] = (legalRow1)?shared_slices[slices_idx+(ty+1)*n_cols+tx]:0;
        shared_mem[(ty+1)*s_stride_x + tx+shared_area] = (legalRow1&&legalSlice1)?shared_slices[slices_idx+(ty+1)*n_cols+tx+sslices_area]:0;
    }
    if(tx==1){
        shared_mem[(ty+1)*s_stride_x + blockDim.x + tx] = (legalRow1&&legalColNX)?shared_slices[slices_idx+(ty+1)*n_cols+tx+blockDim.x]:0;
        shared_mem[(ty+1)*s_stride_x + blockDim.x + tx+shared_area] = (legalRow1&&legalColNX&&legalSlice1)?shared_slices[slices_idx+(ty+1)*n_cols+tx+blockDim.x+sslices_area]:0;
    }
    //----copy third plane from shared_rows, shared_cols, dst----//
    int srows_area = gridDim.y*n_cols*2;
    int scols_area = gridDim.x*n_rows*2;
    int dst_idx= base_global_idx + global_area*2;
    int s_idx  = shared_area*2;
    int srows_idx = srows_area*(base_global_slice+2) + blockIdx.y*n_cols*2 +blockIdx.x*tile_x;
    int scols_idx = scols_area*(base_global_slice+2) + blockIdx.x*n_rows*2 +blockIdx.y*tile_y;
    if(legalSlice2){
        //--copy x=1~blockDim.x, y=1~blockDim.y from dst to shared_mem--//
        if(ty>0){
            shared_mem[s_idx+(ty+1)*s_stride_x+tx+1] = (legalCol1&&legalRow1)?dst[dst_idx+(ty+1)*n_cols+tx+1]:0; 
        }
        //--copy y=0, y=blockDim.y+1 , x=1~blockDim.x from shared_rows to shared_mem--/
        if(ty==0){ //y=0, x=1~blockDim.x
            shared_mem[s_idx+tx+1] = (legalCol1)?shared_rows[srows_idx+tx+1]:0;
            shared_mem[s_idx+(ty+1)*s_stride_x+tx+1] = (legalCol1&&legalRow1)?shared_rows[srows_idx+(ty+1)*n_cols+tx+1]:0;
        }
        if(ty==1){//y=blockDim.y+1, x=1~blockDim.x
            shared_mem[s_idx+(ty+blockDim.y)*s_stride_x+tx+1] = (legalCol1&&legalRowN2)?shared_rows[srows_idx+(ty+2)*n_cols+tx+1]:((legalCol1&&legalRowNY)?dst[dst_idx+(ty+blockDim.y)*n_cols+tx+1]:0);
        }
        //--copy x=0, x=blockDim.x+1, y= 0~blockDim.y+1 from shared_cols to shared_mem--/
        //--[0,0],[0,blockDim.y+1],[blockDim.x+1,0],[blockDim.x+1, blockDim.y+1] never be used
        if(tx==0){  //x=0, y=1~blockDim.y   
            shared_mem[s_idx+(ty+1)*s_stride_x] = (legalRow1)?shared_cols[scols_idx+ty+1]:0;
        }
        if(tx==1){  //x=blockDim.x+1,y=1~blockDim.y
            shared_mem[s_idx+(ty+1)*s_stride_x+tx+blockDim.x] = (legalColN2&&legalRow1)?shared_cols[scols_idx+(tx+2)*n_rows+ty+1]:((legalColNX&&legalRow1)?dst[dst_idx+(ty+1)*n_cols+tx+blockDim.x]:0);
        }
    }
    
	__syncthreads();

#ifdef CUDA_CUDA_DEBUG
	if(blockIdx.z==0 && blockIdx.x==0 && blockIdx.y==0 ){
        if(threadIdx.y==0 || threadIdx.y==1 || threadIdx.y ==2){
            int s_s=2;
//            printf("blockIdx.x=%d,blockIdx.y=%d,blockIdx.z=%d, shared_mem   : addr: z:%d,y:%d,x=%d,val: %f\n",blockIdx.x,blockIdx.y,blockIdx.z,0,threadIdx.y,threadIdx.x,shared_mem[0*shared_area+threadIdx.y*s_stride_x+threadIdx.x]);
//            printf("blockIdx.x=%d,blockIdx.y=%d,blockIdx.z=%d, shared_mem   : addr: z:%d,y:%d,x=%d,val: %f\n",blockIdx.x,blockIdx.y,blockIdx.z,1,blockIdx.x,blockIdx.y, threadIdx.y,threadIdx.x,shared_mem[shared_area+threadIdx.y*s_stride_x+threadIdx.x]);
//            printf("blockIdx.x=%d,blockIdx.y=%d,blockIdx.z=%d, shared_mem   : addr: z:%d,y:%d,x=%d,val: %f\n",blockIdx.x,blockIdx.y,blockIdx.z,2,blockIdx.x,blockIdx.y, threadIdx.y,threadIdx.x,shared_mem[2*shared_area+threadIdx.y*s_stride_x+threadIdx.x]);
        }
        if(threadIdx.x==0&&threadIdx.y==0){
            int addr= 2*shared_area+s_stride_x+1;
            int addr1=2*srows_area+n_cols+1;
            printf("shared_mem   : addr: %d,val: %f\n",addr,shared_mem[addr]);
            printf("shared_rows  : addr: %d,val: %f\n",addr1,shared_rows[addr1]);
        }
    }

	__syncthreads();
#endif
    //====================================copy first 3 slices to shared_mem[]=================================//

    //==============================compute plus copy 1 slices to shared_mem[]===============================//

    int center = 1;
    int north  = 0;
    int south  = 2;
    int curSlice;
    int lenSlice = (legalSliceN)? (tile_z):(n_slices-base_global_slice-1);
    for (curSlice = HALO; curSlice < lenSlice ; curSlice+=1){
        //----compute slice----//
        if(legalCol2 && legalRow2){
            dst[base_global_idx + curSlice*global_area + ty1*n_cols + tx1] = 
                 (   shared_mem[center*shared_area+ty1*s_stride_x+tx]  + shared_mem[center*shared_area+ty1*s_stride_x+tx+2]
                 +   shared_mem[center*shared_area+ty*s_stride_x+tx1]  + shared_mem[center*shared_area+(ty+2)*s_stride_x+tx1]
                 +   shared_mem[north*shared_area +ty1*s_stride_x+tx1] + shared_mem[south*shared_area+ty1*s_stride_x+tx1]
                 +   shared_mem[center*shared_area+ty1*s_stride_x+tx1] )/7.5 ; 
        }
		__syncthreads();

#ifdef CUDA_CUDA_DEBUG
    	if(blockIdx.z==0 && blockIdx.x==0 && blockIdx.y==0 ){
            if((threadIdx.y==0)&&(threadIdx.x==0)){
                printf("dst addr: %d\n", base_global_idx + curSlice*global_area + ty1*n_cols + tx1);
                //printf("curSlice: %d, lenSlice: %d\n", curSlice,lenSlice);
                printf("curSlice: %d, lenSlice: %d,gridDim.x=%d,gridDim.y=%d,gridDim.z=%d\n", curSlice,lenSlice,gridDim.x,gridDim.y,gridDim.z);
                printf("blockIdx.x=%d,blockIdx.y=%d,blockIdx.z=%d, shared_mem   : addr: z:%d,y:%d,x=%d,addr: %d, val: %f\n",blockIdx.x,blockIdx.y,blockIdx.z,center,ty1,tx,center*shared_area+ty1*s_stride_x+tx,shared_mem[center*shared_area+ty1*s_stride_x+tx]);
                printf("blockIdx.x=%d,blockIdx.y=%d,blockIdx.z=%d, shared_mem   : addr: z:%d,y:%d,x=%d,addr: %d,val: %f\n",blockIdx.x,blockIdx.y,blockIdx.z,center,ty1,tx+2,center*shared_area+ty1*s_stride_x+tx+2,shared_mem[center*shared_area+ty1*s_stride_x+tx+2]);
                printf("blockIdx.x=%d,blockIdx.y=%d,blockIdx.z=%d, shared_mem   : addr: z:%d,y:%d,x=%d,addr: %d,val: %f\n",blockIdx.x,blockIdx.y,blockIdx.z,center,ty,tx1,center*shared_area+ty*s_stride_x+tx1,shared_mem[center*shared_area+ty*s_stride_x+tx1]);
                printf("blockIdx.x=%d,blockIdx.y=%d,blockIdx.z=%d, shared_mem   : addr: z:%d,y:%d,x=%d,addr: %d,val: %f\n",blockIdx.x,blockIdx.y,blockIdx.z,center,ty+2,tx1,center*shared_area+(ty+2)*s_stride_x+tx1,shared_mem[center*shared_area+(ty+2)*s_stride_x+tx1]);
                printf("blockIdx.x=%d,blockIdx.y=%d,blockIdx.z=%d, shared_mem   : addr: z:%d,y:%d,x=%d,addr: %d,val: %f\n",blockIdx.x,blockIdx.y,blockIdx.z,north,ty1,tx1,north*shared_area+(ty1)*s_stride_x+tx1,shared_mem[north*shared_area+(ty1)*s_stride_x+tx1]);
                printf("blockIdx.x=%d,blockIdx.y=%d,blockIdx.z=%d, shared_mem   : addr: z:%d,y:%d,x=%d,addr: %d,val: %f\n",blockIdx.x,blockIdx.y,blockIdx.z,south,ty1,tx1,south*shared_area+(ty1)*s_stride_x+tx1,shared_mem[south*shared_area+(ty1)*s_stride_x+tx1]);
                printf("blockIdx.x=%d,blockIdx.y=%d,blockIdx.z=%d, shared_mem   : addr: z:%d,y:%d,x=%d,addr: %d,val: %f\n",blockIdx.x,blockIdx.y,blockIdx.z,center,ty1,tx1,center*shared_area+(ty1)*s_stride_x+tx1,shared_mem[center*shared_area+(ty1)*s_stride_x+tx1]);
            }
//            if(threadIdx.x==0 && threadIdx.y==0){
//                int addr = 10*n_rows*n_cols + n_cols+1;
//                int addr1 = 10*srows_area + 2*n_cols+1;
//                printf("dst addr: %d,val: %f\n", addr, dst[addr] );
//                printf("shared_row addr: %d,val: %f\n", addr1, shared_rows[addr1] );
//            }
//                
//            if(legalCol1 && legalRow1){
//                printf("addr: %d\n",base_global_idx+curSlice*global_area+ty1*n_cols+tx1 );
//            }
        }
#endif
        //----copy next slice to shared_mem[]----//
        int ssSlice = curSlice+2;
        int g_ssSlice = base_global_slice+ssSlice;
        srows_idx = srows_area*g_ssSlice + blockIdx.y*n_cols*2 +blockIdx.x*tile_x;
        scols_idx = scols_area*g_ssSlice + blockIdx.x*n_rows*2 +blockIdx.y*tile_y;
        bool legalSliceSS = g_ssSlice<n_slices;
        dst_idx= base_global_idx + global_area*ssSlice;
        s_idx  = shared_area*north;
        if(legalSliceSS){
            //--copy x=1~blockDim.x, y=1~blockDim.y from dst to shared_mem--//
            if(ty>0){
                shared_mem[s_idx+(ty+1)*s_stride_x+tx+1] = (legalCol1&&legalRow1)?dst[dst_idx+(ty+1)*n_cols+tx+1]:0; 
            }
            //--copy y=0, y=blockDim.y+1 , x=1~blockDim.x from shared_rows to shared_mem--/
            if(ty==0){ //y=0, x=1~blockDim.x
                shared_mem[s_idx+tx+1] = (legalCol1)?shared_rows[srows_idx+tx+1]:0;
                shared_mem[s_idx+(ty+1)*s_stride_x+tx+1] = (legalCol1&&legalRow1)?shared_rows[srows_idx+(ty+1)*n_cols+tx+1]:0;
            }
            if(ty==1){//y=blockDim.y+1, x=1~blockDim.x
                shared_mem[s_idx+(ty+blockDim.y)*s_stride_x+tx+1] = (legalCol1&&legalRowN2)?shared_rows[srows_idx+(ty+2)*n_cols+tx+1]:((legalCol1&&legalRowNY)?dst[dst_idx+(ty+blockDim.y)*n_cols+tx+1]:0);
            }
            //--copy x=0, x=blockDim.x+1, y= 0~blockDim.y+1 from shared_cols to shared_mem--/
            //--[0,0],[0,blockDim.y+1],[blockDim.x+1,0],[blockDim.x+1, blockDim.y+1] never be used
            if(tx==0){  //x=0, y=1~blockDim.y   
                shared_mem[s_idx+(ty+1)*s_stride_x] = (legalRow1)?shared_cols[scols_idx+ty+1]:0;
            }
            if(tx==1){  //x=blockDim.x+1,y=1~blockDim.y
                shared_mem[s_idx+(ty+1)*s_stride_x+tx+blockDim.x] = (legalColN2&&legalRow1)?shared_cols[scols_idx+(tx+2)*n_rows+ty+1]:((legalColNX&&legalRow1)?dst[dst_idx+(ty+1)*n_cols+tx+blockDim.x]:0);
            }
        }
        
        center = ROTATE_UP(center,3);
		south  = ROTATE_UP(south,3);
		north  = ROTATE_UP(north,3);
		__syncthreads();
    }

    //==============================compute plus copy 1 slices to shared_mem[]===============================//

    //=========================copy plus compute last slice in one grid to shared_mem[]==========================//
    int g_nnSlice = base_global_slice+ tile_z + 1;
    bool legalSliceNN = g_nnSlice < n_slices;
    if(legalSliceNN){
        //----copy----//
        //--copy x: 1~blockDim.x , y: 1~blockDimy
        slices_idx = (blockIdx.z+1)*global_area*2 +global_area + blockIdx.y*tile_y*n_cols + blockIdx.x*tile_x;
        
        s_idx = south*shared_area;

        //--copy x: 1~blockDim.x , y: 1~blockDimy
        shared_mem[s_idx+(ty+1)*s_stride_x+tx+1] = (legalRow1&&legalCol1)?shared_slices[slices_idx+(ty+1)*n_cols+tx+1]:0;
        
        //--copy y=0,y=blockDim.y+1, x=1~blockDim.x --//
        if(ty ==0){
            shared_mem[s_idx+ty*s_stride_x+ tx+1] = (legalRow&&legalCol1)?shared_slices[slices_idx+ty*n_cols+tx+1]:0;
        }
        if(ty==1){
            shared_mem[s_idx+(blockDim.y+ty)*s_stride_x+ tx+1] = (legalRowNY&&legalCol1)?shared_slices[slices_idx+(ty+blockDim.y)*n_cols+tx+1]:0;
        }
        //--copy x= 0, x=blockDim.x+1, y=1~blockDim.x--//
        if(tx==0){
            shared_mem[s_idx+(ty+1)*s_stride_x + tx] = (legalRow1)?shared_slices[slices_idx+(ty+1)*n_cols+tx]:0;
        }
        if(tx==1){
            shared_mem[s_idx+(ty+1)*s_stride_x + blockDim.x + tx] = (legalRow1&&legalColNX)?shared_slices[slices_idx+(ty+1)*n_cols+tx+blockDim.x]:0;
        }
        __syncthreads();
        //----compute----//
        if(legalCol2 && legalRow2){
            dst[base_global_idx + curSlice*global_area + ty1*n_cols + tx1] = 
                 (   shared_mem[center*shared_area+ty1*s_stride_x+tx]  + shared_mem[center*shared_area+ty1*s_stride_x+tx+2]
                 +   shared_mem[center*shared_area+ty*s_stride_x+tx1]  + shared_mem[center*shared_area+(ty+2)*s_stride_x+tx1]
                 +   shared_mem[north*shared_area +ty1*s_stride_x+tx1] + shared_mem[south*shared_area+ty1*s_stride_x+tx1]
                 +   shared_mem[center*shared_area+ty1*s_stride_x+tx1] )/7.5 ; 
        }
    }

	__syncthreads();
    //=========================copy plus compute last slice in one grid to shared_mem[]==========================//
#ifdef CUDA_DARTS_DEBUG

	if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.x==0)&&(threadIdx.y==0)&&(threadIdx.z==0)){
		printf("3D kernel finish!\n");
	}
#endif

}


void gpu_kernel37(dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedRows, double * sharedCols, double * sharedSlices,int n_rows,int n_cols, int n_slices,int tile_x, int tile_y, int tile_z){
	int sharedMemSize = sizeof(double)*(1+HALO*2)*((tile_x+2)*(tile_y+2));
#ifdef CUDA_DARTS_DEBUG
		printf("sharedMemSize: %d B, total sharedMemSize: %d B\n",sharedMemSize, sharedMemSize*dimGrid.x*dimGrid.y*dimGrid.z);
		printf("gpu_kernel37: dimGrid.x= %d dimGrid.y= %d, dimGrid.z= %d\n",dimGrid.x,dimGrid.y,dimGrid.z);
#endif
		gpu_stencil37_hack2<<<dimGrid,dimBlock,sharedMemSize>>>(d_dst,sharedRows,sharedCols,sharedSlices,n_rows,n_cols,n_slices,tile_x,tile_y,tile_z);
#ifdef CUDA_DARTS_DEBUG
		printf("gpu kernel37 return to host, but kernel haven't finished!\n");
#endif
}


void gpu_kernel37_stream(hipStream_t &stream, dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedRows, double * sharedCols, double * sharedSlices,int n_rows,int n_cols, int n_slices,int tile_x, int tile_y, int tile_z){
	int sharedMemSize = sizeof(double)*(1+HALO*2)*((tile_x+2)*(tile_y+2));
#ifdef CUDA_DARTS_DEBUG
		printf("sharedMemSize: %d B, total sharedMemSize: %d B\n",sharedMemSize, sharedMemSize*dimGrid.x*dimGrid.y*dimGrid.z);
		printf("gpu_kernel37: dimGrid.x= %d dimGrid.y= %d, dimGrid.z= %d\n",dimGrid.x,dimGrid.y,dimGrid.z);
#endif
		gpu_stencil37_hack2<<<dimGrid,dimBlock,sharedMemSize,stream>>>(d_dst,sharedRows,sharedCols,sharedSlices,n_rows,n_cols,n_slices,tile_x,tile_y,tile_z);
#ifdef CUDA_DARTS_DEBUG
		printf("gpu kernel37 return to host, but kernel haven't finished!\n");
#endif
}



