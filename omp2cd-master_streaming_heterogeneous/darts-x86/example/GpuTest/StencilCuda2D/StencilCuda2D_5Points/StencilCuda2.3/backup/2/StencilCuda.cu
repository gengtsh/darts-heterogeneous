#include "hip/hip_runtime.h"
extern "C" {
#include <hip/hip_runtime.h>
#include "conf.h"
#include "stencil.h"
}
#include <stdio.h>
#define ROTATE_DOWN(val,MAX) ((val-1==-1)?MAX-1:val-1)
#define ROTATE_UP(val,MAX) ((val+1)%MAX)
/**
  * GPU Device kernel for the for 2D stencil
  * First attempt during hackaton
  * M = Rows, N = Cols INCLUDING HALOS
  * In this version now we replace the size of the shared memory to be just 3 rows (actually 1+HALO*2) rows 
  */

//__global__ void gpu_stencil2D_4pt_hack4(double * dst, double * src, int M, int N)
//{
////	printf("kernel begin!\n");
//	//Declaring the shared memory array for source
//	extern	__shared__ double shared_mem[] ;
//
//	//indexes
//	int i, j, k,curRow;
//                           //Cols   *  numRows/Tile * tileIndex  
//	int base_global_row = ( N ) * ( GRID_TILE_Y * blockIdx.y ); 
//	int base_global_col = blockDim.x*blockIdx.x;
//	int base_global_idx = base_global_row + base_global_col ;
//	int center = 1,north = 0,south = 2; //indexes for the current location in the shared memory
//	int t = threadIdx.x;
//	
//	//copy the shared memory to fill the pipeline
//	bool rowLeft = (blockIdx.y==(gridDim.y-1))&&(M%GRID_TILE_Y<3)&&(M%GRID_TILE_Y>0);
//	int  numRowLeft =(rowLeft)?(3-M%GRID_TILE_Y):0;
//	bool noColsLeft = (base_global_col +t )<N;
//	bool noColsLeft2= (base_global_col+t+2)<N;
//	for (i = 0 ; i < 1+HALO*2-numRowLeft ; i ++ ){
//		k = base_global_idx+i*N+t;
//		j = i*(blockDim.x+2) + t;
//		shared_mem [j] = (noColsLeft)?src[k]:0.0;
//		if((t<2) &&(noColsLeft)){
//			shared_mem[j+blockDim.x]=src[k+blockDim.x];
//		}
//	}
//		
//	__syncthreads();
//
//	int tt = (((blockIdx.y+1)*GRID_TILE_Y)>M)?(M%GRID_TILE_Y): GRID_TILE_Y;
//	int ss = (((M%GRID_TILE_Y)==1)&&(blockIdx.y ==(gridDim.y-2)))?-1:0;
//
//	int lastRow = ((blockIdx.y == (gridDim.y-1))?-1:1)+tt +ss ;
////	printf("lastRow:%d \n",lastRow );
//	//Pipelined copy one row and process it
//	for ( curRow = HALO; curRow < lastRow; curRow+=1 )
//	{
//		//Stencil computation
//							//top             + bottom              + left                + right
//		j = threadIdx.x+HALO;
//		if(noColsLeft2){
//			dst[base_global_idx + curRow*N + j] =(shared_mem[north*(blockDim.x+2)+j] + shared_mem[south*(blockDim.x+2)+j] + shared_mem[center*(blockDim.x+2)+j-1] + shared_mem[center*(blockDim.x+2)+j+1] )/5.5;
//		}
//		__syncthreads();
//		//We are copying from src to shared memory.
//		k=base_global_idx+(curRow+2)*N+threadIdx.x;
//		if(k<M*N){
//			shared_mem [north*(blockDim.x+2)+threadIdx.x] =(noColsLeft)? src[k]:0.0;
//		}
//		if((t<2)&&(noColsLeft)&&(k<M*N)){
//			shared_mem[north*(blockDim.x+2)+threadIdx.x+blockDim.x]=src[k+blockDim.x];
//		}	
//		center = ROTATE_UP(center,3);
//		south  = ROTATE_UP(south,3);
//		north  = ROTATE_UP(north,3);
//		__syncthreads();
//	}
//
////	printf("kernel finish!\n");
//}



__global__ void gpu_stencil2D_4pt_hack4(double * dst, double * src, int M, int N)
{
//	printf("kernel begin!\n");
	//Declaring the shared memory array for source
	extern	__shared__ double shared_mem[] ;

	//indexes
	int i,j, k,curRow;
                           //Cols   *  numRows/Tile * tileIndex  
	int base_global_row = ( GRID_TILE_Y * blockIdx.y ); 
	int base_global_col = blockDim.x*blockIdx.x;
	int base_global_idx = N*base_global_row + base_global_col ;
	int center = 1,north = 0,south = 2; //indexes for the current location in the shared memory
	int t = threadIdx.x;
	
	//copy the shared memory to fill the pipeline
	bool legalCol = (base_global_col +t )<N;
	bool legalCol2= (base_global_col+t+2)<N;
	bool legalColn= (base_global_col+t+blockDim.x)<N;
	for (i = 0 ; i < 1+HALO*2 ; i ++ ){
		k = base_global_idx+i*N+t;
		j = i*(blockDim.x+2) + t;
		bool legalRow = (base_global_row+i)<M;
		shared_mem [j] =legalRow?( legalCol?src[k]:0.0):0.0;
		if((t<2)&&legalColn&&legalRow){
			shared_mem[j+blockDim.x]=src[k+blockDim.x];
		}
	}
		
	__syncthreads();

	//Pipelined copy one row and process it
	for ( curRow = HALO; curRow < GRID_TILE_Y+1; curRow+=1 )
	{
		//Stencil computation
		//top + bottom + left + right

		j = threadIdx.x+HALO;
		bool legalRow1 =( base_global_row+curRow+1)<M;
		if((legalCol2)&&(legalRow1)){
			dst[base_global_idx + curRow*N + j] =(shared_mem[north*(blockDim.x+2)+j] + shared_mem[south*(blockDim.x+2)+j] + shared_mem[center*(blockDim.x+2)+j-1] + shared_mem[center*(blockDim.x+2)+j+1] )/5.5;
		}
		__syncthreads();
		//We are copying from src to shared memory.
		int nextRow2 = base_global_row+curRow+2;
		bool legalRow2 = nextRow2<M;
		k = base_global_col+nextRow2*N+t;

		shared_mem [north*(blockDim.x+2)+t] =(legalRow2&&legalCol)?src[k]:0.0;

		if((t<2)&&legalColn&&legalRow2){
			shared_mem[north*(blockDim.x+2)+t+blockDim.x]=src[k+blockDim.x];
		}	
		
		center = ROTATE_UP(center,3);
		south  = ROTATE_UP(south,3);
		north  = ROTATE_UP(north,3);
		__syncthreads();
	}

//	printf("kernel finish!\n");
}


__global__ void gpu_stencil2D_4pt_hack2(double * dst, double * src, int M, int N)
{
//	printf("kernel begin!\n");
	//Declaring the shared memory array for source
	__shared__ double shared_mem[ 1 + HALO*2 ] [ GRID_TILE_X + HALO*2]; //1 is the row I am modifying
	//double * shSrc = shared_mem;

	//indexes
	int i, j, curRow;
                           //Cols   *  numRows/Tile * tileIndex  
	int base_global_idx = ( N ) * ( GRID_TILE_Y * blockIdx.y ) + GRID_TILE_X*blockIdx.x;
	
	int center = 1,north = 0,south = 2; //indexes for the current location in the shared memory

	//copy the shared memory to fill the pipeline
	for (i = 0 ; i < 1+HALO*2 ; i ++ )
		for (j = threadIdx.x ; j < GRID_TILE_X+2*HALO ; j+=blockDim.x)
		{
			shared_mem [i][j] = src[base_global_idx + i*N + j];
		}
	__syncthreads();
	//Pipelined copy one row and process it
	for ( curRow = HALO; curRow < GRID_TILE_Y; curRow+=1 )
	{
		//Stencil computation
		for (j = threadIdx.x + HALO ; j < GRID_TILE_X+HALO ; j+=blockDim.x)
		{
							//top             + bottom              + left                + right
			dst[base_global_idx + curRow*N + j] = (shared_mem[north][j] + shared_mem[south][j] + shared_mem[center][j-1] + shared_mem[center][j+1] )/5.5;
		}
		
		__syncthreads();
		//We are copying from dst to shared memory.
		for (j = threadIdx.x ; j < GRID_TILE_X+2*HALO ; j+=blockDim.x)
		{
			shared_mem [north][j] = src[base_global_idx + (curRow+2)*N + j];
		}
	
		center = ROTATE_UP(center,3);
		south  = ROTATE_UP(south,3);
		north = ROTATE_UP(north,3);
		__syncthreads();
	}

	//Dranning the pipeline
	for (j = threadIdx.x + HALO ; j < GRID_TILE_X+HALO ; j+=blockDim.x)
	{
							//top             + bottom              + left                + right
		dst[base_global_idx + curRow*N + j] = (shared_mem[north][j] + shared_mem[south][j] + shared_mem[center][j-1] + shared_mem[center][j+1] )/5.5;
	}
	__syncthreads();

//	printf("kernel finish!\n");
}



///**
//  * GPU Device kernel for the for 2D stencil
//  * First attempt during hackaton
//  * M = Rows, N = Cols INCLUDING HALOS
//  */
//__global__ void gpu_stencil2D_4pt_hack1(double * dst, double * src, int M, int N)
//{
//
//	//Declaring the shared memory array for source
//	__shared__ double shared_mem[GRID_TILE_Y + HALO*2 ] [ GRID_TILE_X + HALO*2];
//	//double * shSrc = shared_mem;
//
//	//indexes
//	int i, j;
//
//                           //Cols   *  numRows/Tile * tileIndex  
//	int base_global_idx = ( N ) * ( GRID_TILE_Y * blockIdx.y ) + GRID_TILE_X*blockIdx.x;
//
//	//We are copying from dst to shared memory.
//	for (i = 0 ; i < GRID_TILE_Y+2*HALO ; i ++ )
//		for (j = threadIdx.x ; j < GRID_TILE_X+2*HALO ; j+=blockDim.x)
//		{
//			shared_mem [i][j] = src[base_global_idx + i*N + j];
//		}
//
//	__syncthreads();
//
//	//Stencil computation
//	for (i = HALO ; i < GRID_TILE_Y+HALO ; i ++ )
//		for (j = threadIdx.x + HALO ; j < GRID_TILE_X+HALO ; j+=blockDim.x)
//		{
//			                                //top             + bottom              + left                + right
//			dst[base_global_idx + i*N + j] = (shared_mem[i-1][j] + shared_mem[i+1][j] + shared_mem[i][j-1] + shared_mem[i][j+1] )/5.5;
//		}
//
//	__syncthreads();
//}




/**
  * GPU Device kernel for the for 2D stencil
  * M = Rows, N = Cols
  */
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
	//Declaring the shared memory array for source
	extern __shared__ double shared_mem[];
	double * shSrc = shared_mem;

	//indexes
	int i, j;

	//neighbor's values 
	double north, south, east, west;



	//SharedMem Collumns Dimension
	int smColDim = HALO*2+blockDim.y*TILE_SIZE;
	int smRowDim = HALO*2+blockDim.x*TILE_SIZE;

	//Copying to shared memory

	//Inner part
	for ( i = 0 ; i < TILE_SIZE ; i++ )
	{
		for ( j = 0 ; j < TILE_SIZE ; j++ )
		{
			int globalIndex=HALO*N+blockIdx.x*blockDim.x*TILE_SIZE*N+threadIdx.x*TILE_SIZE*N+i*N+blockIdx.y*blockDim.y*TILE_SIZE+threadIdx.y*TILE_SIZE+j+HALO;
			int shMemIndex=HALO*smColDim+threadIdx.x*smColDim*TILE_SIZE+i*smColDim+HALO+threadIdx.y*TILE_SIZE+j;
			shSrc[shMemIndex]=src[globalIndex];
		}
	}

	//Halos

	if (threadIdx.x == 0 && threadIdx.y == 0 ) 
	{

		int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;
		//For Bottom and top row
		for ( i = 0 ; i < HALO ; i++ )
		{
			for ( j = 0 ; j < smColDim ; j++ )
			{
				indexTopHalo = (blockIdx.x*blockDim.x*TILE_SIZE+i)*N + (blockIdx.y*blockDim.y*TILE_SIZE) + j;
				indexBottomHalo = (HALO + (blockIdx.x+1)*blockDim.x*TILE_SIZE)*N + (blockIdx.y*blockDim.y*TILE_SIZE)+j;
				shSrc[i*smColDim+j] = src[indexTopHalo];
				shSrc[(HALO+blockDim.x*TILE_SIZE+i)*smColDim + j] = src[indexBottomHalo];
			}
		}
		
		//For right and left Columns
		for ( i = 0 ; i < HALO ; i++ )
		{
			for ( j = 0 ; j < smRowDim-HALO*2; j ++ )
			{
				indexLeftHalo = (HALO+blockIdx.x*blockDim.x*TILE_SIZE+j)*N + (blockIdx.y*blockDim.y*TILE_SIZE)+i;
				indexRightHalo = (HALO+blockIdx.x*blockDim.x*TILE_SIZE+j)*N + ((blockIdx.y+1)*blockDim.y*TILE_SIZE)+HALO+i;
				shSrc[(HALO+j)*smColDim+i] = src[indexLeftHalo];
				shSrc[(HALO+j+1)*smColDim-HALO+i] = src[indexRightHalo];
			}
		}
	}

	__syncthreads();



	for ( i = 0 ; i < TILE_SIZE ; i++ )
	{
		for ( j = 0 ; j < TILE_SIZE ; j++ )
		{
			int globalIndex=HALO*N+blockIdx.x*blockDim.x*TILE_SIZE*N+threadIdx.x*TILE_SIZE*N+i*N+blockIdx.y*blockDim.y*TILE_SIZE+threadIdx.y*TILE_SIZE+j+HALO;
			int shMemIndex=HALO*smColDim+threadIdx.x*smColDim*TILE_SIZE+i*smColDim+HALO+threadIdx.y*TILE_SIZE+j;


			//Getting the neighbohrs
			north = shSrc[shMemIndex-smColDim];
			south = shSrc[shMemIndex+smColDim];
			east  = shSrc[shMemIndex+1];
			west  = shSrc[shMemIndex-1];
			//Real Stencil operation
			dst[globalIndex] = ( north + south + east + west )/5.5;
//			dst[globalIndex] = ( north + south + east + west )/4;
		}
	}

	__syncthreads();
}




/**
 *  Naïve 4pt stencil code for 2D arrays. 
 */
void
stencil2D4pt ( double* __restrict__ dst,    double* __restrict__ src, 
               const size_t     n_rows, const size_t     n_cols,
               const size_t     n_tsteps )
{
    typedef double (*Array2D)[n_cols];
    volatile Array2D DST = (Array2D) dst,
            SRC = (Array2D) src;
    for (size_t ts = 0; ts < n_tsteps; ++ts) {
        for (size_t i = 1; i < n_rows-1; ++i) {
            for (size_t j = 1; j < n_cols-1; ++j) {
                DST[i][j] = (SRC[i-1][j] + SRC[i+1][j] + SRC[i][j-1] + SRC[i][j+1])/5.5;
            }
        }
        SWAP_PTR(&DST,&SRC);
    }
}

extern "C"
void
stencil2D4pt_gpu( double * __restrict__ dst, double* __restrict__ src,
		  const size_t M, const size_t N, 
		  const size_t NUM_ITERATIONS)//M Rows by N Columns
{
		
	double size = sizeof(double) * M * N;

	//device memory allocation
	double * d_dst, * d_src;
	hipMalloc( (void **) &d_dst, size);
	hipMalloc( (void **) &d_src, size);
	
	//dimmensions for indexes
	// TODO the -2 is to remove the borders
	dim3 dimBlock(MAX_BLOCK_DIM,MAX_BLOCK_DIM);
	int gridx = (N-2)/(MAX_BLOCK_DIM*TILE_SIZE) + (((N-2)%(MAX_BLOCK_DIM*TILE_SIZE) == 0)? 0:1 ) ;
	int gridy = (M-2)/(MAX_BLOCK_DIM*TILE_SIZE) + (((M-2)%(MAX_BLOCK_DIM*TILE_SIZE) == 0)? 0:1 ) ;
	dim3 dimGrid(gridx,gridy);

	//Shared memory size = inside + halo
	int shMemSize=MAX_BLOCK_DIM*TILE_SIZE*MAX_BLOCK_DIM*TILE_SIZE*sizeof(double)+(HALO*MAX_BLOCK_DIM*TILE_SIZE+HALO*HALO)*4*sizeof(double);
	
	//Hackaton dimensions
	dim3 dimGrid_hack1((N-HALO*2)/GRID_TILE_X,(M-HALO*2)/GRID_TILE_Y);

	//Copying the device memory
	hipMemcpy(d_src, src, size, hipMemcpyHostToDevice);
	hipMemcpy(d_dst, dst, size, hipMemcpyHostToDevice);

	//printf("CUDA Stencil Code running... cycles = %d. dim = %d by %d \n",NUM_ITERATIONS,M,N);
	
    int time_step = NUM_ITERATIONS;

    while (time_step-- > 0) 
    {
    	//gpu_stencil2D_4pt<<<dimGrid,dimBlock,shMemSize>>>(d_dst,d_src,M,N);
		//gpu_stencil2D_4pt_hack1<<<dimGrid_hack1,NUM_THREADS>>>(d_dst,d_src,M,N); //JOSE Hackathon!
		//printf("before: d_src[10] = %ld",d_src[10]);

		gpu_stencil2D_4pt_hack2<<<dimGrid_hack1,NUM_THREADS>>>(d_dst,d_src,M,N);
		//Inline swapping.
		
		//printf("after: d_src[10] = %ld",d_src[10]);
		double * temp;
		if ( NUM_ITERATIONS%2 ==0 || time_step !=0)
		{
			temp=d_src;
			d_src=d_dst;
			d_dst=temp;
		}
	}
	
	
	//Copying memory back from device to DRAM
	//hipMemcpy(src, d_src, size, hipMemcpyDeviceToHost);
	hipMemcpy(dst, d_dst, size, hipMemcpyDeviceToHost);
	hipMemcpy(src, d_src, size, hipMemcpyDeviceToHost);
	
	//Free device memory
	hipFree(d_src); hipFree(d_dst);
}

//void*
//stencil_run(void* arg)
//{
//    stencil_t* stencil = (stencil_t*)arg;
//    STENCIL_COMPUTE(stencil->stencil,stencil->arg);
//    return NULL;
//}


void gpu_kernel4(dim3 dimGrid,dim3 dimBlock,double * d_dst, double * d_src, int M, int N){
		int sharedMemSize = sizeof(double)*(1+HALO*2)*(dimBlock.x+2);
#ifdef CUDA_DARTS_DEBUG
		printf("sharedMemSize: %dKB, total sharedMemSize: %dKB\n",sharedMemSize/1024, sharedMemSize*dimGrid.x*dimGrid.y/1024);
#endif
		gpu_stencil2D_4pt_hack4<<<dimGrid,dimBlock,sharedMemSize>>>(d_dst,d_src,M,N);
}

void gpu_kernel1(dim3 dimGrid_hack1,double * d_dst, double * d_src, int M, int N){
		gpu_stencil2D_4pt_hack2<<<dimGrid_hack1,NUM_THREADS>>>(d_dst,d_src,M,N);
}

void gpu_kernel3(hipStream_t &stream,dim3 dimGrid_hack1,double * d_dst, double * d_src, int M, int N){
		gpu_stencil2D_4pt_hack2<<<dimGrid_hack1,NUM_THREADS,0,stream>>>(d_dst,d_src,M,N);

}

void gpu_kernel2(dim3 dimGrid_hack1,double *dst, double *src, double size, size_t ts, double * d_dst, double * d_src, int M, int N){
	double * tmp;
	while (--ts!=0){
		printf("ts:%ld \n", ts);
		gpu_stencil2D_4pt_hack2<<<dimGrid_hack1,NUM_THREADS>>>(d_dst,d_src,M,N);
		tmp = d_src;
		d_src = d_dst;
		d_dst=tmp;
	}
}
