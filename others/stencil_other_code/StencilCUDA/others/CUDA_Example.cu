//Example CUDA code, written and commented by Jose Monsalve
//Taken from CUDA C/C++ Basics
//Supercomputing 2011 Tutorial
//by NVIDIA

/**
  This code executes c=a+b in a single thread in a GPU device.
It is a really simple code that is intended to show the memory 
movement between host and device, but not the division of the 
work between the differetn GPU threads.

First, we define the kernel (function that is executed in the 
GPU device), second, we initialize the values in the host (CPU)
and we allocate (reserve) some memory in the GPU). Then we move 
these values to the GPU device through explicitely memory movement
and we start the execution of the add kernel. Finally, after the 
computation is done, we copy the information back from the GPU and 
display it
**/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


//Simple add kernel, this function will be executed in the GPU device.
//C=A+B
__global__ void add(int *a, int *b, int *c)
{
	extern __shared__ int shared_mem[];
	int * shmem=shared_mem;
	shmem[threadIdx.x]=threadIdx.x;
	a[threadIdx.x]=shmem[threadIdx.x];
	b[threadIdx.x]=shmem[threadIdx.x];
	c[threadIdx.x]=a[threadIdx.x]+b[threadIdx.x];
}

//main func
int main(void)
{
	//Initializing the host variables
	//The *d_X are pointers that are not accessible in the host
	//directly but they represent a way to refer to the data in the 
	//GPU device
	int a[10], b[10], c[10];
	int *d_a, *d_b, *d_c;
	int size = sizeof(int)*10;

	//Initial values in the host 

	//Initializing memory in the GPU device, 
	//reserving the space, but there is not value yet (requires explicit movement)
	hipMalloc ( (void **) &d_a, size );
	hipMalloc ( (void **) &d_b, size );
	hipMalloc ( (void **) &d_c, size );
	

	//moving the values that we just initialize in the host
	//to the GPU device (Explicit memory movement)
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	printf("Adding %d and %d in the device ... \n",a,b);
	
	//starting the kernel function
	add<<<1,10,10*sizeof(int)>>>(d_a,d_b,d_c);
	
	//Bringing back the result, which is stored in the GPU device 
	//And needs to be manually obtained.
	hipMemcpy(c,d_c,size, hipMemcpyDeviceToHost);

	printf("Result is \n");
	//printing the result
	for ( int i = 0; i < 10 ; i++)
		printf("%d\t",c[i]);
	printf("\n");
	
	//cleaning device.
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
