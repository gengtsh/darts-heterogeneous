#include "hip/hip_runtime.h"
extern "C" {
#include <hip/hip_runtime.h>
#include "conf.h"
#include "stencil.h"
}
#include <stdint.h>
#include <stdio.h>
#define ROTATE_DOWN(val,MAX) ((val-1==-1)?MAX-1:val-1)
#define ROTATE_UP(val,MAX) ((val+1)%MAX)
/**
  * GPU Device kernel for the for 2D stencil
  * First attempt during hackaton
  * M = Rows, N = Cols INCLUDING HALOS
  * In this version now we replace the size of the shared memory to be just 3 rows (actually 1+HALO*2) rows 
  */

//__global__ void gpu_stencil2D_4pt_hack4(double * dst, double * src, int M, int N)
//{
////	printf("kernel begin!\n");
//	//Declaring the shared memory array for source
//	extern	__shared__ double shared_mem[] ;
//
//	//indexes
//	int i, j, k,curRow;
//                           //Cols   *  numRows/Tile * tileIndex  
//	int base_global_row = ( N ) * ( GRID_TILE_Y * blockIdx.y ); 
//	int base_global_col = blockDim.x*blockIdx.x;
//	int base_global_idx = base_global_row + base_global_col ;
//	int center = 1,north = 0,south = 2; //indexes for the current location in the shared memory
//	int t = threadIdx.x;
//	
//	//copy the shared memory to fill the pipeline
//	bool rowLeft = (blockIdx.y==(gridDim.y-1))&&(M%GRID_TILE_Y<3)&&(M%GRID_TILE_Y>0);
//	int  numRowLeft =(rowLeft)?(3-M%GRID_TILE_Y):0;
//	bool noColsLeft = (base_global_col +t )<N;
//	bool noColsLeft2= (base_global_col+t+2)<N;
//	for (i = 0 ; i < 1+HALO*2-numRowLeft ; i ++ ){
//		k = base_global_idx+i*N+t;
//		j = i*(blockDim.x+2) + t;
//		shared_mem [j] = (noColsLeft)?src[k]:0.0;
//		if((t<2) &&(noColsLeft)){
//			shared_mem[j+blockDim.x]=src[k+blockDim.x];
//		}
//	}
//		
//	__syncthreads();
//
//	int tt = (((blockIdx.y+1)*GRID_TILE_Y)>M)?(M%GRID_TILE_Y): GRID_TILE_Y;
//	int ss = (((M%GRID_TILE_Y)==1)&&(blockIdx.y ==(gridDim.y-2)))?-1:0;
//
//	int lastRow = ((blockIdx.y == (gridDim.y-1))?-1:1)+tt +ss ;
////	printf("lastRow:%d \n",lastRow );
//	//Pipelined copy one row and process it
//	for ( curRow = HALO; curRow < lastRow; curRow+=1 )
//	{
//		//Stencil computation
//							//top             + bottom              + left                + right
//		j = threadIdx.x+HALO;
//		if(noColsLeft2){
//			dst[base_global_idx + curRow*N + j] =(shared_mem[north*(blockDim.x+2)+j] + shared_mem[south*(blockDim.x+2)+j] + shared_mem[center*(blockDim.x+2)+j-1] + shared_mem[center*(blockDim.x+2)+j+1] )/5.5;
//		}
//		__syncthreads();
//		//We are copying from src to shared memory.
//		k=base_global_idx+(curRow+2)*N+threadIdx.x;
//		if(k<M*N){
//			shared_mem [north*(blockDim.x+2)+threadIdx.x] =(noColsLeft)? src[k]:0.0;
//		}
//		if((t<2)&&(noColsLeft)&&(k<M*N)){
//			shared_mem[north*(blockDim.x+2)+threadIdx.x+blockDim.x]=src[k+blockDim.x];
//		}	
//		center = ROTATE_UP(center,3);
//		south  = ROTATE_UP(south,3);
//		north  = ROTATE_UP(north,3);
//		__syncthreads();
//	}
//
////	printf("kernel finish!\n");
//}



__global__ void gpu_stencil2D_4pt_hack4(double * dst, double * src, int M, int N)
{
#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.x==0)){
		printf("kernel begin!\n");
	}
#endif
	//Declaring the shared memory array for source
	extern	__shared__ double shared_mem[] ;

	//indexes
	int i,j, k,curRow;
                           //Cols   *  numRows/Tile * tileIndex  
	int base_global_row = ( GRID_TILE_Y * blockIdx.y ); 
	int base_global_col = blockDim.x*blockIdx.x;
	int base_global_idx = N*base_global_row + base_global_col ;
	int center = 1,north = 0,south = 2; //indexes for the current location in the shared memory
	int t = threadIdx.x;
	
	//copy the shared memory to fill the pipeline
	bool legalCol = (base_global_col +t )<N;
	bool legalCol2= (base_global_col+t+2)<N;
	bool legalColn= (base_global_col+t+blockDim.x)<N;
	for (i = 0 ; i < 1+HALO*2 ; i ++ ){
		k = base_global_idx+i*N+t;
		j = i*(blockDim.x+2) + t;
		bool legalRow = (base_global_row+i)<M;
		shared_mem [j] =legalRow?( legalCol?src[k]:0.0):0.0;
		if((t<2)&&legalColn&&legalRow){
			shared_mem[j+blockDim.x]=src[k+blockDim.x];
		}
	}
		
	__syncthreads();

	//Pipelined copy one row and process it
	for ( curRow = HALO; curRow < GRID_TILE_Y+1; curRow+=1 )
	{
		//Stencil computation
		//top + bottom + left + right

		j = threadIdx.x+HALO;
		bool legalRow1 =( base_global_row+curRow+1)<M;
		if((legalCol2)&&(legalRow1)){
			dst[base_global_idx + curRow*N + j] =(shared_mem[north*(blockDim.x+2)+j] + shared_mem[south*(blockDim.x+2)+j] + shared_mem[center*(blockDim.x+2)+j-1] + shared_mem[center*(blockDim.x+2)+j+1] )/5.5;
		}
		__syncthreads();
		//We are copying from src to shared memory.
		int nextRow2 = base_global_row+curRow+2;
		bool legalRow2 = nextRow2<M;
		k = base_global_col+nextRow2*N+t;

		shared_mem [north*(blockDim.x+2)+t] =(legalRow2&&legalCol)?src[k]:0.0;

		if((t<2)&&legalColn&&legalRow2){
			shared_mem[north*(blockDim.x+2)+t+blockDim.x]=src[k+blockDim.x];
		}	
		
		center = ROTATE_UP(center,3);
		south  = ROTATE_UP(south,3);
		north  = ROTATE_UP(north,3);
		__syncthreads();
	}
#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.x==0)){
		printf("kernel finish!\n");
	}
#endif
}


__global__ void gpu_stencil2D_4pt_hack5_cp_rows(double * dst, double * shared_cols, double *shared_rows,int tile_y,int M, int N){


#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.x==0)){
		printf("copy rows begin!\n");
	}
#endif

	int base_global_row = (tile_y  * blockIdx.y ); 
	int base_global_col = blockDim.x*blockIdx.x;
	int base_global_idx = N*base_global_row + base_global_col ;
	int nextRow = base_global_row+1;
	bool legalNextRow = (nextRow<M)?1:0;
	int t = threadIdx.x;
	bool legalCurCol = (base_global_col + t)<N;
	int idx = (base_global_row/tile_y)*2*N + t+base_global_col;
	int idx_nextrow = idx + N;
	if(legalCurCol){
		shared_rows[idx] = dst[base_global_idx + t];
	}
	if(legalNextRow&&legalCurCol){
		shared_rows[idx_nextrow] = dst[base_global_idx + N+t];
	}
	__syncthreads();


#ifdef CUDA_CUDA_DEBUG
//	if(threadIdx.x==0){
//		printf("blockIdx.x = %d,blockIdx.y = %d\n",blockIdx.x,blockIdx.y);
//	}
//	if(blockIdx.y==1 && threadIdx.x==0){
//		printf("addr: %d\n",idx_nextrow);
//	}
	if(blockIdx.y==0 && blockIdx.x==2 && (t==0 || t==1)){	
		printf("addr:%d, val = %f\n", idx_nextrow,shared_rows[idx_nextrow]);
	}
#endif

#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.x==0)){
		printf("copy rows finish!\n");
	}
#endif
}

__global__ void gpu_stencil2D_4pt_hack5_cp_cols(double * dst, double * shared_cols, double *shared_rows,int tile_x,int tile_y, int M, int N){

#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.y==0)){
		printf("copy cols begin!\n");
	}
#endif

	int base_global_row = tile_y  * blockIdx.y; 
	int base_global_col = tile_x  * blockIdx.x;
	int base_global_idx = N*base_global_row + base_global_col ;
	int nextCol = base_global_col+1;
	bool legalNextCol = (nextCol<N);
	int t = threadIdx.y;
	int idx = 2*M*blockIdx.x + t + base_global_row;
	int idx_nextCol = idx + M ;
	bool legalCurRow = (base_global_row + t)<M;
	if(legalCurRow){
		shared_cols[idx] = dst[base_global_idx + t*N];
	}
	if(legalNextCol && legalCurRow){
		shared_cols[idx_nextCol] = dst[base_global_idx + t*N+1];
	}
	__syncthreads();


#ifdef CUDA_CUDA_DEBUG
//	if(threadIdx.y==0){
//		printf("blockDimy = %d\n",blockDim.y);
//	}
	if(blockIdx.x==1 && t<5){
		printf("addr: %d ,%f,\n",idx_nextCol,shared_cols[idx_nextCol]);
	}
#endif

#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.y==0)){
		printf("copy cols finish!\n");
	}
#endif
}

__global__ void gpu_stencil2D_4pt_hack5(double * dst, double * shared_cols, double *shared_rows,int tile_y,int M, int N)
{
#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.x==0)){
		printf("kernel begin!\n");
	}
#endif
	//Declaring the shared memory array for source
	extern	__shared__ double shared_mem[] ;

	//indexes
	int i,j, k,curRow;
                           //Cols   *  numRows/Tile * tileIndex  
	int base_global_row = ( tile_y * blockIdx.y ); 
	int base_global_col = blockDim.x*blockIdx.x;
	int base_global_idx = N*base_global_row + base_global_col ;
	int center = 1,north = 0,south = 2; //indexes for the current location in the shared memory
	int t = threadIdx.x;

	//copy the shared memory to fill the pipeline
	bool legalCol = (base_global_col + t )<N;
	bool legalCol1 = (base_global_col + t +1)<N;
	bool legalCol2= (base_global_col+t+2)<N;
	bool legalColn= (base_global_col+t+blockDim.x)<N;

	shared_mem [t] = shared_rows[base_global_col + t + blockIdx.y * N*2];
	if(t==2 || t==3){
		shared_mem [blockDim.x + t-2] = shared_rows[base_global_col+blockIdx.y*N*2+blockDim.x  + t-2];
	}
#ifdef CUDA_CUDA_DEBUG
//	if(blockIdx.y==0 && blockIdx.x==1 && (t==2||t==3)){
//		printf("addr: %d,val: %f\n",blockDim.x+t-2,shared_mem[blockDim.x+t-2]);
//	}
#endif
	for (i = 1 ; i < 1+HALO*2 ; i ++ ){
		k = base_global_idx+i*N+t;
		j = i*(blockDim.x+2) + t;
		bool legalRow = (base_global_row+i)<M;
		shared_mem [j+1] =legalRow?( legalCol1?dst[k+1]:0.0):0.0;
	
		if((t==1)&&legalColn&&legalRow){
			shared_mem[j+blockDim.x]=(blockIdx.x == (gridDim.x-1))?dst[k+blockDim.x]:shared_cols[blockIdx.x*2*M+3*M+i+base_global_row];
		}
		if(t==0){
			shared_mem[j] = shared_cols[blockIdx.x*2*M+base_global_row+i];
		}
	}
		
	__syncthreads();


#ifdef CUDA_CUDA_DEBUG
	if(blockIdx.y==0 && blockIdx.x==1 && (t==1||t==0)){
		printf("addr: %d,val: %f\n",blockDim.x+(blockDim.x+2)+t,shared_mem[blockDim.x + (blockDim.x+2)+t]);

		printf("addr: %d,val: %f\n",2*(blockDim.x+2)+blockDim.x+t,shared_mem[2*(blockDim.x+2) + blockDim.x+t]);
	}
#endif

	//Pipelined copy one row and process it
	for ( curRow = HALO; curRow < tile_y; curRow+=1 )
	{
		//Stencil computation
		//top + bottom + left + right

		j = threadIdx.x+HALO;
		bool legalRow1 =( base_global_row+curRow+1)<M;
		if((legalCol2)&&(legalRow1)){
			dst[base_global_idx + curRow*N + j] =(shared_mem[north*(blockDim.x+2)+j] + shared_mem[south*(blockDim.x+2)+j] + shared_mem[center*(blockDim.x+2)+j-1] + shared_mem[center*(blockDim.x+2)+j+1] )/5.5;
		}
		__syncthreads();
		
		//We are copying from src to shared memory.
		int nextRow2 = base_global_row+curRow+2;
		bool legalRow2 = nextRow2<M;
		k = base_global_col+nextRow2*N+t;

		shared_mem [north*(blockDim.x+2)+t+1] =(legalRow2&&legalCol1)?dst[k+1]:0.0;

		if((t==1)&&legalColn&&legalRow2){
			shared_mem[north*(blockDim.x+2)+t+blockDim.x]=(blockIdx.x == (gridDim.x-1))?dst[k+blockDim.x]:shared_cols[blockIdx.x*2*M+3*M+nextRow2];
		}	
		if((t==0)&&legalRow2){
			shared_mem[north*(blockDim.x+2)+t] = shared_cols[nextRow2+blockIdx.x*2*M];
		}
		
		center = ROTATE_UP(center,3);
		south  = ROTATE_UP(south,3);
		north  = ROTATE_UP(north,3);
		__syncthreads();
	}
	int lastRow1 = base_global_row+curRow+1;
	bool legalLastRow1 = (lastRow1)<M;
	
	if(legalLastRow1){
		
		shared_mem[south*(blockDim.x+2)+t] = shared_rows[base_global_col + t + blockIdx.y * N*2+N*3] ;

		if(t==2 || t==3){
			shared_mem [south*(blockDim.x+2) +blockDim.x + t-2] = shared_rows[base_global_col+blockIdx.y*N*2+3*N+blockDim.x  + t-2];
		}
#ifdef CUDA_CUDA_DEBUG
//		if((blockIdx.x==1)&&((t==2)||(t==3))&&(blockIdx.y==0)){ 
//			printf("addr: %d,val: %f\n",base_global_col+blockIdx.y*N*2+3*N+blockDim.x  + t-2 ,shared_mem[south*(blockDim.x+2)+blockDim.x+t-2]);
//		}
#endif
		__syncthreads();
	}
	if((legalCol2)&& legalLastRow1){
			dst[base_global_idx + curRow*N + j] =(shared_mem[north*(blockDim.x+2)+j] + shared_mem[south*(blockDim.x+2)+j] + shared_mem[center*(blockDim.x+2)+j-1] + shared_mem[center*(blockDim.x+2)+j+1] )/5.5;
	
	}


#ifdef CUDA_DARTS_DEBUG
	if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.x==0)){
		printf("kernel finish!\n");
	}
#endif
}


__global__ void gpu_stencil2D_4pt_hack2(double * dst, double * src, int M, int N)
{
//	printf("kernel begin!\n");
	//Declaring the shared memory array for source
	__shared__ double shared_mem[ 1 + HALO*2 ] [ GRID_TILE_X + HALO*2]; //1 is the row I am modifying
	//double * shSrc = shared_mem;

	//indexes
	int i, j, curRow;
                           //Cols   *  numRows/Tile * tileIndex  
	int base_global_idx = ( N ) * ( GRID_TILE_Y * blockIdx.y ) + GRID_TILE_X*blockIdx.x;
	
	int center = 1,north = 0,south = 2; //indexes for the current location in the shared memory

	//copy the shared memory to fill the pipeline
	for (i = 0 ; i < 1+HALO*2 ; i ++ )
		for (j = threadIdx.x ; j < GRID_TILE_X+2*HALO ; j+=blockDim.x)
		{
			shared_mem [i][j] = src[base_global_idx + i*N + j];
		}
	__syncthreads();
	//Pipelined copy one row and process it
	for ( curRow = HALO; curRow < GRID_TILE_Y; curRow+=1 )
	{
		//Stencil computation
		for (j = threadIdx.x + HALO ; j < GRID_TILE_X+HALO ; j+=blockDim.x)
		{
							//top             + bottom              + left                + right
			dst[base_global_idx + curRow*N + j] = (shared_mem[north][j] + shared_mem[south][j] + shared_mem[center][j-1] + shared_mem[center][j+1] )/5.5;
		}
		
		__syncthreads();
		//We are copying from dst to shared memory.
		for (j = threadIdx.x ; j < GRID_TILE_X+2*HALO ; j+=blockDim.x)
		{
			shared_mem [north][j] = src[base_global_idx + (curRow+2)*N + j];
		}
	
		center = ROTATE_UP(center,3);
		south  = ROTATE_UP(south,3);
		north = ROTATE_UP(north,3);
		__syncthreads();
	}

	//Dranning the pipeline
	for (j = threadIdx.x + HALO ; j < GRID_TILE_X+HALO ; j+=blockDim.x)
	{
							//top             + bottom              + left                + right
		dst[base_global_idx + curRow*N + j] = (shared_mem[north][j] + shared_mem[south][j] + shared_mem[center][j-1] + shared_mem[center][j+1] )/5.5;
	}
	__syncthreads();

//	printf("kernel finish!\n");
}



///**
//  * GPU Device kernel for the for 2D stencil
//  * First attempt during hackaton
//  * M = Rows, N = Cols INCLUDING HALOS
//  */
//__global__ void gpu_stencil2D_4pt_hack1(double * dst, double * src, int M, int N)
//{
//
//	//Declaring the shared memory array for source
//	__shared__ double shared_mem[GRID_TILE_Y + HALO*2 ] [ GRID_TILE_X + HALO*2];
//	//double * shSrc = shared_mem;
//
//	//indexes
//	int i, j;
//
//                           //Cols   *  numRows/Tile * tileIndex  
//	int base_global_idx = ( N ) * ( GRID_TILE_Y * blockIdx.y ) + GRID_TILE_X*blockIdx.x;
//
//	//We are copying from dst to shared memory.
//	for (i = 0 ; i < GRID_TILE_Y+2*HALO ; i ++ )
//		for (j = threadIdx.x ; j < GRID_TILE_X+2*HALO ; j+=blockDim.x)
//		{
//			shared_mem [i][j] = src[base_global_idx + i*N + j];
//		}
//
//	__syncthreads();
//
//	//Stencil computation
//	for (i = HALO ; i < GRID_TILE_Y+HALO ; i ++ )
//		for (j = threadIdx.x + HALO ; j < GRID_TILE_X+HALO ; j+=blockDim.x)
//		{
//			                                //top             + bottom              + left                + right
//			dst[base_global_idx + i*N + j] = (shared_mem[i-1][j] + shared_mem[i+1][j] + shared_mem[i][j-1] + shared_mem[i][j+1] )/5.5;
//		}
//
//	__syncthreads();
//}




/**
  * GPU Device kernel for the for 2D stencil
  * M = Rows, N = Cols
  */
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
	//Declaring the shared memory array for source
	extern __shared__ double shared_mem[];
	double * shSrc = shared_mem;

	//indexes
	int i, j;

	//neighbor's values 
	double north, south, east, west;



	//SharedMem Collumns Dimension
	int smColDim = HALO*2+blockDim.y*TILE_SIZE;
	int smRowDim = HALO*2+blockDim.x*TILE_SIZE;

	//Copying to shared memory

	//Inner part
	for ( i = 0 ; i < TILE_SIZE ; i++ )
	{
		for ( j = 0 ; j < TILE_SIZE ; j++ )
		{
			int globalIndex=HALO*N+blockIdx.x*blockDim.x*TILE_SIZE*N+threadIdx.x*TILE_SIZE*N+i*N+blockIdx.y*blockDim.y*TILE_SIZE+threadIdx.y*TILE_SIZE+j+HALO;
			int shMemIndex=HALO*smColDim+threadIdx.x*smColDim*TILE_SIZE+i*smColDim+HALO+threadIdx.y*TILE_SIZE+j;
			shSrc[shMemIndex]=src[globalIndex];
		}
	}

	//Halos

	if (threadIdx.x == 0 && threadIdx.y == 0 ) 
	{

		int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;
		//For Bottom and top row
		for ( i = 0 ; i < HALO ; i++ )
		{
			for ( j = 0 ; j < smColDim ; j++ )
			{
				indexTopHalo = (blockIdx.x*blockDim.x*TILE_SIZE+i)*N + (blockIdx.y*blockDim.y*TILE_SIZE) + j;
				indexBottomHalo = (HALO + (blockIdx.x+1)*blockDim.x*TILE_SIZE)*N + (blockIdx.y*blockDim.y*TILE_SIZE)+j;
				shSrc[i*smColDim+j] = src[indexTopHalo];
				shSrc[(HALO+blockDim.x*TILE_SIZE+i)*smColDim + j] = src[indexBottomHalo];
			}
		}
		
		//For right and left Columns
		for ( i = 0 ; i < HALO ; i++ )
		{
			for ( j = 0 ; j < smRowDim-HALO*2; j ++ )
			{
				indexLeftHalo = (HALO+blockIdx.x*blockDim.x*TILE_SIZE+j)*N + (blockIdx.y*blockDim.y*TILE_SIZE)+i;
				indexRightHalo = (HALO+blockIdx.x*blockDim.x*TILE_SIZE+j)*N + ((blockIdx.y+1)*blockDim.y*TILE_SIZE)+HALO+i;
				shSrc[(HALO+j)*smColDim+i] = src[indexLeftHalo];
				shSrc[(HALO+j+1)*smColDim-HALO+i] = src[indexRightHalo];
			}
		}
	}

	__syncthreads();



	for ( i = 0 ; i < TILE_SIZE ; i++ )
	{
		for ( j = 0 ; j < TILE_SIZE ; j++ )
		{
			int globalIndex=HALO*N+blockIdx.x*blockDim.x*TILE_SIZE*N+threadIdx.x*TILE_SIZE*N+i*N+blockIdx.y*blockDim.y*TILE_SIZE+threadIdx.y*TILE_SIZE+j+HALO;
			int shMemIndex=HALO*smColDim+threadIdx.x*smColDim*TILE_SIZE+i*smColDim+HALO+threadIdx.y*TILE_SIZE+j;


			//Getting the neighbohrs
			north = shSrc[shMemIndex-smColDim];
			south = shSrc[shMemIndex+smColDim];
			east  = shSrc[shMemIndex+1];
			west  = shSrc[shMemIndex-1];
			//Real Stencil operation
			dst[globalIndex] = ( north + south + east + west )/5.5;
//			dst[globalIndex] = ( north + south + east + west )/4;
		}
	}

	__syncthreads();
}




/**
 *  Naïve 4pt stencil code for 2D arrays. 
 */
void
stencil2D4pt ( double* __restrict__ dst,    double* __restrict__ src, 
               const size_t     n_rows, const size_t     n_cols,
               const size_t     n_tsteps )
{
    typedef double (*Array2D)[n_cols];
    volatile Array2D DST = (Array2D) dst,
            SRC = (Array2D) src;
    for (size_t ts = 0; ts < n_tsteps; ++ts) {
        for (size_t i = 1; i < n_rows-1; ++i) {
            for (size_t j = 1; j < n_cols-1; ++j) {
                DST[i][j] = (SRC[i-1][j] + SRC[i+1][j] + SRC[i][j-1] + SRC[i][j+1])/5.5;
            }
        }
        SWAP_PTR(&DST,&SRC);
    }
}

//extern "C"
//void
//stencil2D4pt_gpu( double * __restrict__ dst, double* __restrict__ src,
//		  const size_t M, const size_t N, 
//		  const size_t NUM_ITERATIONS)//M Rows by N Columns
//{
//		
//	double size = sizeof(double) * M * N;
//
//	//device memory allocation
//	double * d_dst, * d_src;
//	hipMalloc( (void **) &d_dst, size);
//	hipMalloc( (void **) &d_src, size);
//	
//	//dimmensions for indexes
//	// TODO the -2 is to remove the borders
//	dim3 dimBlock(MAX_BLOCK_DIM,MAX_BLOCK_DIM);
//	int gridx = (N-2)/(MAX_BLOCK_DIM*TILE_SIZE) + (((N-2)%(MAX_BLOCK_DIM*TILE_SIZE) == 0)? 0:1 ) ;
//	int gridy = (M-2)/(MAX_BLOCK_DIM*TILE_SIZE) + (((M-2)%(MAX_BLOCK_DIM*TILE_SIZE) == 0)? 0:1 ) ;
//	dim3 dimGrid(gridx,gridy);
//
//	//Shared memory size = inside + halo
//	int shMemSize=MAX_BLOCK_DIM*TILE_SIZE*MAX_BLOCK_DIM*TILE_SIZE*sizeof(double)+(HALO*MAX_BLOCK_DIM*TILE_SIZE+HALO*HALO)*4*sizeof(double);
//	
//	//Hackaton dimensions
//	dim3 dimGrid_hack1((N-HALO*2)/GRID_TILE_X,(M-HALO*2)/GRID_TILE_Y);
//
//	//Copying the device memory
//	hipMemcpy(d_src, src, size, hipMemcpyHostToDevice);
//	hipMemcpy(d_dst, dst, size, hipMemcpyHostToDevice);
//
//	//printf("CUDA Stencil Code running... cycles = %d. dim = %d by %d \n",NUM_ITERATIONS,M,N);
//	
//    int time_step = NUM_ITERATIONS;
//
//    while (time_step-- > 0) 
//    {
//    	//gpu_stencil2D_4pt<<<dimGrid,dimBlock,shMemSize>>>(d_dst,d_src,M,N);
//		//gpu_stencil2D_4pt_hack1<<<dimGrid_hack1,NUM_THREADS>>>(d_dst,d_src,M,N); //JOSE Hackathon!
//		//printf("before: d_src[10] = %ld",d_src[10]);
//
//		gpu_stencil2D_4pt_hack2<<<dimGrid_hack1,NUM_THREADS>>>(d_dst,d_src,M,N);
//		//Inline swapping.
//		
//		//printf("after: d_src[10] = %ld",d_src[10]);
//		double * temp;
//		if ( NUM_ITERATIONS%2 ==0 || time_step !=0)
//		{
//			temp=d_src;
//			d_src=d_dst;
//			d_dst=temp;
//		}
//	}
//	
//	
//	//Copying memory back from device to DRAM
//	//hipMemcpy(src, d_src, size, hipMemcpyDeviceToHost);
//	hipMemcpy(dst, d_dst, size, hipMemcpyDeviceToHost);
//	hipMemcpy(src, d_src, size, hipMemcpyDeviceToHost);
//	
//	//Free device memory
//	hipFree(d_src); hipFree(d_dst);
//}




//void*
//stencil_run(void* arg)
//{
//    stencil_t* stencil = (stencil_t*)arg;
//    STENCIL_COMPUTE(stencil->stencil,stencil->arg);
//    return NULL;
//}


void gpu_kernel4(dim3 dimGrid,dim3 dimBlock,double * d_dst, double * d_src, int M, int N){
		int sharedMemSize = sizeof(double)*(1+HALO*2)*(dimBlock.x+2);
#ifdef CUDA_DARTS_DEBUG
		printf("sharedMemSize: %d B, total sharedMemSize: %d B\n",sharedMemSize, sharedMemSize*dimGrid.x*dimGrid.y);
#endif
		gpu_stencil2D_4pt_hack4<<<dimGrid,dimBlock,sharedMemSize>>>(d_dst,d_src,M,N);
#ifdef CUDA_DARTS_DEBUG
		printf("gpu kernel return to host, but kernel haven't finished!\n");
#endif

}
void gpu_kernel5(dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedCols, double * sharedRows, int tile_y,int M, int N){
		int sharedMemSize = sizeof(double)*(1+HALO*2)*(dimBlock.x+2);
#ifdef CUDA_DARTS_DEBUG
		printf("sharedMemSize: %d B, total sharedMemSize: %d B\n",sharedMemSize, sharedMemSize*dimGrid.x*dimGrid.y);
#endif
		gpu_stencil2D_4pt_hack5<<<dimGrid,dimBlock,sharedMemSize>>>(d_dst,sharedCols,sharedRows,tile_y,M,N);
#ifdef CUDA_DARTS_DEBUG
		printf("gpu kernel return to host, but kernel haven't finished!\n");
#endif

}


void gpu_kernel5_cp_rows(dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedCols, double * sharedRows, int tile_y,int M, int N){

		gpu_stencil2D_4pt_hack5_cp_rows<<<dimGrid,dimBlock>>>(d_dst,sharedCols,sharedRows,tile_y,M,N);
}

void gpu_kernel5_cp_cols(dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedCols, double * sharedRows, int tile_x,int tile_y,int M, int N){

		gpu_stencil2D_4pt_hack5_cp_cols<<<dimGrid,dimBlock>>>(d_dst,sharedCols,sharedRows,tile_x,tile_y,M,N);
}


void gpu_kernel5_stream(hipStream_t &stream, dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedCols, double * sharedRows, int tile_y,int M, int N){
		int sharedMemSize = sizeof(double)*(1+HALO*2)*(dimBlock.x+2);
#ifdef CUDA_DARTS_DEBUG
		printf("Kernel5 stream: sharedMemSize: %d B, total sharedMemSize: %d B\n",sharedMemSize, sharedMemSize*dimGrid.x*dimGrid.y);
#endif

        gpu_stencil2D_4pt_hack5<<<dimGrid,dimBlock,sharedMemSize,stream>>>(d_dst,sharedCols,sharedRows,tile_y,M,N);
#ifdef CUDA_DARTS_DEBUG
		printf("gpu kernel return to host, but kernel haven't finished!\n");
#endif


}


void gpu_kernel5_stream_cp_rows(hipStream_t &stream ,dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedCols, double * sharedRows, int tile_y,int M, int N){

#ifdef CUDA_DARTS_DEBUG
		printf("Kernel5 stream: copy Rows. \n");
#endif
        gpu_stencil2D_4pt_hack5_cp_rows<<<dimGrid,dimBlock,0,stream>>>(d_dst,sharedCols,sharedRows,tile_y,M,N);

}

void gpu_kernel5_stream_cp_cols(hipStream_t &stream,dim3 dimGrid,dim3 dimBlock,double * d_dst, double * sharedCols, double * sharedRows, int tile_x,int tile_y,int M, int N){

#ifdef CUDA_DARTS_DEBUG
		printf("Kernel5 stream: copy cols. \n");
#endif
        gpu_stencil2D_4pt_hack5_cp_cols<<<dimGrid,dimBlock,0,stream>>>(d_dst,sharedCols,sharedRows,tile_x,tile_y,M,N);

}

void gpu_kernel1(dim3 dimGrid_hack1,double * d_dst, double * d_src, int M, int N){
		gpu_stencil2D_4pt_hack2<<<dimGrid_hack1,NUM_THREADS>>>(d_dst,d_src,M,N);
}

void gpu_kernel3(hipStream_t &stream,dim3 dimGrid_hack1,double * d_dst, double * d_src, int M, int N){
		gpu_stencil2D_4pt_hack2<<<dimGrid_hack1,NUM_THREADS,0,stream>>>(d_dst,d_src,M,N);

}

void gpu_kernel2(dim3 dimGrid_hack1,double *dst, double *src, double size, size_t ts, double * d_dst, double * d_src, int M, int N){
	double * tmp;
	while (--ts!=0){
		printf("ts:%ld \n", ts);
		gpu_stencil2D_4pt_hack2<<<dimGrid_hack1,NUM_THREADS>>>(d_dst,d_src,M,N);
		tmp = d_src;
		d_src = d_dst;
		d_dst=tmp;
	}
}
bool checkGpu(hipStream_t *stream, size_t n){
    for (size_t i=0;i<n;++i){
  
#ifdef CUDA_DARTS_DEBUG
		printf("checkGpu: %d \n",i);
#endif
        if (hipSuccess != hipStreamQuery(stream[i]))
            return false;
    }
    return true;
}


extern "C"
void
stencil2D4pt_gpu( double * __restrict__ h_dst, double* __restrict__ h_src, const size_t nRows, const size_t nCols, const size_t timestep)//M Rows by N Columns
{


	double *d_dst ;
	double *d_sharedCols ;
	double *d_sharedRows ;
	double d_size = sizeof(double) * nRows * nCols;
	int64_t d_size_sharedCols ;
	int64_t d_size_sharedRows ;
	
	size_t gpuMemMax = 0;
	size_t gpu_mem_total_t = 0;
	size_t gpu_mem_avail_t = 0;
	size_t gpu_mem_valid_t = 0;
	hipMemGetInfo(&gpu_mem_avail_t,&gpu_mem_total_t);
	gpu_mem_valid_t = gpu_mem_avail_t - XMB;
    gpuMemMax =(2*GB)> gpu_mem_valid_t?gpu_mem_avail_t: 2*GB;

    int tile_y = GRID_TILE_Y;
    int tile_x = NUM_THREADS;
   
	d_size_sharedCols = sizeof(double)*nRows* (std::ceil(1.0*nCols/NUM_THREADS)) *2;
	d_size_sharedRows = sizeof(double)*nCols*(std::ceil(1.0*nRows/tile_y))*2;
	double req_size = sizeof(double)* nRows*nCols +  d_size_sharedCols + d_size_sharedRows  ;

	uint64_t nRowsGpu;
	uint64_t nRowsGpuMax;
    
	int nGPU = 1;
	uint64_t gpuPos = 0;

	hipError_t err1,err2,err3,err4,err5;
	
	if(req_size<gpuMemMax){
		nGPU = 1;
		nRowsGpu = nRows;

		int blockDimx =( (nCols-2)>NUM_THREADS)?NUM_THREADS:(nCols-2);
		int blockDimy = 1;
		int gridDimx = std::ceil(1.0*(nCols-2)/blockDimx);
		int gridDimy = std::ceil(1.0*nRowsGpu/tile_y); //GRID_TILE_Y=10, it needs to change.
		
		dim3 dimGrid(gridDimx,gridDimy);
		dim3 dimBlock(blockDimx,blockDimy);
		d_size_sharedCols = sizeof(double)*nRowsGpu*gridDimx*2;
		d_size_sharedRows = sizeof(double)*nCols*gridDimy*2;

		err1 = hipMalloc( (void **) &d_dst, d_size);
		err2 = hipMalloc( (void **) &d_sharedCols, d_size_sharedCols);
		err3 = hipMalloc( (void **) &d_sharedRows, d_size_sharedRows);

#ifdef CUDA_ERROR_CHECKING
		if(err1!=hipSuccess){
			printf("GpuKernelWithAllTimeSteps: cuda malloc1: %s \n",hipGetErrorString(err1));
			exit(-1);
		}
		if(err2!=hipSuccess){
			printf("GpuKernelWithAllTimeSteps: cuda malloc2: %s \n",hipGetErrorString(err2));
			exit(-1);
		}

		if(err3!=hipSuccess){
			printf("GpuKernelWithAllTimeSteps: cuda malloc3: %s \n ",hipGetErrorString(err3));
			exit(-1);
		}

#endif
		size_t pos1 = gpuPos*nCols;	

		err4 = hipMemcpy(d_dst, h_src+pos1, d_size, hipMemcpyHostToDevice);

#ifdef CUDA_ERROR_CHECKING
		if(err4!=hipSuccess){
		
			printf("GpuKernelWithAllTimeSteps: cuda memcpyHostToDevice d_dst: %s \n ",hipGetErrorString(err4));
			exit(-1);
		}
#endif

		int blockDimx_rows =( nCols>NUM_THREADS)?NUM_THREADS:nCols;
		int blockDimy_rows = 1;
		int gridDimx_rows = std::ceil(1.0*nCols/blockDimx_rows);
		int gridDimy_rows = std::ceil(1.0*nRowsGpu/tile_y);
	
		int blockDimx_cols = 1 ;
		int blockDimy_cols = (nRowsGpu>NUM_THREADS)?NUM_THREADS:nRows;
		int gridDimx_cols = gridDimx;
		int gridDimy_cols = std::ceil(1.0*nRowsGpu/blockDimy_cols);


		dim3 dimGrid_rows(gridDimx_rows,gridDimy_rows);
		dim3 dimBlock_rows(blockDimx_rows,blockDimy_rows);
	
		dim3 dimGrid_cols(gridDimx_cols,gridDimy_cols);
		dim3 dimBlock_cols(blockDimx_cols,blockDimy_cols);

		size_t ts = timestep; 
	
		while(ts-- >0){
			gpu_kernel5_cp_rows(dimGrid_rows,dimBlock_rows,d_dst, d_sharedCols, d_sharedRows, tile_y,nRowsGpu, nCols);
			gpu_kernel5_cp_cols(dimGrid_cols,dimBlock_cols,d_dst, d_sharedCols, d_sharedRows, tile_x,tile_x,nRowsGpu, nCols);
			gpu_kernel5(dimGrid,dimBlock,d_dst,d_sharedCols,d_sharedRows,tile_y,nRowsGpu,nCols);
		}
		
		err5 = hipDeviceSynchronize();

#ifdef CUDA_ERROR_CHECKING
		if(err5!=hipSuccess){
			printf("GpuKernelWithAllTimeSteps: cuda deviceSynchronize:  %s \n ",hipGetErrorString(err5));
			exit(-1);
		}
#endif
	
#ifdef VERIFICATION
        if(timestep%2==0){
			SWAP_PTR(&h_dst ,&h_src);
        }
#endif
	
        err1=hipMemcpy(h_dst+pos1, d_dst,d_size, hipMemcpyDeviceToHost);

#ifdef CUDA_ERROR_CHECKING
		if(err1!=hipSuccess){
			printf(" GpuKernelWithAllTimeSteps: cuda memcpyDeviceToHost:  %s \n ",hipGetErrorString(err1));
			
			exit(-1);
		}
#endif

	    err1 = hipFree(d_dst);
#ifdef CUDA_ERROR_CHECKING
		if(err1!=hipSuccess){
			printf(" GpuKernelWithAllTimeSteps: cuda free d_dst:  %s \n ",hipGetErrorString(err1));
			
			exit(-1);
		}
#endif		
	
	
	
	}else{
		nGPU = std::ceil(req_size/gpuMemMax); 
		nRowsGpu = nRows;
		int nStream = 4 ;
		hipStream_t *stream ;
		stream = new hipStream_t[nStream];
		for(int i=0;i<nStream;++i){
			hipStreamCreate(&stream[i]);
		}

		int vnStream = nStream*nGPU;
		int nTile_y = nRows/(tile_y * vnStream);
		
		int chunk = nTile_y*tile_y;
		int chunk2= chunk+2;
		
		int nRowsGpuBlock = nStream*chunk2 + nRows-nGPU*nStream*chunk;
		int64_t nRowsGpuStream;
		int64_t d_size_stream;
		
		int blockDimx =( (nCols-2)>NUM_THREADS)?NUM_THREADS:(nCols-2);
		int blockDimy = 1;
		int gridDimx = std::ceil(1.0*(nCols-2)/blockDimx);
		int gridDimy = std::ceil(1.0*nRowsGpuBlock/tile_y); 
		
		dim3 dimBlock(blockDimx,blockDimy);
		//dim3 dimGrid(gridDimx,gridDimy);
		
		d_size = sizeof(double)*nRowsGpuBlock*nCols;
		d_size_sharedCols = sizeof(double) * nRowsGpuBlock*gridDimx*2 ;
		d_size_sharedRows = sizeof(double) * nCols* gridDimy*2;

		err1 = hipMalloc( (void **) &d_dst, d_size);
		err2 = hipMalloc( (void **) &d_sharedCols, d_size_sharedCols);
		err3 = hipMalloc( (void **) &d_sharedRows, d_size_sharedRows);


#ifdef CUDA_ERROR_CHECKING
	    if(err1!=hipSuccess){
	        
			printf("GpuKernelPureGpuWithStreams: cuda malloc d_dst:  %s \n ",hipGetErrorString(err1));
			exit(-1);
	    }
	    if(err2!=hipSuccess){
			printf("GpuKernelPureGpuWithStreams: cuda mallock d_sharedRows :  %s \n ",hipGetErrorString(err2));
			exit(-1);
	    }
	    
	    if(err3!=hipSuccess){
			printf("GpuKernelPureGpuWithStreams: cuda mallock d_sharedCols :  %s \n ",hipGetErrorString(err3));
			exit(-1);
	    }
#endif
    
	    int blockDimx_rows =( nCols>NUM_THREADS)?NUM_THREADS:nCols;
	    int blockDimy_rows = 1;
	    int gridDimx_rows = std::ceil(1.0*nCols/blockDimx_rows);
	    int gridDimy_rows; 
	    
	    int blockDimx_cols = 1 ;
	    int blockDimy_cols;
	    int gridDimx_cols = gridDimx;  
	    int gridDimy_cols; 
	    
	    dim3 dimBlock_rows(blockDimx_rows,blockDimy_rows);
	    //dim3 dimGrid_rows(gridDimx_rows,gridDimy_rows);
	
	    uint64_t h_pos;
	    uint64_t d_pos;
	    uint64_t pos0 = gpuPos*nCols;
		
		size_t ts = timestep;

	    while(ts-- >0){
	        for (size_t i = 0; i<nGPU; ++i){
	            for (size_t j =0; j<nStream;++j){
	                int ps = i*nStream+j;
	                nRowsGpuStream = ((i==(nGPU-1))&&(j==(nStream-1)))? (nRows-ps*chunk) :chunk2;
	                h_pos = pos0+ps*chunk*nCols;
	                d_pos = j*chunk2*nCols; 
	
#ifdef CUDA_ERROR_CHECKING
	                err3 = hipGetLastError();
	                if(hipSuccess != err3){
						printf("GpuKernelPureGpuWithStreams multiple streams: kernel5 stream error :  %s \n ",hipGetErrorString(err3));
						exit(-1);
	                }
#endif
	
	                d_size_stream = sizeof(double)*nCols*nRowsGpuStream;
	                err1 = hipMemcpyAsync(d_dst+d_pos, h_src+h_pos, d_size_stream, hipMemcpyHostToDevice,stream[j]);
#ifdef CUDA_ERROR_CHECKING
	                if(err1!=hipSuccess){
						printf("GpuKernelWithStream multiple streams: cuda MemcpyAsync from host to device :  %s \n ",hipGetErrorString(err1));
						
						exit(-1);
	                }
#endif
	            
	                gridDimy_rows = std::ceil(1.0*nRowsGpuStream/tile_y);
	                dim3 dimGrid_rows(gridDimx_rows,gridDimy_rows);
	                gpu_kernel5_stream_cp_rows(stream[j],dimGrid_rows,dimBlock_rows,d_dst+d_pos , d_sharedCols, d_sharedRows+j*nTile_y*2*nCols, tile_y,nRowsGpuStream, nCols);
	            
#ifdef CUDA_ERROR_CHECKING
	                err3 = hipGetLastError();
	                if(hipSuccess != err3){
						printf("GpuKernelWithStream multiple streams: kernel5 cuda cp rows :  %s \n ",hipGetErrorString(err3));
						exit(-1);
	                }
#endif
	            
	                blockDimy_cols = (nRowsGpuStream>NUM_THREADS)?NUM_THREADS:nRowsGpuStream;
	                gridDimy_cols = std::ceil(1.0*nRowsGpuStream/blockDimy_cols);
	        	    dim3 dimBlock_cols(blockDimx_cols,blockDimy_cols);
	                dim3 dimGrid_cols(gridDimx_cols,gridDimy_cols);
	                int addrCol = j*chunk2*2*gridDimx_cols; 
	                gpu_kernel5_stream_cp_cols(stream[j],dimGrid_cols,dimBlock_cols,d_dst+d_pos, d_sharedCols+addrCol, d_sharedRows, tile_x,tile_x,nRowsGpuStream, nCols);
	
#ifdef CUDA_ERROR_CHECKING
	                err3 = hipGetLastError();
	                if(hipSuccess != err3){
						printf("GpuKernelWithStream multiple streams: kernel5 cuda cp cols :  %s \n ",hipGetErrorString(err3));
						exit(-1);
	                }
#endif
	            
		            int gridDimy_stream = std::ceil(1.0*nRowsGpuStream/tile_y);
		            dim3 dimGrid_stream(gridDimx,gridDimy_stream);
	                gpu_kernel5_stream(stream[j] ,dimGrid_stream,dimBlock,d_dst+d_pos,d_sharedCols+addrCol,d_sharedRows+j*nTile_y*2*nCols,tile_y,nRowsGpuStream,nCols);
	            
#ifdef CUDA_ERROR_CHECKING
	                err3 = hipGetLastError();
	                if(hipSuccess != err3){
						printf("GpuKernelWithStream multiple streams: kernel5 cuda computation :  %s \n ",hipGetErrorString(err3));
						exit(-1);
	                }
#endif
	            
		            err3=hipMemcpyAsync(h_dst+h_pos+nCols, d_dst+d_pos+nCols,d_size_stream-(nCols)*2*sizeof(double), hipMemcpyDeviceToHost,stream[j]);
	
#ifdef CUDA_ERROR_CHECKING
	                err3 = hipGetLastError();
	                if(hipSuccess != err3){
						printf("GpuKernelWithStream multiple streams: kernel5 Asyn Memory copy from device to host :  %s \n ",hipGetErrorString(err3));
						exit(-1);
	                }
#endif
	            }
	        }
	   
		    err4 = hipDeviceSynchronize();
#ifdef CUDA_ERROR_CHECKING
		    if(err4!=hipSuccess){
				printf("GpuKernelPureGpuWithStreams: cuda deviceSynchronize: %s \n ",hipGetErrorString(err4));
				exit(-1);
		    }
#endif
		    SWAP_PTR(&h_dst ,&h_src);
	    }
	
		err4 = hipDeviceSynchronize();
	    err1 = hipFree(d_dst);
		err2 = hipFree(d_sharedRows);
	    err3 = hipFree(d_sharedCols);

#ifdef CUDA_ERROR_CHECKING
		if(err4!=hipSuccess){
			printf("GpuKernelPureGpuWithStreams: cuda deviceSynchronize :  %s \n ",hipGetErrorString(err4));
			exit(-1);
		}
	
	    if(err1!=hipSuccess){
			printf("GpuKernelPureGpuWithStreams: cuda memcpy free d_dst :  %s \n ",hipGetErrorString(err1));
			exit(-1);
	    }
	
	    if(err2!=hipSuccess){
			printf("GpuKernelPureGpuWithStreams: cuda memcpy free d_sharedRows :  %s \n ",hipGetErrorString(err2));
			exit(-1);
	    }
	
	    if(err3!=hipSuccess){
			printf("GpuKernelPureGpuWithStreams: cuda memcpy free d_sharedCols:  %s \n ",hipGetErrorString(err3));
			exit(-1);
	    }
#endif
		if(timestep%2){
			SWAP_PTR(&h_dst ,&h_src);
		}

		for(int i=0;i<nStream;++i){
			hipStreamDestroy(stream[i]);
		}
		delete [] stream;

	}



}
